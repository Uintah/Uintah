/*
 * The MIT License
 *
 * Copyright (c) 1997-2017 The University of Utah
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to
 * deal in the Software without restriction, including without limitation the
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 */
 
#include <Core/Util/GPU.h>
 
namespace Uintah {

//______________________________________________________________________
//  Returns true if threadID and blockID are 0.
//  Useful in conditional statements for limiting output.
//
__device__
bool
isThread0_Blk0(){
  int blockID  = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z; 
  int threadID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
  
  bool test (blockID == 0 && threadID == 0);
  return test;
}

//______________________________________________________________________
//  Returns true if threadID = 0 for this block
//  Useful in conditional statements for limiting output.
//
__device__
bool
isThread0(){
  int threadID = threadIdx.x +  threadIdx.y +  threadIdx.z;
  bool test (threadID == 0 );
  return test;
}

//______________________________________________________________________
// Output the threadID
//
__device__
void 
printThread(){ 
  int threadID = threadIdx.x +  threadIdx.y +  threadIdx.z;
  printf( "Thread [%i,%i,%i], ID: %i\n", threadIdx.x,threadIdx.y,threadIdx.z, threadID);
}

//______________________________________________________________________
// Output the blockID
//
__device__
void 
printBlock(){ 
  int blockID  = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
  printf( "Block  [%i,%i,%i], ID: %i\n", blockIdx.x,blockIdx.y,blockIdx.z, blockID);
}

}  // end namespace Uintah
