#include "hip/hip_runtime.h"
/*
 * The MIT License
 *
 * Copyright (c) 1997-2014 The University of Utah
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to
 * deal in the Software without restriction, including without limitation the
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 */
/* GPU DataWarehouse device&host access*/

#include <CCA/Components/Schedulers/GPUDataWarehouse.h>
#ifndef __CUDA_ARCH__
#include <string.h>
#endif
//#include <Core/Util/GPU.h>

// This belongs in GPU.h
__device__ bool isThread0_Blk0(){
  int blockID  = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z; 
  int threadID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
  
  bool test = (blockID == 0 && threadID == 0);
  return test;
}

namespace Uintah {

//______________________________________________________________________
//
HOST_DEVICE void
GPUDataWarehouse::get(const GPUGridVariableBase& var, char const* name, int patchID, int matlIndex)
{
  GPUDataWarehouse::dataItem* item = getItem(name, patchID, matlIndex);
  if (item){
    var.setArray3(item->var_offset, item->var_size, item->var_ptr);
  }else{    
#ifdef __CUDA_ARCH__
    int numThreads = blockDim.x*blockDim.y*blockDim.z;
    int threadID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
    
    int i=threadID;
    while(i<d_numItems){
      printf( "   Available labels: \"%s\"\n", d_varDB[i].label );
      i=i+numThreads;
    }
    if( isThread0_Blk0() ) {
      printf("  ERROR: GPUDataWarehouse::get( \"%s\", patchID: %i, matl: %i )  unknown variable\n\n", name, patchID, matlIndex);
      assert(0);
    }

#else
    printf("\t ERROR: GPUDataWarehouse::get( \"%s\", patchID: %i, matl: %i )  unknown variable\n", name, patchID, matlIndex);
#endif
  }
}

//______________________________________________________________________
//
HOST_DEVICE void
GPUDataWarehouse::getModifiable(GPUGridVariableBase& var, char const* name, int patchID, int matlIndex)
{
  GPUDataWarehouse::dataItem* item=getItem(name, patchID, matlIndex);
  if (item) {
    var.setArray3(item->var_offset, item->var_size, item->var_ptr);
  }else{
#ifdef __CUDA_ARCH__
    int numThreads = blockDim.x*blockDim.y*blockDim.z;
    int threadID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
    
    
    int i=threadID;
    while(i<d_numItems){
      printf( "  Available Labels: \"%s\"\n", d_varDB[i].label );
      i=i+numThreads;
    }
    if( isThread0_Blk0() ) {
      printf("  ERROR: GPUDataWarehouse::getModifiable( \"%s\", patchID: %i, matl: %i )  unknown variable\n\n", name, patchID, matlIndex);
      assert(0);
    }

#else
    printf("  ERROR: GPUDataWarehouse::getModifiable( \"%s\", patchID: %i, matl: %i )  unknown variable\n", name, patchID, matlIndex);
#endif
  }
}

//______________________________________________________________________
//
HOST_DEVICE void 
GPUDataWarehouse::put(GPUGridVariableBase &var, char const* name, int patchID, int matlIndex, bool overWrite)
{
#ifdef __CUDA_ARCH__  // need to limit output
  printf("ERROR:\nGPUDataWarehouse::put( %s )  You cannot use this on the device.  All memory should be allocated on the CPU with hipMalloc\n",name);
#else
  
  //__________________________________
  //cpu code 
  if (d_numItems==MAX_ITEM) {
    printf("out of GPUDataWarehouse space");
    exit(-1);
  }
  
  int i=d_numItems;
  d_numItems++; 
  strncpy(d_varDB[i].label, name, MAX_NAME);
  d_varDB[i].domainID  = patchID;
  d_varDB[i].matlIndex = matlIndex;
  var.getArray3(d_varDB[i].var_offset, d_varDB[i].var_size, d_varDB[i].var_ptr);
  
  if (d_debug){
    printf("host put \"%s\" (patch: %d) loc %p into GPUDW %p on device %d, size [%d,%d,%d]\n", name, patchID, d_varDB[i].var_ptr, d_device_copy, d_device_id, d_varDB[i].var_size.x, d_varDB[i].var_size.y, d_varDB[i].var_size.z);
  }
  d_dirty=true;
#endif
}

//______________________________________________________________________
//
HOST_DEVICE void 
GPUDataWarehouse::allocateAndPut(GPUGridVariableBase &var, char const* name, int patchID, int matlIndex, int3 low, int3 high)
{
#ifdef __CUDA_ARCH__  // need to limit output
  printf("ERROR:\nGPUDataWarehouse::allocateAndPut( %s )  You cannot use this on the device.  All memory should be allocated on the CPU with hipMalloc\n",name);
#else
  //__________________________________
  //  cpu code
  hipError_t retVal;
  int3 size   = make_int3(high.x-low.x, high.y-low.y, high.z-low.z);
  int3 offset = low;
  void* addr  = NULL;
  
  var.setArray3(offset, size, addr);
  CUDA_RT_SAFE_CALL( retVal = hipSetDevice(d_device_id) );
  CUDA_RT_SAFE_CALL( retVal = hipMalloc(&addr, var.getMemSize()) );
  
  if (d_debug && retVal == hipSuccess) {
    printf("hipMalloc for \"%s\", size %ld from (%d,%d,%d) to (%d,%d,%d) ", name, var.getMemSize(),
            low.x, low.y, low.z, high.x, high.y, high.z);
    printf(" at %p on device %d\n", addr, d_device_id);
  }
  
  var.setArray3(offset, size, addr);
  put(var, name, patchID, matlIndex);

#endif
}

//______________________________________________________________________
//
HOST_DEVICE void
GPUDataWarehouse::get(const GPUParticleVariableBase& var, char const* name, int patchID, int matlID)
{
  GPUDataWarehouse::dataItem* item = getItem(name, patchID, matlID);
  if (item){
    var.setData(item->num_elems, item->var_ptr);
  }else{
#ifdef __CUDA_ARCH__
    int numThreads = blockDim.x*blockDim.y*blockDim.z;
    int threadID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;

    int i=threadID;
    while(i<d_numItems){
      printf( "   Available labels: \"%s\"\n", d_varDB[i].label );
      i=i+numThreads;
    }
    if( isThread0_Blk0() ) {
      printf("  ERROR: GPUDataWarehouse::get( \"%s\", patchID: %i, matl: %i )  unknown variable\n\n", name, patchID, matlID);
      assert(0);
    }

#else
    printf("\t ERROR: GPUDataWarehouse::get( \"%s\", patchID: %i, matl: %i )  unknown variable\n", name, patchID, matlID);
#endif
  }
}

//______________________________________________________________________
//
HOST_DEVICE void
GPUDataWarehouse::getModifiable(GPUParticleVariableBase& var, char const* name, int patchID, int matlID)
{
  GPUDataWarehouse::dataItem* item = getItem(name, patchID, -1 /* matlID */);
  if (item) {
    var.setData(item->num_elems, item->var_ptr);
  } else {
#ifdef __CUDA_ARCH__
    int numThreads = blockDim.x*blockDim.y*blockDim.z;
    int threadID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;


    int i=threadID;
    while(i<d_numItems){
      printf( "  Available Labels: \"%s\"\n", d_varDB[i].label );
      i=i+numThreads;
    }
    if( isThread0_Blk0() ) {
      printf("  ERROR: GPUDataWarehouse::getModifiable( \"%s\", patchID: %i, matl: %i )  unknown variable\n\n", name, patchID, matlID);
      assert(0);
    }

#else
    printf("  ERROR: GPUDataWarehouse::getModifiable( \"%s\", patchID: %i, matl: %i )  unknown variable\n", name, patchID, matlID);
#endif
  }
}

//______________________________________________________________________
//
HOST_DEVICE void
GPUDataWarehouse::put(GPUParticleVariableBase& var, char const* name, int patchID, int matlIndex, bool overWrite)
{
#ifdef __CUDA_ARCH__  // need to limit output
  printf("ERROR:\nGPUDataWarehouse::put( %s )  You cannot use this on the device.  All memory should be allocated on the CPU with hipMalloc\n", name);
#else

  //__________________________________
  //cpu code
  if (d_numItems==MAX_ITEM) {
    printf("out of GPUDataWarehouse space");
    exit(-1);
  }

  int i=d_numItems;
  d_numItems++;
  strncpy(d_varDB[i].label, name, MAX_NAME);
  d_varDB[i].domainID  = patchID;
  d_varDB[i].matlIndex = -1; // matlIndex;

  var.getData(d_varDB[i].num_elems, d_varDB[i].var_ptr);

  if (d_debug){
    printf("host put \"%s\" (patch: %d) loc %p into GPUDW %p on device %d, size %lu\n", name, patchID, d_varDB[i].var_ptr, d_device_copy, d_device_id, d_varDB[i].num_elems);
  }
  d_dirty=true;
#endif
}

//______________________________________________________________________
//
HOST_DEVICE void
GPUDataWarehouse::allocateAndPut(GPUParticleVariableBase& var, char const* name, int patchID, int matlID, size_t numElems)
{
#ifdef __CUDA_ARCH__  // need to limit output
  printf("ERROR:\nGPUDataWarehouse::allocateAndPut( %s )  You cannot use this on the device.  All memory should be allocated on the CPU with hipMalloc()\n",name);
#else
  //__________________________________
  //  cpu code
  hipError_t retVal;
  size_t numVals = numElems;
  void* addr  = NULL;

  CUDA_RT_SAFE_CALL( retVal = hipSetDevice(d_device_id) );
  CUDA_RT_SAFE_CALL( retVal = hipMalloc(&addr, var.getMemSize()) );

  if (d_debug && retVal == hipSuccess) {
    printf("hipMalloc for \"%s\", size %ld", name, var.getMemSize());
    printf(" at %p on device %d\n", addr, d_device_id);
  }

  var.setData(numVals, addr);
  put(var, name, patchID, matlID);
#endif
}

//______________________________________________________________________
//
HOST_DEVICE void
GPUDataWarehouse::get(const GPUReductionVariableBase& var, char const* name, int patchID, int matlID)
{
  GPUDataWarehouse::dataItem* item = getItem(name, patchID, matlID);
  if (item){
    var.setData(item->num_elems, item->var_ptr);
  }else{
#ifdef __CUDA_ARCH__
    int numThreads = blockDim.x*blockDim.y*blockDim.z;
    int threadID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;

    int i=threadID;
    while(i<d_numItems){
      printf( "   Available labels: \"%s\"\n", d_varDB[i].label );
      i=i+numThreads;
    }
    if( isThread0_Blk0() ) {
      printf("  ERROR: GPUDataWarehouse::get( \"%s\", patchID: %i, matl: %i )  unknown variable\n\n", name, patchID, matlID);
      assert(0);
    }

#else
    printf("\t ERROR: GPUDataWarehouse::get( \"%s\", patchID: %i, matl: %i )  unknown variable\n", name, patchID, matlID);
#endif
  }
}

//______________________________________________________________________
//
HOST_DEVICE void
GPUDataWarehouse::getModifiable(GPUReductionVariableBase& var, char const* name, int patchID, int matlID)
{
  GPUDataWarehouse::dataItem* item = getItem(name, patchID, -1 /* matlID */);
  if (item) {
    var.setData(item->num_elems, item->var_ptr);
  } else {
#ifdef __CUDA_ARCH__
    int numThreads = blockDim.x*blockDim.y*blockDim.z;
    int threadID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;


    int i=threadID;
    while(i<d_numItems){
      printf( "  Available Labels: \"%s\"\n", d_varDB[i].label );
      i=i+numThreads;
    }
    if( isThread0_Blk0() ) {
      printf("  ERROR: GPUDataWarehouse::getModifiable( \"%s\", patchID: %i, matl: %i )  unknown variable\n\n", name, patchID, matlID);
      assert(0);
    }

#else
    printf("  ERROR: GPUDataWarehouse::getModifiable( \"%s\", patchID: %i, matl: %i )  unknown variable\n", name, patchID, matlID);
#endif
  }
}

//______________________________________________________________________
//
HOST_DEVICE void
GPUDataWarehouse::put(GPUReductionVariableBase& var, char const* name, int patchID, int matlIndex, bool overWrite)
{
#ifdef __CUDA_ARCH__  // need to limit output
  printf("ERROR:\nGPUDataWarehouse::put( %s )  You cannot use this on the device.  All memory should be allocated on the CPU with hipMalloc\n", name);
#else

  //__________________________________
  //cpu code
  if (d_numItems==MAX_ITEM) {
    printf("out of GPUDataWarehouse space");
    exit(-1);
  }

  int i=d_numItems;
  d_numItems++;
  strncpy(d_varDB[i].label, name, MAX_NAME);
  d_varDB[i].domainID  = patchID;
  d_varDB[i].matlIndex = -1; // matlIndex;

  var.getData(d_varDB[i].num_elems, d_varDB[i].var_ptr);

  if (d_debug){
    printf("host put \"%s\" (patch: %d) loc %p into GPUDW %p on device %d, size %lu\n", name, patchID, d_varDB[i].var_ptr, d_device_copy, d_device_id, d_varDB[i].num_elems);
  }
  d_dirty=true;
#endif
}

//______________________________________________________________________
//
HOST_DEVICE void
GPUDataWarehouse::allocateAndPut(GPUReductionVariableBase& var, char const* name, int patchID, int matlID, int numElems)
{
#ifdef __CUDA_ARCH__  // need to limit output
  printf("ERROR:\nGPUDataWarehouse::allocateAndPut( %s )  You cannot use this on the device.  All memory should be allocated on the CPU with hipMalloc()\n",name);
#else
  //__________________________________
  //  cpu code
  hipError_t retVal;
  size_t numVals = numElems;
  void* addr  = NULL;

  CUDA_RT_SAFE_CALL( retVal = hipSetDevice(d_device_id) );
  CUDA_RT_SAFE_CALL( retVal = hipMalloc(&addr, var.getMemSize()) );

  if (d_debug && retVal == hipSuccess) {
    printf("hipMalloc for \"%s\", size %ld", name, var.getMemSize());
    printf(" at %p on device %d\n", addr, d_device_id);
  }

  var.setData(numVals, addr);
  put(var, name, patchID, matlID);
#endif
}

//______________________________________________________________________
//
HOST_DEVICE GPUDataWarehouse::dataItem*
GPUDataWarehouse::getItem(char const* name, int patchID, int matlIndex)
{
#ifdef __CUDA_ARCH__
  __shared__ int index;
  int numThreads = blockDim.x*blockDim.y*blockDim.z;
  int blockID = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
  int threadID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
  int i=threadID;
  char const* s1 = name;
  __syncthreads();
  index = -1;

  if (d_debug && threadID == 0 && blockID==0) {
    printf("device getting item \"%s\" from GPUDW %p", name, this);
    printf("size (%d vars)\n Available labels:", d_numItems);
  }

  //sync before get
  __syncthreads();

  while(i<d_numItems){
    int strmatch=0;
    char* s2 = &(d_varDB[i].label[0]);
    while (!(strmatch = *(unsigned char *) s1 - *(unsigned char *) s2) && *s2) { //strcmp
      ++s1, ++s2;
    }

    if (strmatch==0 && d_varDB[i].domainID==patchID && d_varDB[i].matlIndex==matlIndex){
      index = i;
    }
    i=i+numThreads;
  }
  //sync before return;
  __syncthreads();

  if (index==-1) {
    return NULL;
  } else {
    return &d_varDB[index];
  }
#else
  //__________________________________
  // cpu code
  int i= 0;
  while(i<d_numItems){
    if (!strncmp(d_varDB[i].label, name, MAX_NAME) &&  d_varDB[i].domainID==patchID && d_varDB[i].matlIndex==matlIndex) {
      break;
    }
    i++;
  }

  if (i==d_numItems) {
    printf("ERROR:\nGPUDataWarehouse::get( %s ) host get unknown variable from GPUDataWarehouse", name);
    exit(-1);
  }

  if (d_debug){
    printf("host got \"%s\" loc %p from GPUDW %p on device %u\n", name, d_varDB[i].var_ptr, d_device_copy, d_device_id);
  }
  return &d_varDB[i];
#endif
}

//______________________________________________________________________
//
HOST_DEVICE bool
GPUDataWarehouse::exist(char const* name, int patchID, int matlID)
{
#ifdef __CUDA_ARCH__
  printf("exist() is only for framework code\n");
#else
  //__________________________________
  //  cpu code
  int i= 0;
  while(i<d_numItems){
    if (!strncmp(d_varDB[i].label, name, MAX_NAME) &&  d_varDB[i].domainID==patchID && d_varDB[i].matlIndex==matlID) {
      return true;
    }
    i++;
  }
#endif 
return false;
}

//______________________________________________________________________
//
HOST_DEVICE bool
GPUDataWarehouse::remove(char const* name, int patchID, int matlID)
{
#ifdef __CUDA_ARCH__
  printf("remove() is only for framework code\n");
#else
  int i= 0;
  while(i<d_numItems){
    if (!strncmp(d_varDB[i].label, name, MAX_NAME) &&  d_varDB[i].domainID==patchID && d_varDB[i].matlIndex==matlID) {
      hipError_t retVal;
      CUDA_RT_SAFE_CALL(retVal = hipFree(d_varDB[i].var_ptr));

      if (d_debug){
        printf("cuda Free for \"%s\" at %p on device %d\n" , d_varDB[i].label, d_varDB[i].var_ptr, d_device_id );
      }

      d_varDB[i].label[0] = 0; // leave a hole in the flat array, not deleted.
      d_dirty=true;
    }
    i++;
  }
#endif 
  return false;
}

//______________________________________________________________________
//
HOST_DEVICE void
GPUDataWarehouse::init_device(int id)
{
#ifdef __CUDA_ARCH__
  // no meaning in device method
#else
  hipError_t retVal;
  d_device_id = id;
  CUDA_RT_SAFE_CALL(retVal = hipSetDevice( d_device_id ));
  CUDA_RT_SAFE_CALL( retVal = hipMalloc((void**)&d_device_copy, sizeof(GPUDataWarehouse)));
  
  if(d_debug){
    printf("Init GPUDW on-device copy %lu bytes to %p on device %d\n", sizeof(GPUDataWarehouse), d_device_copy, d_device_id);
  }
  
  d_dirty=true;
#endif 
}

//______________________________________________________________________
//
HOST_DEVICE void
GPUDataWarehouse::syncto_device()
{
#ifdef __CUDA_ARCH__
  // no meaning in device method
#else
  if (!d_device_copy) {
    printf("ERROR:\nGPUDataWarehouse::syncto_device()\nNo device copy\n");
    exit(-1);
  }
  // TODO: only sync the difference
  if (d_dirty){
    hipError_t retVal;
    CUDA_RT_SAFE_CALL(retVal = hipSetDevice( d_device_id ));
    CUDA_RT_SAFE_CALL (retVal = hipMemcpy( d_device_copy,this, sizeof(GPUDataWarehouse), hipMemcpyHostToDevice));
    
    if (d_debug) {
      printf("sync GPUDW %p to device %d\n", d_device_copy, d_device_id);
    }
  }
  d_dirty=false;
#endif
}

//______________________________________________________________________
//
HOST_DEVICE void
GPUDataWarehouse::clear() 
{
#ifdef __CUDA_ARCH__
  // no meaning in device method
#else

  hipError_t retVal;
  CUDA_RT_SAFE_CALL(retVal = hipSetDevice( d_device_id ));
  for (int i=0; i<d_numItems; i++) {
    if (d_varDB[i].label[0] != 0){
      CUDA_RT_SAFE_CALL(retVal = hipFree(d_varDB[i].var_ptr));
      
      if (d_debug){
        printf("hipFree for \"%s\" at %p on device %d\n", d_varDB[i].label, d_varDB[i].var_ptr, d_device_id );
      }
    }
  }

  d_numItems=0;
  if ( d_device_copy ) {
    CUDA_RT_SAFE_CALL(retVal =  hipFree( d_device_copy ));
    if(d_debug){
      printf("Delete GPUDW on-device copy at %p on device %d \n",  d_device_copy, d_device_id);
    }
  }
#endif
}
//______________________________________________________________________
//


}
