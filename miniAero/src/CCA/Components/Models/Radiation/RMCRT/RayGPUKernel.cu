#include "hip/hip_runtime.h"
/*
 * The MIT License
 *
 * Copyright (c) 1997-2015 The University of Utah
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to
 * deal in the Software without restriction, including without limitation the
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 */

#include <CCA/Components/Models/Radiation/RMCRT/RayGPU.cuh>
#include <Core/Grid/Variables/GPUGridVariable.h>

#include <CCA/Components/Schedulers/GPUDataWarehouse.h>
#include <Core/Grid/Variables/Stencil7.h>
#include <Core/Grid/Variables/GPUStencil7.h>
#include <sci_defs/cuda_defs.h>
#include <sci_defs/uintah_defs.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// TURN ON debug flag in src/Core/Math/MersenneTwister.h to compare with Ray:CPU
#define DEBUG -9 // 1: divQ, 2: boundFlux, 3: scattering
//#define FIXED_RANDOM_NUM

//__________________________________
//  To Do
//  - dynamic block size?
//  - use labelNames
//  - investigate the performance with different patch configurations
//  - deterministic random numbers
//  - Ray steps
//  - What's up with data onion and raylocation call?


namespace Uintah {

using namespace SCIRun;

//---------------------------------------------------------------------------
// Kernel: The GPU ray tracer kernel
//---------------------------------------------------------------------------
template< class T>
__global__ void rayTraceKernel(dim3 dimGrid,
                               dim3 dimBlock,
                               int matl,
                               patchParams patch,
                               hiprandState* randNumStates,
                               RMCRT_flags RT_flags,
                               varLabelNames labelNames,
                               GPUDataWarehouse* abskg_gdw,
                               GPUDataWarehouse* sigmaT4_gdw,
                               GPUDataWarehouse* celltype_gdw,
                               GPUDataWarehouse* old_gdw,
                               GPUDataWarehouse* new_gdw)
{

  int blockID  = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z; 
  int threadID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
  
  // calculate the thread indices
  int tidX = threadIdx.x + blockIdx.x * blockDim.x + patch.loEC.x;
  int tidY = threadIdx.y + blockIdx.y * blockDim.y + patch.loEC.y;
  
  const GPUGridVariable< T > sigmaT4OverPi;
  const GPUGridVariable< T > abskg;              // Need to use getRegion() to get the data
  const GPUGridVariable<int> celltype;

  GPUGridVariable<double> divQ;
  GPUGridVariable<GPUStencil7> boundFlux;
  GPUGridVariable<double> radiationVolQ;
 
    
  sigmaT4_gdw->get( sigmaT4OverPi , "sigmaT4",  patch.ID, matl );       // Should be using labelNames struct
  celltype_gdw->get( celltype,     "cellType", patch.ID, matl );
  
  if(RT_flags.usingFloats){
    abskg_gdw->get( abskg , "abskgRMCRT",   patch.ID, matl );
  }else{
    abskg_gdw->get( abskg , "abskg",   patch.ID, matl );
  }

  if( RT_flags.modifies_divQ ){
    new_gdw->getModifiable( divQ,         "divQ",          patch.ID, matl );
    new_gdw->getModifiable( boundFlux,    "boundFlux",     patch.ID, matl );
    new_gdw->getModifiable( radiationVolQ,"radiationVolq", patch.ID, matl );
  }else{
    new_gdw->get( divQ,         "divQ",          patch.ID, matl );         // these should be allocateAntPut() calls
    new_gdw->get( boundFlux,    "boundFlux",     patch.ID, matl );
    new_gdw->get( radiationVolQ,"radiationVolq", patch.ID, matl );
    
    
    // Extra Cell Loop
    if (tidX >= patch.loEC.x && tidY >= patch.loEC.y && tidX < patch.hiEC.x && tidY < patch.hiEC.y) { // patch boundary check
      #pragma unroll
      for (int z = patch.loEC.z; z < patch.hiEC.z; z++) { // loop through z slices
        gpuIntVector c = make_int3(tidX, tidY, z);
        divQ[c]          = 0.0;
        radiationVolQ[c] = 0.0;
      }
    }
  }
  

  
/*`==========TESTING==========*/
#if 0  
 //__________________________________
 // Sanity check code used to test the "iterators" 
  // Extra Cell Loop
  if (threadIdx.y == 2 ) {
    printf( "outside loops thread[%d, %d] tID[%d, %d]\n",threadIdx.x, threadIdx.y, tidX, tidY);
  }
  
  if (tidX >= patch.loEC.x && tidY >= patch.loEC.y && tidX < patch.hiEC.x && tidY < patch.hiEC.y) { // patch boundary check
    for (int z = patch.loEC.z; z < patch.hiEC.z; z++) { // loop through z slices
      gpuIntVector c = make_int3(tidX, tidY, z);
      divQ[c] = 0;

      if (c.y == 2 && c.z == 2 ) {
        printf( " EC thread[%d, %d] tID[%d, %d]\n",threadIdx.x, threadIdx.y, tidX, tidY);
      }
    }
  }  

  if (tidX >= patch.lo.x && tidY >= patch.lo.y && tidX < patch.hi.x && tidY < patch.hi.y) { // patch boundary check
    for (int z = patch.lo.z; z < patch.hi.z; z++) { // loop through z slices
      gpuIntVector c = make_int3(tidX, tidY, z);
      if (c.y == 2 && c.z == 2 ) {
        printf( " int thread[%d, %d] tID[%d, %d]\n",threadIdx.x, threadIdx.y, tidX, tidY);
      }
      divQ[c] = c.x + c.y + c.z; 
    }
  }
  return;
#endif 
/*===========TESTING==========`*/

  double DyDx = patch.dx.y/patch.dx.x;
  double DzDx = patch.dx.z/patch.dx.x;
  
  //______________________________________________________________________
  //           R A D I O M E T E R
  //______________________________________________________________________
  // TO BE FILLED IN
  
  //______________________________________________________________________
  //          B O U N D A R Y F L U X
  //______________________________________________________________________
  if( RT_flags.solveBoundaryFlux ){

    __shared__ int3 dirIndexOrder[6];
    __shared__ int3 dirSignSwap[6];
    __shared__ int3 locationIndexOrder[6];
    __shared__ int3 locationShift[6];
    
    //_____________________________________________
    //   Ordering for Surface Method
    // This block of code is used to properly place ray origins, and orient ray directions
    // onto the correct face.  This is necessary, because by default, the rays are placed
    // and oriented onto a default face, then require adjustment onto the proper face.
    dirIndexOrder[EAST]   = make_int3(2, 1, 0);
    dirIndexOrder[WEST]   = make_int3(2, 1, 0);
    dirIndexOrder[NORTH]  = make_int3(0, 2, 1);
    dirIndexOrder[SOUTH]  = make_int3(0, 2, 1);
    dirIndexOrder[TOP]    = make_int3(0, 1, 2);
    dirIndexOrder[BOT]    = make_int3(0, 1, 2);

    // Ordering is slightly different from 6Flux since here, rays pass through origin cell from the inside faces.
    dirSignSwap[EAST]     = make_int3(-1, 1,  1);
    dirSignSwap[WEST]     = make_int3( 1, 1,  1);
    dirSignSwap[NORTH]    = make_int3( 1, -1, 1);
    dirSignSwap[SOUTH]    = make_int3( 1, 1,  1);
    dirSignSwap[TOP]      = make_int3( 1, 1, -1);
    dirSignSwap[BOT]      = make_int3( 1, 1,  1);

    locationIndexOrder[EAST]  = make_int3(1,0,2);
    locationIndexOrder[WEST]  = make_int3(1,0,2);
    locationIndexOrder[NORTH] = make_int3(0,1,2);
    locationIndexOrder[SOUTH] = make_int3(0,1,2);
    locationIndexOrder[TOP]   = make_int3(0,2,1);
    locationIndexOrder[BOT]   = make_int3(0,2,1);

    locationShift[EAST]   = make_int3(1, 0, 0);
    locationShift[WEST]   = make_int3(0, 0, 0);
    locationShift[NORTH]  = make_int3(0, 1, 0);
    locationShift[SOUTH]  = make_int3(0, 0, 0);
    locationShift[TOP]    = make_int3(0, 0, 1);
    locationShift[BOT]    = make_int3(0, 0, 0);  
    
    //__________________________________
    // GPU equivalent of GridIterator loop - calculate sets of rays per thread
    if (tidX >= patch.lo.x && tidY >= patch.lo.y && tidX < patch.hi.x && tidY < patch.hi.y) { // patch boundary check
      #pragma unroll
      for (int z = patch.lo.z; z < patch.hi.z; z++) { // loop through z slices
      
        gpuIntVector origin = make_int3(tidX, tidY, z);  // for each thread
        
        boundFlux[origin].initialize(0.0);

        BoundaryFaces boundaryFaces;

 /*
        if(RT_flags.benchMark==4 || RT_flags.benchMark==5){
          boundaryFaces.addFace(5);
        }
*/
        
        // which surrounding cells are boundaries
        boundFlux[origin].p = has_a_boundaryDevice(origin, celltype, boundaryFaces);

        //__________________________________
        // Loop over boundary faces of the cell and compute incident radiative flux
        #pragma unroll
        for( int i = 0; i<boundaryFaces.size(); i++) {
          
          int RayFace = boundaryFaces.faceArray[i];
          int UintahFace[6] = {WEST,EAST,SOUTH,NORTH,BOT,TOP};
          
          double sumI     = 0;
          double sumProjI = 0;
          double sumI_prev= 0;

          //__________________________________
          // Flux ray loop
          #pragma unroll
          for (int iRay=0; iRay < RT_flags.nFluxRays; iRay++){

            gpuVector direction_vector, ray_location; 
            double cosTheta;

            rayDirection_cellFaceDevice( randNumStates, origin, dirIndexOrder[RayFace], dirSignSwap[RayFace], iRay,
                                   direction_vector, cosTheta );
                                   
            rayLocation_cellFaceDevice( randNumStates, origin, locationIndexOrder[RayFace], locationShift[RayFace], 
                                  DyDx, DzDx, ray_location);            

            updateSumIDevice< T >( direction_vector, ray_location, origin, patch.dx, sigmaT4OverPi, abskg, celltype, sumI, randNumStates, RT_flags);

            sumProjI += cosTheta * (sumI - sumI_prev);   // must subtract sumI_prev, since sumI accumulates intensity

            sumI_prev = sumI;

          } // end of flux ray loop

          //__________________________________
          //  Compute Net Flux to the boundary
          int face = UintahFace[RayFace];            
          boundFlux[origin][ face ] = sumProjI * 2 *M_PI/RT_flags.nFluxRays;

/*`==========TESTING==========*/
#if DEBUG == 2
          printf( "\n      [%d, %d, %d]  face: %d sumProjI:  %g BF: %g\n", 
                    origin.x, origin.y, origin.z, face, sumProjI, boundFlux[origin][ face ]);
#endif
/*===========TESTING==========`*/

        } // boundary faces loop
      }  // z slices loop
    }  // X-Y Thread loop
  }
  
  
  //______________________________________________________________________
  //         S O L V E   D I V Q
  //______________________________________________________________________
  if( RT_flags.solveDivQ ){
    // GPU equivalent of GridIterator loop - calculate sets of rays per thread
    if (tidX >= patch.lo.x && tidY >= patch.lo.y && tidX < patch.hi.x && tidY < patch.hi.y) { // patch boundary check
      #pragma unroll
      for (int z = patch.lo.z; z < patch.hi.z; z++) { // loop through z slices

        gpuIntVector origin = make_int3(tidX, tidY, z);  // for each thread
        double sumI = 0;
        
        //__________________________________
        // ray loop
        #pragma unroll
        for (int iRay = 0; iRay < RT_flags.nDivQRays; iRay++) {
        
          gpuVector direction_vector = findRayDirectionDevice( randNumStates );
          
          gpuVector ray_location = rayLocationDevice( randNumStates, origin, DyDx,  DzDx, RT_flags.CCRays );
         
          updateSumIDevice< T >( direction_vector, ray_location, origin, patch.dx,  sigmaT4OverPi, abskg, celltype, sumI, randNumStates, RT_flags);
        } //Ray loop
 
        //__________________________________
        //  Compute divQ
        divQ[origin] = 4.0 * M_PI * abskg[origin] * ( sigmaT4OverPi[origin] - (sumI/RT_flags.nDivQRays) );

        // radiationVolq is the incident energy per cell (W/m^3) and is necessary when particle heat transfer models (i.e. Shaddix) are used 
        radiationVolQ[origin] = 4.0 * M_PI * abskg[origin] *  (sumI/RT_flags.nDivQRays) ;
        
/*`==========TESTING==========*/
#if DEBUG == 1
          printf( "\n      [%d, %d, %d]  sumI: %g  divQ: %g radiationVolq: %g  abskg: %g,    sigmaT4: %g \n", 
                    origin.x, origin.y, origin.z, sumI,divQ[origin], radiationVolQ[origin],abskg[origin], sigmaT4OverPi[origin]);
#endif
/*===========TESTING==========`*/
      }  // end z-slice loop
    }  // end domain boundary check
  }  // solve divQ
//}  // thread1
}  // end ray trace kernel

//______________________________________________________________________
//
//______________________________________________________________________
__device__ gpuVector findRayDirectionDevice( hiprandState* randNumStates )
{
  // Random Points On Sphere
  // add fuzz to prevent infs in 1/dirVector calculation
  double plusMinus_one = 2.0 * randDblExcDevice( randNumStates ) - 1.0 + DBL_EPSILON;
  double r = sqrt(1.0 - plusMinus_one * plusMinus_one);             // Radius of circle at z
  double theta = 2.0 * M_PI * randDblExcDevice( randNumStates );    // Uniform betwen 0-2Pi

  gpuVector dirVector;
  dirVector.x = r*cos(theta);   // Convert to cartesian coordinates
  dirVector.y = r*sin(theta);
  dirVector.z = plusMinus_one;

  return dirVector;
}

//______________________________________________________________________
// Compute the Ray direction from a cell face
__device__ void rayDirection_cellFaceDevice( hiprandState* randNumStates,
                                             const gpuIntVector& origin,
                                             const gpuIntVector& indexOrder, 
                                             const gpuIntVector& signOrder,
                                             const int iRay,
                                             gpuVector& directionVector,
                                             double& cosTheta)
{

  // Surface Way to generate a ray direction from the positive z face
  double phi   = 2 * M_PI * randDevice( randNumStates ); // azimuthal angle.  Range of 0 to 2pi
  double theta = acos( randDevice( randNumStates ) );      // polar angle for the hemisphere
  cosTheta = cos( theta );
  double sinTheta = sin( theta );

  //Convert to Cartesian
  gpuVector tmp;
  tmp[0] =  sinTheta * cos( phi );
  tmp[1] =  sinTheta * sin( phi );
  tmp[2] =  cosTheta;

  // Put direction vector as coming from correct face,
  directionVector[0] = tmp[indexOrder[0]] * signOrder[0];
  directionVector[1] = tmp[indexOrder[1]] * signOrder[1];
  directionVector[2] = tmp[indexOrder[2]] * signOrder[2];
}


//______________________________________________________________________
//
__device__ gpuVector rayLocationDevice( hiprandState* randNumStates,
                                      const gpuIntVector origin,
                                      const double DyDx, 
                                      const double DzDx,
                                      const bool useCCRays)
{
  gpuVector location;
  if( useCCRays == false ){
    location.x =   (double) origin.x +  randDevice( randNumStates ) ;
    location.y =   (double) origin.y +  randDevice( randNumStates ) * DyDx ;
    location.z =   (double) origin.z +  randDevice( randNumStates)  * DzDx ;
  }else{
    location.x =   origin.x +  0.5 ;
    location.y =   origin.y +  0.5 * DyDx ;
    location.z =   origin.z +  0.5 * DzDx ;
  }
  return location;
}

//______________________________________________________________________
//  Compute the Ray location from a cell face
__device__ void rayLocation_cellFaceDevice( hiprandState* randNumStates,
                                            const gpuIntVector& origin,
                                            const gpuIntVector &indexOrder, 
                                            const gpuIntVector &shift, 
                                            const double &DyDx, 
                                            const double &DzDx,
                                            gpuVector& location)
{
  gpuVector tmp;
  tmp[0] =  randDevice( randNumStates ) ;
  tmp[1] =  0;
  tmp[2] =  randDevice( randNumStates ) * DzDx ;
  
  // Put point on correct face
  location[0] = tmp[indexOrder[0]] + (double)shift[0];
  location[1] = tmp[indexOrder[1]] + (double)shift[1] * DyDx;
  location[2] = tmp[indexOrder[2]] + (double)shift[2] * DzDx;

  location[0] += (double) origin.x;
  location[1] += (double) origin.y;
  location[2] += (double) origin.z;
}

//______________________________________________________________________
//
__device__ bool has_a_boundaryDevice(const gpuIntVector &c, 
                                     const GPUGridVariable<int>& celltype, 
                                     BoundaryFaces &boundaryFaces){

  gpuIntVector adj = c;
  bool hasBoundary = false;

  adj[0] = c[0] - 1;     // west

  if ( celltype[adj]+1 ){              // cell type of flow is -1, so when cellType+1 isn't false, we
    boundaryFaces.addFace( WEST );     // know we're at a boundary
    hasBoundary = true;
  }

  adj[0] += 2;           // east

  if ( celltype[adj]+1 ){
    boundaryFaces.addFace( EAST );
    hasBoundary = true;
  }

  adj[0] -= 1;
  adj[1] = c[1] - 1;     // south

  if ( celltype[adj]+1 ){
    boundaryFaces.addFace( SOUTH );
    hasBoundary = true;
  }

  adj[1] += 2;           // north

  if ( celltype[adj]+1 ){
    boundaryFaces.addFace( NORTH );
    hasBoundary = true;
  }

  adj[1] -= 1;
  adj[2] = c[2] - 1;     // bottom

  if ( celltype[adj]+1 ){
    boundaryFaces.addFace( BOT );
    hasBoundary = true;
  }

  adj[2] += 2;           // top

  if ( celltype[adj]+1 ){
    boundaryFaces.addFace( TOP );
    hasBoundary = true;
  }

  return (hasBoundary);
}


//______________________________________________________________________
//
__device__ void findStepSizeDevice(int step[],
                                   bool sign[],
                                   const gpuVector& inv_direction_vector){
  // get new step and sign
  for ( int d= 0; d<3; d++ ){
  
    if (inv_direction_vector[d]>0){
      step[d] = 1;
      sign[d] = 1;
    }else{
      step[d] = -1;
      sign[d] = 0;
    }
  }
}


//______________________________________________________________________
//
__device__ void reflect(double& fs,
                         gpuIntVector& cur,
                         gpuIntVector& prevCell,
                         const double abskg,
                         bool& in_domain,
                         int& step,
                         bool& sign,
                         double& ray_direction)
{
  fs = fs * (1 - abskg);

  //put cur back inside the domain
  cur = prevCell;
  in_domain = true;

  // apply reflection condition
  step *= -1;                // begin stepping in opposite direction
  sign = (sign==1) ? 0 : 1;  //  swap sign from 1 to 0 or vice versa
  ray_direction *= -1;
}

//______________________________________________________________________
template< class T >
__device__ void updateSumIDevice ( gpuVector& ray_direction,
                                   gpuVector& ray_location,
                                   const gpuIntVector& origin,
                                   const gpuVector& Dx,
                                   const GPUGridVariable< T >& sigmaT4OverPi,
                                   const GPUGridVariable< T >& abskg,
                                   const GPUGridVariable<int>& celltype,
                                   double& sumI,
                                   hiprandState* randNumStates,
                                   RMCRT_flags RT_flags)

{

 
  gpuIntVector cur = origin;
  gpuIntVector prevCell = cur;
  // Step and sign for ray marching
  int step[3];                                          // Gives +1 or -1 based on sign    
  bool sign[3];                                                                            
                                                                                           
  gpuVector inv_ray_direction = 1.0/ray_direction;
/*`==========TESTING==========*/
#if DEBUG == 1
  printf("        updateSumI: [%d,%d,%d] ray_dir [%g,%g,%g] ray_loc [%g,%g,%g]\n", origin.x, origin.y, origin.z,ray_direction.x, ray_direction.y, ray_direction.z, ray_location.x, ray_location.y, ray_location.z);
  printf("        inv_ray_dir [%g,%g,%g]\n", inv_ray_direction.x,inv_ray_direction.y,inv_ray_direction.z);
#endif
/*===========TESTING==========`*/  

  findStepSizeDevice(step, sign, inv_ray_direction);
  gpuVector D_DxRatio = make_double3(1, Dx.y/Dx.x, Dx.z/Dx.x );

  gpuVector tMax;         // (mixing bools, ints and doubles)
  tMax.x = (origin.x + sign[0]               - ray_location.x) * inv_ray_direction.x ; 
  tMax.y = (origin.y + sign[1] * D_DxRatio.y - ray_location.y) * inv_ray_direction.y ; 
  tMax.z = (origin.z + sign[2] * D_DxRatio.z - ray_location.z) * inv_ray_direction.z ; 

  //Length of t to traverse one cell
  gpuVector tDelta;
  tDelta   = Abs(inv_ray_direction) * D_DxRatio;                              

  //Initializes the following values for each ray
  bool in_domain     = true;
  double tMax_prev   = 0;
  double intensity   = 1.0;
  double fs          = 1.0;
  int nReflect       = 0;                 // Number of reflections                         
  double optical_thickness      = 0;                                                       
  double expOpticalThick_prev   = 1.0;                                                     


#ifdef RAY_SCATTER
  double scatCoeff = RT_flags.sigmaScat;          //[m^-1]  !! HACK !! This needs to come from data warehouse
  if (scatCoeff == 0) scatCoeff = 1e-99;  // avoid division by zero

  // Determine the length at which scattering will occur
  // See CCA/Components/Arches/RMCRT/PaulasAttic/MCRT/ArchesRMCRT/ray.cc
  double scatLength = -log( randDblExcDevice( randNumStates ) ) / scatCoeff;
  double curLength = 0;
#endif

  //+++++++Begin ray tracing+++++++++++++++++++
  //Threshold while loop
  while ( intensity > RT_flags.threshold ){

    DIR face = NONE;
    
    while (in_domain){

      prevCell = cur;
      double disMin = -9;          // Represents ray segment length.

      //__________________________________
      //  Determine which cell the ray will enter next
      if ( tMax.x < tMax.y ){        // X < Y
        if ( tMax.x < tMax.z ){      // X < Z
          face = X;
        } else {
          face = Z;
        }
      } else {
        if( tMax.y < tMax.z ){       // Y < Z
          face = Y;
        } else {
          face = Z;
        }
      }
      
      //__________________________________
      //  update marching variables
      cur[face]  = cur[face] + step[face];
      disMin     = (tMax[face] - tMax_prev);
      tMax_prev  = tMax[face];
      tMax[face] = tMax[face] + tDelta[face];

      ray_location.x = ray_location.x + (disMin  * ray_direction.x);
      ray_location.y = ray_location.y + (disMin  * ray_direction.y);
      ray_location.z = ray_location.z + (disMin  * ray_direction.z);
      
/*`==========TESTING==========*/
#if DEBUG == 1
if(origin.x == 0 && origin.y == 0 && origin.z ==0){
    printf( "            cur [%d,%d,%d] prev [%d,%d,%d] ", cur.x, cur.y, cur.z, prevCell.x, prevCell.y, prevCell.z);
    printf( " face %d ", face ); 
    printf( "tMax [%g,%g,%g] ",tMax.x,tMax.y, tMax.z);
    printf( "rayLoc [%g,%g,%g] ",ray_location.x,ray_location.y, ray_location.z);
    printf( "inv_dir [%g,%g,%g] ",inv_ray_direction.x,inv_ray_direction.y, inv_ray_direction.z); 
    printf( "disMin %g \n",disMin ); 
   
    printf( "            abskg[prev] %g  \t sigmaT4OverPi[prev]: %g \n",abskg[prevCell],  sigmaT4OverPi[prevCell]);
    printf( "            abskg[cur]  %g  \t sigmaT4OverPi[cur]:  %g  \t  cellType: %i\n",abskg[cur], sigmaT4OverPi[cur], celltype[cur] );
} 
#endif

/*===========TESTING==========`*/
      in_domain = (celltype[cur]==-1);  //cellType of -1 is flow
    
      optical_thickness += Dx.x * abskg[prevCell]*disMin; // as long as tDeltaY,Z tMax.y(),Z and ray_location[1],[2]..
      // were adjusted by DyDx  or DzDx, this line is now correct for noncubic domains.

      RT_flags.nRaySteps ++;

      //Eqn 3-15(see below reference) while
      //Third term inside the parentheses is accounted for in Inet. Chi is accounted for in Inet calc.
      double expOpticalThick = exp(-optical_thickness);

      sumI += sigmaT4OverPi[prevCell] * ( expOpticalThick_prev - expOpticalThick ) * fs;

      expOpticalThick_prev = expOpticalThick;


#ifdef RAY_SCATTER
      curLength += disMin * Dx.x;
      if (curLength > scatLength && in_domain){

        // get new scatLength for each scattering event
        scatLength = -log( randDblExcDevice( randNumStates ) ) / scatCoeff; 

        ray_direction     = findRayDirectionDevice( randNumStates ); 
        
        inv_ray_direction = 1.0/ray_direction;

        // get new step and sign
        int stepOld = step[face];
        findStepSizeDevice( step, sign, inv_ray_direction);

        // if sign[face] changes sign, put ray back into prevCell (back scattering)
        // a sign change only occurs when the product of old and new is negative
        if( step[face] * stepOld < 0 ){
          cur = prevCell;
        }

        // get new tMax (mixing bools, ints and doubles)
        tMax.x = ( ( cur.x + sign[0]               - ray_location.x) * inv_ray_direction.x );
        tMax.y = ( ( cur.y + sign[1] * D_DxRatio.y - ray_location.y) * inv_ray_direction.y );
        tMax.z = ( ( cur.z + sign[2] * D_DxRatio.z - ray_location.z) * inv_ray_direction.z );

        // Length of t to traverse one cell
        tDelta    = Abs(inv_ray_direction) * D_DxRatio;
        tMax_prev = 0;
        curLength = 0;  // allow for multiple scattering events per ray

/*`==========TESTING==========*/
#if DEBUG == 3
        printf( "%i, %i, %i, tmax: %g, %g, %g  tDelta: %g, %g, %g \n", cur.x, cur.y, cur.z, tMax.x, tMax.y, tMax.z, tDelta.x, tDelta.y , tDelta.z );
#endif
/*===========TESTING==========`*/

        //if(_benchmark == 4 || _benchmark ==5) scatLength = 1e16; // only for Siegel Benchmark4 benchmark5. Only allows 1 scatter event.
      }
#endif

    } //end domain while loop.  ++++++++++++++
    
    //  wall emission 12/15/11
    double wallEmissivity = abskg[cur];

    if (wallEmissivity > 1.0){       // Ensure wall emissivity doesn't exceed one. 
      wallEmissivity = 1.0;
    } 

    intensity = exp(-optical_thickness);

    sumI += wallEmissivity * sigmaT4OverPi[cur] * intensity;

    intensity = intensity * fs;
    

    // when a ray reaches the end of the domain, we force it to terminate. 
    if( !RT_flags.allowReflect ) intensity = 0;


/*`==========TESTING==========*/
#if DEBUG == 1
if(origin.x == 0 && origin.y == 0 && origin.z ==0 ){
    printf( "            cur [%d,%d,%d] intensity: %g expOptThick: %g, fs: %g allowReflect: %i \n", 
            cur.x, cur.y, cur.z, intensity,  exp(-optical_thickness), fs,RT_flags.allowReflect );
    
} 
__syncthreads();
#endif 
/*===========TESTING==========`*/
    //__________________________________
    //  Reflections
    if ( (intensity > RT_flags.threshold) && RT_flags.allowReflect){
      reflect( fs, cur, prevCell, abskg[cur], in_domain, step[face], sign[face], ray_direction[face]);
      ++nReflect;
    }

  }  // threshold while loop.
} // end of updateSumI function


//---------------------------------------------------------------------------
// Returns random number between 0 & 1.0 including 0 & 1.0
// See src/Core/Math/MersenneTwister.h for equation 
//---------------------------------------------------------------------------
__device__ double randDevice(hiprandState* globalState)
{
  int tid = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
  hiprandState localState = globalState[tid];
  double val = hiprand(&localState);
  globalState[tid] = localState;

#ifdef FIXED_RANDOM_NUM
  return 0.3;
#else
  return (double)val * (1.0/4294967295.0);
#endif  
  
}

//---------------------------------------------------------------------------
// Returns random number between 0 & 1.0 excluding 0 & 1.0 
// See src/Core/Math/MersenneTwister.h for equation 
//---------------------------------------------------------------------------
__device__ double randDblExcDevice(hiprandState* globalState)
{
  int tid = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
  
  hiprandState localState = globalState[tid];
  double val = hiprand(&localState);
  globalState[tid] = localState;

#ifdef FIXED_RANDOM_NUM
  return 0.3;
#else
  return ( (double)val + 0.5 ) * (1.0/4294967296.0);
#endif
}

//______________________________________________________________________
//  Each thread gets same seed, a different sequence number, no offset 
//  This will create repeatable results.
__global__ void setupRandNum_kernel(hiprandState* randNumStates)
{
  int tID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
  hiprand_init(1234, tID, 0, &randNumStates[tID]);
}

//______________________________________________________________________
template< class T>
__host__ void launchRayTraceKernel(dim3 dimGrid,
                                   dim3 dimBlock,
                                   int matlIndex,
                                   patchParams patch,
                                   hipStream_t* stream,
                                   RMCRT_flags RT_flags,
                                   varLabelNames labelNames,
                                   GPUDataWarehouse* abskg_gdw,
                                   GPUDataWarehouse* sigmaT4_gdw,
                                   GPUDataWarehouse* celltype_gdw,
                                   GPUDataWarehouse* old_gdw,
                                   GPUDataWarehouse* new_gdw)
{
  // setup random number generator states on the device, 1 for each thread
  hiprandState* randNumStates;
  int numStates = dimGrid.x * dimGrid.y * dimBlock.x * dimBlock.y * dimBlock.z;
  CUDA_RT_SAFE_CALL( hipMalloc((void**)&randNumStates, numStates * sizeof(hiprandState)) );
  
  
  setupRandNum_kernel<<< dimGrid, dimBlock>>>( randNumStates );
  
  rayTraceKernel< T ><<< dimGrid, dimBlock, 0, *stream >>>(dimGrid, 
                                                      dimBlock, 
                                                      matlIndex,
                                                      patch,
                                                      randNumStates,
                                                      RT_flags,
                                                      labelNames,
                                                      abskg_gdw,
                                                      sigmaT4_gdw,
                                                      celltype_gdw,
                                                      old_gdw,
                                                      new_gdw);
    // free device-side RNG states
    CUDA_RT_SAFE_CALL( hipFree(randNumStates) );
}

//______________________________________________________________________
//  Explicit template instantiations

template
__host__ void launchRayTraceKernel<double>(dim3 dimGrid,
                                           dim3 dimBlock,
                                           int matlIndex,
                                           patchParams patch,
                                           hipStream_t* stream,
                                           RMCRT_flags RT_flags,
                                           varLabelNames labelNames,
                                           GPUDataWarehouse* abskg_gdw,
                                           GPUDataWarehouse* sigmaT4_gdw,
                                           GPUDataWarehouse* celltype_gdw,
                                           GPUDataWarehouse* old_gdw,
                                           GPUDataWarehouse* new_gdw);
                                   
template
__host__ void launchRayTraceKernel<float>(dim3 dimGrid,
                                          dim3 dimBlock,
                                          int matlIndex,
                                          patchParams patch,
                                          hipStream_t* stream,
                                          RMCRT_flags RT_flags,
                                          varLabelNames labelNames,
                                          GPUDataWarehouse* abskg_gdw,
                                          GPUDataWarehouse* sigmaT4_gdw,
                                          GPUDataWarehouse* celltype_gdw,
                                          GPUDataWarehouse* old_gdw,
                                          GPUDataWarehouse* new_gdw);

} //end namespace Uintah
