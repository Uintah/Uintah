#include "hip/hip_runtime.h"
/*

The MIT License

Copyright (c) 1997-2011 Center for the Simulation of Accidental Fires and
Explosions (CSAFE), and  Scientific Computing and Imaging Institute (SCI), 
University of Utah.

License for the specific language governing rights and limitations under
Permission is hereby granted, free of charge, to any person obtaining a 
copy of this software and associated documentation files (the "Software"),
to deal in the Software without restriction, including without limitation 
the rights to use, copy, modify, merge, publish, distribute, sublicense, 
and/or sell copies of the Software, and to permit persons to whom the 
Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included 
in all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS 
OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, 
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL 
THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER 
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING 
FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
DEALINGS IN THE SOFTWARE.

 */



#include <CCA/Components/Examples/AdvectSlabsGPU.h>
#include <CCA/Components/Examples/ExamplesLabel.h>
#include <CCA/Ports/LoadBalancer.h>
#include <Core/ProblemSpec/ProblemSpec.h>
#include <Core/Grid/Variables/CCVariable.h>
#include <Core/Grid/Variables/CellIterator.h>
#include <Core/Grid/SimulationState.h>
#include <Core/Grid/Task.h>
#include <Core/Grid/Grid.h>
#include <Core/Grid/Level.h>
#include <Core/Grid/SimpleMaterial.h>
#include <Core/Grid/Variables/VarTypes.h>
#include <Core/Parallel/ProcessorGroup.h>
#include <CCA/Ports/Scheduler.h>
#include <Core/Malloc/Allocator.h>
#include <Core/Parallel/Parallel.h>
#include <Core/Malloc/Allocator.h>
#include <Core/Util/Endian.h>
#include <Core/Util/FancyAssert.h>

using namespace Uintah;

AdvectSlabsGPU::AdvectSlabsGPU(const ProcessorGroup* myworld)
: UintahParallelComponent(myworld)
{
  mass_label = VarLabel::create("mass", 
                                CCVariable<double>::getTypeDescription());
  massAdvected_label = VarLabel::create("massAdvected", 
                                        CCVariable<double>::getTypeDescription());

  //__________________________________
  //  outflux/influx slabs
  OF_slab[RIGHT] = RIGHT;         IF_slab[RIGHT]  = LEFT;
  OF_slab[LEFT]  = LEFT;          IF_slab[LEFT]   = RIGHT;
  OF_slab[TOP]   = TOP;           IF_slab[TOP]    = BOTTOM;
  OF_slab[BOTTOM]= BOTTOM;        IF_slab[BOTTOM] = TOP;
  OF_slab[FRONT] = FRONT;         IF_slab[FRONT]  = BACK;
  OF_slab[BACK]  = BACK;          IF_slab[BACK]   = FRONT;

  // Slab adjacent cell
  S_ac[RIGHT]  =  IntVector( 1, 0, 0);
  S_ac[LEFT]   =  IntVector(-1, 0, 0);
  S_ac[TOP]    =  IntVector( 0, 1, 0);
  S_ac[BOTTOM] =  IntVector( 0,-1, 0);
  S_ac[FRONT]  =  IntVector( 0, 0, 1);
  S_ac[BACK]   =  IntVector( 0, 0,-1);
}

AdvectSlabsGPU::~AdvectSlabsGPU()
{

}

void AdvectSlabsGPU::problemSetup(const ProblemSpecP& params,
                                  const ProblemSpecP& restart_prob_spec,
                                  GridP&, SimulationStateP& sharedState)
{
  sharedState_ = sharedState;
  ProblemSpecP ps = params->findBlock("AdvectSlabsGPU");
  ps->require("delt", delt_);
  mymat_ = scinew SimpleMaterial();
  sharedState->registerSimpleMaterial(mymat_);
}

void AdvectSlabsGPU::scheduleInitialize(const LevelP& level,
                                        SchedulerP& sched)
{
  Task* task = scinew Task("initialize",
                           this, &AdvectSlabsGPU::initialize);
  task->computes(mass_label);
  task->computes(massAdvected_label);
  sched->addTask(task, level->eachPatch(), sharedState_->allMaterials());
}

void AdvectSlabsGPU::scheduleComputeStableTimestep(const LevelP& level,
                                                   SchedulerP& sched)
{
  Task* task = scinew Task("computeStableTimestep",
                           this, &AdvectSlabsGPU::computeStableTimestep);
  task->computes(sharedState_->get_delt_label(),level.get_rep());
  sched->addTask(task, level->eachPatch(), sharedState_->allMaterials());
}

void
AdvectSlabsGPU::scheduleTimeAdvance( const LevelP& level, SchedulerP& sched)
{
  Task* task = scinew Task("timeAdvance",
                           this, &AdvectSlabsGPU::timeAdvance);

  task->requires(Task::OldDW, mass_label, Ghost::AroundCells, 1);
  task->computes(mass_label);
  task->computes(massAdvected_label);
  sched->addTask(task, level->eachPatch(), sharedState_->allMaterials());

}

void AdvectSlabsGPU::computeStableTimestep(const ProcessorGroup*,
                                           const PatchSubset* patches,
                                           const MaterialSubset*,
                                           DataWarehouse*, DataWarehouse* new_dw)
{
  new_dw->put(delt_vartype(delt_), sharedState_->get_delt_label(),getLevel(patches));
}

void AdvectSlabsGPU::initialize(const ProcessorGroup*,
                                const PatchSubset* patches,
                                const MaterialSubset* matls,
                                DataWarehouse*old_dw, DataWarehouse* new_dw)
{
  for(int p=0;p<patches->size();p++){
    const Patch* patch = patches->get(p);

    new_dw->allocateTemporary(d_OFS, patch, Ghost::AroundCells,1);
    for(int m = 0;m<matls->size();m++){
      int matl = matls->get(m);

      CCVariable<double> mass, massAd;
      new_dw->allocateAndPut(mass,   mass_label,         matl, patch, Ghost::AroundCells, 1);
      new_dw->allocateAndPut(massAd, massAdvected_label, matl, patch, Ghost::AroundCells, 1);
      mass.initialize(0.0);
      massAd.initialize(0.0);

      for(CellIterator iter = patch->getCellIterator(); !iter.done(); iter++)
      {
        // set initial value for fluxes
        for(int face = TOP; face <= BACK; face++ )  {
          d_OFS[*iter].d_fflux[face]= 1;
        }
        // set up the initial mass
        mass[*iter]=1;
      }
    }
  }
}

/// KERNEL FOR TIME ADVANCE
// @brief A kernel that applies the stencil used in timeAdvance(...)
// @param domainSize a three component vector that gives the size of the domain as (x,y,z)
// @param domainLower a three component vector that gives the lower corner of the work area as (x,y,z)
// @param ghostLayers the number of layers of ghost cells
// @param mass pointer to the source mass allocated on the device
// @param massAd pointer to the  massAdvected allocated on the device
// @param OFS pointer to an array of 6 element double arrays
// @param invol inverse of the volume of a single cell
__global__ void timeAdvanceKernelAdvectSlabs(uint3 domainSize,
                                             uint3 domainLower,
                                             int ghostLayers,
                                             double *mass,
                                             double *massAd,
                                             double *newMass,
                                             double *ofs,
                                             double invol) {

  // calculate the thread indices
  int tidX = blockDim.x * blockIdx.x + threadIdx.x;
  int tidY = blockDim.y * blockIdx.y + threadIdx.y;

 
  // Indexing
  int num_slices = domainSize.z - ghostLayers;
  int dx = domainSize.x;
  int dy = domainSize.y;
  int dxOFS = dx+1;
  int dyOFS = dy+1;

  double q_face_flux[6];
  double faceVol[6];

  if (tidX < (dx-1) && tidY < (dy-1) && tidX > 0 && tidY > 0) {
    int tidYminus = tidY - 1;
    int tidYplus  = tidY + 1;
    int tidXminus = tidX - 1;
    int tidXplus  = tidX + 1;
    for (int slice = ghostLayers; slice < num_slices; slice++) {
      // Variables needed for each cell
      double sum_q_face_flux = 0.0;
      int cell2 = INDEX3D(dx,dy, tidX,tidY, slice);
      int cell  = 6*cell2; //INDEX3D(dxOFS,dyOFS, tidX,tidY, slice);
      double influxVol  = 0.0;
      double outfluxVol = 0.0;
      double massCell   = mass[cell2];
      int adjCell;

      // Schematic of layout
      // enum FACE {TOP(0), BOTTOM(1), RIGHT(2), LEFT(3), FRONT(4), BACK(5)};
      //__________________________________
      //  outflux/influx slabs
      //  OF_slab[RIGHT] = RIGHT;         IF_slab[RIGHT]  = LEFT;
      //  OF_slab[LEFT]  = LEFT;          IF_slab[LEFT]   = RIGHT;
      //  OF_slab[TOP]   = TOP;           IF_slab[TOP]    = BOTTOM;
      //  OF_slab[BOTTOM]= BOTTOM;        IF_slab[BOTTOM] = TOP;
      //  OF_slab[FRONT] = FRONT;         IF_slab[FRONT]  = BACK;
      //  OF_slab[BACK]  = BACK;          IF_slab[BACK]   = FRONT;

      // Corresponding CPU code
      // double outfluxVol = d_OFS[c ].d_fflux[OF_slab[f]];
      // double influxVol  = d_OFS[ac].d_fflux[IF_slab[f]];


      // Unrolled 'for' loop
      // Above
      adjCell    = INDEX3D(dxOFS,dyOFS, tidX, tidYplus, slice);
      outfluxVol = ofs[cell];
      influxVol  = ofs[(6*adjCell+1)];

      q_face_flux[0]   = mass[adjCell]*influxVol - massCell*outfluxVol;
      faceVol[0]       = outfluxVol + influxVol;
      sum_q_face_flux += q_face_flux[0];

      // Below
      adjCell    = INDEX3D(dxOFS,dyOFS, tidX, tidYminus, slice);
      outfluxVol = ofs[(cell+1)];
      influxVol  = ofs[(6*adjCell)];

      q_face_flux[1]   = mass[adjCell]*influxVol - massCell*outfluxVol;
      faceVol[1]       = outfluxVol + influxVol;
      sum_q_face_flux += q_face_flux[1];

      // Right
      adjCell    = INDEX3D(dxOFS,dyOFS, tidXplus, tidY, slice);
      outfluxVol = ofs[(cell+2)];
      influxVol  = ofs[(6*adjCell+3)];

      q_face_flux[2]   = mass[adjCell]*influxVol - massCell*outfluxVol;
      faceVol[2]       = outfluxVol + influxVol;
      sum_q_face_flux += q_face_flux[2];

      // Left
      adjCell    = INDEX3D(dxOFS,dyOFS, tidXminus, tidY, slice);
      outfluxVol = ofs[(cell+3)];
      influxVol  = ofs[(6*adjCell+2)];

      q_face_flux[3]   = mass[adjCell]*influxVol - massCell*outfluxVol;
      faceVol[3]       = outfluxVol + influxVol;
      sum_q_face_flux += q_face_flux[3];

      // Front
      adjCell    = INDEX3D(dxOFS,dyOFS, tidX, tidY, (slice-1));
      outfluxVol = ofs[(cell+4)];
      influxVol  = ofs[(6*adjCell+5)];

      q_face_flux[4]   = mass[adjCell]*influxVol - massCell*outfluxVol;
      faceVol[4]       = outfluxVol + influxVol;
      sum_q_face_flux += q_face_flux[4];

      // Back
      adjCell    = INDEX3D(dxOFS,dyOFS, tidX, tidY, (slice+1));
      outfluxVol = ofs[(cell+5)];
      influxVol  = ofs[(6*adjCell+4)];

      q_face_flux[5]   = mass[adjCell]*influxVol - massCell*outfluxVol;
      faceVol[5]       = outfluxVol + influxVol;
      sum_q_face_flux += q_face_flux[5];

      // Sum all the Advected masses and adjust the new mass
      massAd[cell2]  = sum_q_face_flux*invol;
      newMass[cell2] = massCell - massAd[cell2];
    }
  }
}

void AdvectSlabsGPU::timeAdvance(const ProcessorGroup* pg,
                                 const PatchSubset* patches,
                                 const MaterialSubset* matls,
                                 DataWarehouse* old_dw, DataWarehouse* new_dw)
{
  int previousPatchSize = 0;// this is to see if we need to release and reallocate between computations
  int size = 0;
  int ghostLayers = 1;

  // declare device and host memory
  double* mass_device;
  double* newMass_device;
  double* massAd_device;
  double* ofs_device;
  double* massAd_host;
  double* mass_host;
  double* newMass_host;
  double* ofs_host;

  // find the "best" device for hipSetDevice()
  int num_devices, device;
  hipGetDeviceCount(&num_devices);
  if (num_devices > 1) {
    int max_multiprocessors = 0, max_device = 0;
    for (device = 0; device < num_devices; device++) {
      hipDeviceProp_t properties;
      hipGetDeviceProperties(&properties, device);
      if (max_multiprocessors < properties.multiProcessorCount) {
        max_multiprocessors = properties.multiProcessorCount;
        max_device = device;
      }
    }
    hipSetDevice(max_device);
  }

  struct fflux ff;
  for(int p=0;p<patches->size();p++) {
   // std::cout << "prev patch size: " << previousPatchSize << std::endl;
    const Patch* patch = patches->get(p);
    Vector dx = patch->dCell();
    double invvol = 1.0/(dx.x() * dx.y() * dx.z());

    d_OFS.initialize(ff);
    for(int m = 0;m<matls->size();m++){
      int matl = matls->get(m);

      // variable to get
      constCCVariable<double> mass;
      CCVariable<double>      mass2;
      CCVariable<double>      massAd;

      old_dw->get(mass, mass_label, matl, patch, Ghost::AroundCells, 1);
      new_dw->allocateAndPut(mass2, mass_label, matl, patch, Ghost::AroundCells, 1 );
      new_dw->allocateAndPut(massAd, massAdvected_label, matl, patch, Ghost::AroundCells, 1 );


      // Here the extents of the patch are extracted and the size of the domain is memory
      // needed is calculated.  Any memory allocation occur here.
      IntVector l = patch->getNodeLowIndex();
      IntVector h = patch->getNodeHighIndex();

      // Sizes //
      // Size of mass and advectedMass variables
      IntVector s = h - l;
      int xdim = s.x(), ydim = s.y(), zdim = s.z();
      size = xdim * ydim * zdim * sizeof(double);
      // Size of the OFS variable
      IntVector ofsSize = d_OFS.getWindow()->getData()->size();
      int sizeOFS = 6*ofsSize.x()*ofsSize.y()*ofsSize.z()*sizeof(double);

      l += IntVector(patch->getBCType(Patch::xminus) == Patch::Neighbor ? 0 : 1,
                     patch->getBCType(Patch::yminus) == Patch::Neighbor ? 0 : 1,
                     patch->getBCType(Patch::zminus) == Patch::Neighbor ? 0 : 1);
      h -= IntVector(patch->getBCType(Patch::xplus) == Patch::Neighbor ? 0 : 1,
                     patch->getBCType(Patch::yplus) == Patch::Neighbor ? 0 : 1,
                     patch->getBCType(Patch::zplus) == Patch::Neighbor ? 0 : 1);

      // check if we need to reallocate
      if (size != previousPatchSize) {
        if (previousPatchSize != 0) {
          hipFree(mass_device);
          hipFree(newMass_device);
          hipFree(massAd_device);
          hipFree(ofs_device);
        }
        hipMalloc(&mass_device, size);
        // Kernel error checking
        hipError_t error = hipGetLastError();
        if(error!=hipSuccess) {
          fprintf(stderr,"ERROR1: %s\n", hipGetErrorString(error) );
          exit(-1);
        }
        hipMalloc(&massAd_device, size);
        // Kernel error checking
        error = hipGetLastError();
        if(error!=hipSuccess) {
          fprintf(stderr,"ERROR3: %s\n", hipGetErrorString(error) );
          exit(-1);
        }
        hipMalloc(&ofs_device, sizeOFS);
        // Kernel error checking
        error = hipGetLastError();
        if(error!=hipSuccess) {
          fprintf(stderr,"ERROR3: %s\n", hipGetErrorString(error) );
          exit(-1);
        }
        hipMalloc(&newMass_device, size);
        // Kernel error checking
        error = hipGetLastError();
        if(error!=hipSuccess) {
          fprintf(stderr,"ERROR3: %s\n", hipGetErrorString(error) );
          exit(-1);
        }
      }

      //__________________________________
      //  Memory Allocation
      mass_host    = (double*)mass.getWindow()->getData()->getPointer();
      newMass_host = (double*)mass2.getWindow()->getData()->getPointer();
      massAd_host  = (double*)massAd.getWindow()->getData()->getPointer();
      ofs_host     = (double*)d_OFS.getWindow()->getData()->getPointer();
#ifdef GJDKLAFJKDLASJCLKDMSALKCJDASKLNGDA
      std::cout << "Dims of OFS: (" << ofsSize.x() <<","<< ofsSize.y() << "," << ofsSize.z() << ")" << std::endl;
      std::cout << "Size: " << ofsSize.x()*ofsSize.y()*ofsSize.z() << std::endl;
      std::cout << "Size of OFS: " <<  ofsSize.x()*ofssize.y()*ofsSize.z()* sizeof(struct fflux) << std::endl;
      std::cout << "Size: " << size/sizeof(double) << " x6= " << 6*size << std::endl;
#endif
      // allocate space on the device
      hipMemcpy(mass_device, mass_host, size,    hipMemcpyHostToDevice);
      //hipMemcpy(ofs_device,  ofs_host,  sizeOFS, hipMemcpyHostToDevice);


      uint3 domainSize  = make_uint3(xdim, ydim, zdim);
      uint3 domainLower = make_uint3(l.x(), l.y(), l.z());
      // Threads per block must be power of 2 in each direction.  Here
      //  8 is chosen as a test value in the x and y and 1 in the z,
      //  as each of these (x,y) threads streams through the z direction.
      dim3 threadsPerBlock(8, 8, 1);

      // Set up the number of blocks of threads in each direction accounting for any
      //  non-power of 8 end pieces.
      int xBlocks = xdim / 8;
      if( xdim % 8 != 0)
      {
        xBlocks++;
      }
      int yBlocks = ydim / 8;
      if( ydim % 8 != 0)
      {
        yBlocks++;
      }
      dim3 totalBlocks(xBlocks,yBlocks);

      // launch kernel
      timeAdvanceKernelAdvectSlabs<<< totalBlocks, threadsPerBlock >>>(domainSize, domainLower, ghostLayers, mass_device,  massAd_device, newMass_device, ofs_device, invvol);

      // Kernel error checking
      hipError_t error = hipGetLastError();
      if(error!=hipSuccess) {
        fprintf(stderr,"ERROR5: %s\n", hipGetErrorString(error) );
        exit(-1);
      }

      hipDeviceSynchronize();
      hipMemcpy(newMass_host, newMass_device, size, hipMemcpyDeviceToHost);
      //hipMemcpy(massAd_host,  massAd_device,  size, hipMemcpyDeviceToHost);
    }
  }

  // free up allocated memory
  hipFree(mass_device);
  hipFree(massAd_device);
  hipFree(newMass_device);
  hipFree(ofs_device);
}
