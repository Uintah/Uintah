#include "hip/hip_runtime.h"
/*
 * The MIT License
 *
 * Copyright (c) 1997-2016 The University of Utah
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to
 * deal in the Software without restriction, including without limitation the
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 */

#include <CCA/Components/Models/Radiation/RMCRT/RayGPU.cuh>
#include <CCA/Components/Schedulers/GPUDataWarehouse.h>
#include <CCA/Components/Schedulers/GPUMemoryPool.h>

#include <Core/Grid/Variables/GPUGridVariable.h>
#include <Core/Grid/Variables/GPUStencil7.h>
#include <Core/Grid/Variables/Stencil7.h>
#include <Core/Util/GPU.h>

#include <sci_defs/cuda_defs.h>
#include <sci_defs/uintah_defs.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DEBUG -9                 // 1: divQ, 2: boundFlux, 3: scattering
#define FIXED_RANDOM_NUM         // also edit in src/Core/Math/MersenneTwister.h to compare with Ray:CPU

//__________________________________
//  To Do
//  - Add rayDirectionHyperCube_cellFace()
//  - Need to implement transferFrom so use can use calc_frequency > 1
//  - Temporal scheduling
//  - restarts are not working.
//  - Investigate using multiple GPUs per node.
//  - Implement fixed and dynamic ROI.
//  - dynamic block size?
//  - Implement labelNames in unified memory.
//  - investigate the performance with different patch configurations
//  - deterministic random numbers
//  - Ray steps


//__________________________________
//
//  To use cuda-gdb on a single GPU you must set the environmental variable
//  CUDA_DEBUGGER_SOFTWARE_PREEMPTION=1
//
// mpirun -np 1 xterm -e cuda-gdb sus -gpu -nthreads 2 <args>
//__________________________________

namespace Uintah {

//---------------------------------------------------------------------------
// Kernel: The GPU ray tracer kernel
//---------------------------------------------------------------------------
template< class T>
__global__ void rayTraceKernel( dim3 dimGrid,
                                dim3 dimBlock,
                                const int matl,
                                levelParams level,
                                patchParams patch,
                                hiprandState* randNumStates,
                                RMCRT_flags RT_flags,
                                varLabelNames* labelNames,
                                GPUDataWarehouse* abskg_gdw,
                                GPUDataWarehouse* sigmaT4_gdw,
                                GPUDataWarehouse* cellType_gdw,
                                GPUDataWarehouse* old_gdw,
                                GPUDataWarehouse* new_gdw )
{

    // Not used right now
//  int blockID  = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
//  int threadID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;

  // calculate the thread indices
  int tidX = threadIdx.x + blockIdx.x * blockDim.x + patch.loEC.x;
  int tidY = threadIdx.y + blockIdx.y * blockDim.y + patch.loEC.y;

  const GPUGridVariable< T > sigmaT4OverPi;
  const GPUGridVariable< T > abskg;              // Need to use getRegion() to get the data
  const GPUGridVariable<int> cellType;

  GPUGridVariable<double> divQ;
  GPUGridVariable<GPUStencil7> boundFlux;
  GPUGridVariable<double> radiationVolQ;

//  sigmaT4_gdw->print();

  sigmaT4_gdw->getLevel( sigmaT4OverPi, "sigmaT4",  matl, level.index);
  cellType_gdw->getLevel( cellType,     "cellType", matl, level.index);

  if(RT_flags.usingFloats){
    abskg_gdw->getLevel( abskg, "abskgRMCRT",  matl, level.index);
  }else{
    abskg_gdw->getLevel( abskg, "abskg",       matl, level.index);
  }

  if( RT_flags.modifies_divQ ){
    new_gdw->getModifiable( divQ,         "divQ",           patch.ID, matl );
    new_gdw->getModifiable( boundFlux,    "RMCRTboundFlux", patch.ID, matl );
    new_gdw->getModifiable( radiationVolQ,"radiationVolq",  patch.ID, matl );
  }else{
    new_gdw->get( divQ,         "divQ",           patch.ID, matl );         // these should be allocateAndPut() calls
    new_gdw->get( boundFlux,    "RMCRTboundFlux", patch.ID, matl );
    new_gdw->get( radiationVolQ,"radiationVolq",  patch.ID, matl );


    // Extra Cell Loop
    if ( (tidX >= patch.loEC.x) && (tidY >= patch.loEC.y) && (tidX < patch.hiEC.x) && (tidY < patch.hiEC.y) ) { // patch boundary check
      #pragma unroll
      for (int z = patch.loEC.z; z < patch.hiEC.z; z++) { // loop through z slices
        GPUIntVector c = make_int3(tidX, tidY, z);
        divQ[c]          = 0.0;
        radiationVolQ[c] = 0.0;
      }
    }
  }

  //__________________________________
  //  Sanity checks
#if 0
  if (isThread0()) {
   printf("  GPUVariable Sanity check level: %i, patch: %i \n",level.index, patch.ID);
  }
#endif
  GPUVariableSanityCK(abskg,         patch.loEC, patch.hiEC);
  GPUVariableSanityCK(sigmaT4OverPi, patch.loEC, patch.hiEC);


  bool doLatinHyperCube = (RT_flags.rayDirSampleAlgo == LATIN_HYPER_CUBE);

  //______________________________________________________________________
  //           R A D I O M E T E R
  //______________________________________________________________________
  // TO BE FILLED IN

  //______________________________________________________________________
  //          B O U N D A R Y F L U X
  //______________________________________________________________________
  if( RT_flags.solveBoundaryFlux ){

    __shared__ int3 dirIndexOrder[6];
    __shared__ int3 dirSignSwap[6];

    //_____________________________________________
    //   Ordering for Surface Method
    // This block of code is used to properly place ray origins, and orient ray directions
    // onto the correct face.  This is necessary, because by default, the rays are placed
    // and oriented onto a default face, then require adjustment onto the proper face.
    dirIndexOrder[EAST]   = make_int3(2, 1, 0);
    dirIndexOrder[WEST]   = make_int3(2, 1, 0);
    dirIndexOrder[NORTH]  = make_int3(0, 2, 1);
    dirIndexOrder[SOUTH]  = make_int3(0, 2, 1);
    dirIndexOrder[TOP]    = make_int3(0, 1, 2);
    dirIndexOrder[BOT]    = make_int3(0, 1, 2);

    // Ordering is slightly different from 6Flux since here, rays pass through origin cell from the inside faces.
    dirSignSwap[EAST]     = make_int3(-1, 1,  1);
    dirSignSwap[WEST]     = make_int3( 1, 1,  1);
    dirSignSwap[NORTH]    = make_int3( 1, -1, 1);
    dirSignSwap[SOUTH]    = make_int3( 1, 1,  1);
    dirSignSwap[TOP]      = make_int3( 1, 1, -1);
    dirSignSwap[BOT]      = make_int3( 1, 1,  1);

    //__________________________________
    // GPU equivalent of GridIterator loop - calculate sets of rays per thread
    if ( (tidX >= patch.lo.x) && (tidY >= patch.lo.y) && (tidX < patch.hi.x) && (tidY < patch.hi.y) ) { // patch boundary check
      #pragma unroll
      for (int z = patch.lo.z; z < patch.hi.z; z++) { // loop through z slices

        GPUIntVector origin = make_int3(tidX, tidY, z);  // for each thread

        boundFlux[origin].initialize(0.0);

        BoundaryFaces boundaryFaces;

         // which surrounding cells are boundaries
        boundFlux[origin].p = has_a_boundaryDevice(origin, cellType, boundaryFaces);
        
        GPUPoint CC_pos = level.getCellPosition(origin);

        //__________________________________
        // Loop over boundary faces of the cell and compute incident radiative flux
        #pragma unroll
        for( int i = 0; i<boundaryFaces.size(); i++) {

          int RayFace = boundaryFaces.faceArray[i];
          int UintahFace[6] = {WEST,EAST,SOUTH,NORTH,BOT,TOP};

          double sumI     = 0;
          double sumProjI = 0;
          double sumI_prev= 0;

          //__________________________________
          // Flux ray loop
          #pragma unroll
          for (int iRay=0; iRay < RT_flags.nFluxRays; iRay++){

            GPUVector direction_vector;
            GPUVector rayOrigin;
            double cosTheta;

//  Need to add rayDirectionHyperCube_cellFace Call  -Todd                     <<<<<<<<<<<<<<<<<<<<<<<<<,,,

            rayDirection_cellFaceDevice( randNumStates, origin, dirIndexOrder[RayFace], dirSignSwap[RayFace], iRay,
                                   direction_vector, cosTheta );

            rayLocation_cellFaceDevice( randNumStates, RayFace, patch.dx, CC_pos, rayOrigin);

            updateSumIDevice< T >( level, direction_vector, rayOrigin, origin, patch.dx, sigmaT4OverPi, abskg, cellType, sumI, randNumStates, RT_flags);

            sumProjI += cosTheta * (sumI - sumI_prev);   // must subtract sumI_prev, since sumI accumulates intensity

            sumI_prev = sumI;

          } // end of flux ray loop

          //__________________________________
          //  Compute Net Flux to the boundary
          int face = UintahFace[RayFace];
          boundFlux[origin][ face ] = sumProjI * 2 *M_PI/RT_flags.nFluxRays;

/*`==========TESTING==========*/
#if DEBUG == 2
          printf( "\n      [%d, %d, %d]  face: %d sumProjI:  %g BF: %g\n",
                    origin.x, origin.y, origin.z, face, sumProjI, boundFlux[origin][ face ]);
#endif
/*===========TESTING==========`*/

        } // boundary faces loop
      }  // z slices loop
    }  // X-Y Thread loop
  }


  //______________________________________________________________________
  //         S O L V E   D I V Q
  //______________________________________________________________________
  if( RT_flags.solveDivQ ){
    const int nDivQRays = RT_flags.nDivQRays;               // for readability

    //int rand_i[ doLatinHyperCube ? nDivQRays : 0 ];                                        // only needed for LHC scheme
    const int size = 1000;                                         // FIX ME Todd
    int rand_i[ size ];                                      // FIX ME TODD

    // GPU equivalent of GridIterator loop - calculate sets of rays per thread
    if ( (tidX >= patch.lo.x) && (tidY >= patch.lo.y) && (tidX < patch.hi.x) && (tidY < patch.hi.y) ) { // patch boundary check
      #pragma unroll
      for (int z = patch.lo.z; z < patch.hi.z; z++) {       // loop through z slices

        GPUIntVector origin = make_int3(tidX, tidY, z);     // for each thread
        double sumI = 0;
        GPUPoint CC_pos = level.getCellPosition(origin);

        if (doLatinHyperCube){
          randVectorDevice(rand_i, size,randNumStates);
        }

        //__________________________________
        // ray loop
        #pragma unroll

        for (int iRay = 0; iRay < nDivQRays; iRay++) {

          GPUVector direction_vector;
          if ( doLatinHyperCube ){                          // Latin-Hyper-Cube sampling
            direction_vector = findRayDirectionHyperCubeDevice(randNumStates, nDivQRays, rand_i[iRay], iRay );
          }else{                                            // Naive Monte-Carlo sampling
            direction_vector = findRayDirectionDevice( randNumStates );
          }
          
          GPUVector rayOrigin = rayOriginDevice( randNumStates, CC_pos, patch.dx, RT_flags.CCRays );

          updateSumIDevice< T >( level, direction_vector, rayOrigin, origin, patch.dx,  sigmaT4OverPi, abskg, cellType, sumI, randNumStates, RT_flags);
        } //Ray loop

        //__________________________________
        //  Compute divQ
        divQ[origin] = -4.0 * M_PI * abskg[origin] * ( sigmaT4OverPi[origin] - (sumI/RT_flags.nDivQRays) );

        // radiationVolq is the incident energy per cell (W/m^3) and is necessary when particle heat transfer models (i.e. Shaddix) are used
        radiationVolQ[origin] = 4.0 * M_PI * abskg[origin] *  (sumI/RT_flags.nDivQRays) ;

/*`==========TESTING==========*/
#if DEBUG == 1
        if( isDbgCellDevice( origin ) ){
          printf( "\n      [%d, %d, %d]  sumI: %1.16e  divQ: %1.16e radiationVolq: %1.16e  abskg: %1.16e,    sigmaT4: %1.16e \n",
                    origin.x, origin.y, origin.z, sumI,divQ[origin], radiationVolQ[origin],abskg[origin], sigmaT4OverPi[origin]);
        }
#endif
/*===========TESTING==========`*/
      }  // end z-slice loop
    }  // end domain boundary check
  }  // solve divQ
}  // end ray trace kernel

//---------------------------------------------------------------------------
// Kernel: The GPU ray tracer data onion kernel
//---------------------------------------------------------------------------
// hard-wired for 2-levels now, but this should be fast and fixes
__constant__ levelParams d_levels[d_MAXLEVELS];

template< class T>
__global__ void rayTraceDataOnionKernel( dim3 dimGrid,
                                         dim3 dimBlock,
                                         int matl,
                                         patchParams finePatch,
                                         gridParams gridP,
                                         GPUIntVector fineLevel_ROI_Lo,
                                         GPUIntVector fineLevel_ROI_Hi,
                                         int3* regionLo,
                                         int3* regionHi,
                                         hiprandState* randNumStates,
                                         RMCRT_flags RT_flags,
                                         GPUDataWarehouse* abskg_gdw,
                                         GPUDataWarehouse* sigmaT4_gdw,
                                         GPUDataWarehouse* cellType_gdw,
                                         GPUDataWarehouse* old_gdw,
                                         GPUDataWarehouse* new_gdw )
{

    // Not used right now
//  int blockID  = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
//  int threadID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;


  // calculate the thread indices
  int tidX = threadIdx.x + blockIdx.x * blockDim.x + finePatch.loEC.x;
  int tidY = threadIdx.y + blockIdx.y * blockDim.y + finePatch.loEC.y;

#if 0
  if (tidX == 1 && tidY == 1) {
    printf("\nGPU levelParams\n");

    printf("Level-0 ");
    d_levels[0].print();

    printf("Level-1 ");
    d_levels[1].print();
  }
#endif


  int maxLevels = gridP.maxLevels;
  int fineL = maxLevels - 1;

  //__________________________________
  //
  const GPUGridVariable<T>    abskg[d_MAXLEVELS];
  const GPUGridVariable<T>    sigmaT4OverPi[d_MAXLEVELS];
  const GPUGridVariable<int>  cellType[d_MAXLEVELS];

//  new_gdw->print();

  //__________________________________
  // coarse level data for the entire level
  for (int l = 0; l < maxLevels; ++l) {
    if (d_levels[l].hasFinerLevel) {
      abskg_gdw->getLevel( abskg[l],           "abskg",    matl, l);
      sigmaT4_gdw->getLevel( sigmaT4OverPi[l], "sigmaT4",  matl, l);
      cellType_gdw->getLevel( cellType[l],     "cellType", matl, l);

      GPUVariableSanityCK(abskg[l],        d_levels[l].regionLo,d_levels[l].regionHi);
      GPUVariableSanityCK(sigmaT4OverPi[l],d_levels[l].regionLo,d_levels[l].regionHi);
    }
  }

  //__________________________________
  //  fine level data for the region of interest.
  //  ToDo:  replace get with getRegion() calls so
  //  so the halo can be > 0
  if ( RT_flags.whichROI_algo == patch_based ) {

    abskg_gdw->get(abskg[fineL],           "abskg",    finePatch.ID, matl, fineL);
    sigmaT4_gdw->get(sigmaT4OverPi[fineL], "sigmaT4",  finePatch.ID, matl, fineL);
    cellType_gdw->get(cellType[fineL],     "cellType", finePatch.ID, matl, fineL);

    GPUVariableSanityCK(abskg[fineL],        fineLevel_ROI_Lo,fineLevel_ROI_Hi);
    GPUVariableSanityCK(sigmaT4OverPi[fineL],fineLevel_ROI_Lo,fineLevel_ROI_Hi);
  }

  GPUGridVariable<double> divQ;
  GPUGridVariable<GPUStencil7> boundFlux;
  GPUGridVariable<double> radiationVolQ;

  //__________________________________
  //  fine level data for this patch
  if( RT_flags.modifies_divQ ){
    new_gdw->getModifiable( divQ,         "divQ",          finePatch.ID, matl, fineL );
    new_gdw->getModifiable( boundFlux,    "boundFlux",     finePatch.ID, matl, fineL );
    new_gdw->getModifiable( radiationVolQ,"radiationVolq", finePatch.ID, matl, fineL );
  }else{
    new_gdw->get( divQ,         "divQ",           finePatch.ID, matl, fineL );         // these should be allocateAntPut() calls
    new_gdw->get( boundFlux,    "RMCRTboundFlux", finePatch.ID, matl, fineL );
    new_gdw->get( radiationVolQ,"radiationVolq",  finePatch.ID, matl, fineL );


    //__________________________________
    // initialize Extra Cell Loop
    if ( (tidX >= finePatch.loEC.x) && (tidY >= finePatch.loEC.y) && (tidX < finePatch.hiEC.x) && (tidY < finePatch.hiEC.y) ) { // finePatch boundary check
      #pragma unroll
      for (int z = finePatch.loEC.z; z < finePatch.hiEC.z; z++) { // loop through z slices
        GPUIntVector c = make_int3(tidX, tidY, z);
        divQ[c]          = 0.0;
        radiationVolQ[c] = 0.0;
      }
    }
  }

  //______________________________________________________________________
  //           R A D I O M E T E R
  //______________________________________________________________________
  // TO BE FILLED IN



  //______________________________________________________________________
  //          B O U N D A R Y F L U X
  //______________________________________________________________________
  if( RT_flags.solveBoundaryFlux ){
    // TO BE FILLED IN
  }


#if 1
  //______________________________________________________________________
  //         S O L V E   D I V Q
  //______________________________________________________________________
  if( RT_flags.solveDivQ ) {

    // GPU equivalent of GridIterator loop - calculate sets of rays per thread
    if ( (tidX >= finePatch.lo.x) && (tidY >= finePatch.lo.y) && (tidX < finePatch.hi.x) && (tidY < finePatch.hi.y) ) { // finePatch boundary check
      #pragma unroll
      for (int z = finePatch.lo.z; z < finePatch.hi.z; z++) { // loop through z slices

        GPUIntVector origin = make_int3(tidX, tidY, z);  // for each thread

/*`==========TESTING==========*/
#if 0
        if( !isDbgCellDevice( origin ) ){
          return;
        }
     printf(" origin[%i,%i,%i] finePatchID: %i \n", origin.x, origin.y, origin.z, finePatch.ID);
#endif
/*===========TESTING==========`*/

        double sumI = 0;

        //__________________________________
        // ray loop
        #pragma unroll
        for (int iRay = 0; iRay < RT_flags.nDivQRays; iRay++) {

          GPUVector ray_direction = findRayDirectionDevice( randNumStates );

          GPUVector rayOrigin = rayOriginDevice( randNumStates, origin, d_levels[fineL].DyDx, d_levels[fineL].DzDx , RT_flags.CCRays );

          updateSumI_MLDevice<T>(ray_direction, rayOrigin, origin, gridP,
                                 fineLevel_ROI_Lo, fineLevel_ROI_Hi,
                                 regionLo, regionHi,
                                 sigmaT4OverPi, abskg, cellType, sumI, randNumStates, RT_flags);
        } //Ray loop

        //__________________________________
        //  Compute divQ
        divQ[origin] = -4.0 * M_PI * abskg[fineL][origin] * ( sigmaT4OverPi[fineL][origin] - (sumI/RT_flags.nDivQRays) );

        // radiationVolq is the incident energy per cell (W/m^3) and is necessary when particle heat transfer models (i.e. Shaddix) are used
        radiationVolQ[origin] = 4.0 * M_PI * abskg[fineL][origin] *  (sumI/RT_flags.nDivQRays) ;


/*`==========TESTING==========*/
#if DEBUG == 1
       if( isDbgCellDevice(origin) ){
          printf( "\n      [%d, %d, %d]  sumI: %g  divQ: %g radiationVolq: %g  abskg: %g,    sigmaT4: %g \n",
                    origin.x, origin.y, origin.z, sumI,divQ[origin], radiationVolQ[origin],abskg[fineL][origin], sigmaT4OverPi[fineL][origin]);
       }
#endif
/*===========TESTING==========`*/

      }  // end z-slice loop
    }  // end ROI loop
  }  // solve divQ
#endif

}

//______________________________________________________________________
//
//______________________________________________________________________
__device__ GPUVector findRayDirectionDevice( hiprandState* randNumStates )
{
  // Random Points On Sphere
  // add fuzz to prevent infs in 1/dirVector calculation
  double plusMinus_one = 2.0 * randDblExcDevice( randNumStates ) - 1.0 + DBL_EPSILON;
  double r = sqrt(1.0 - plusMinus_one * plusMinus_one);             // Radius of circle at z
  double theta = 2.0 * M_PI * randDblExcDevice( randNumStates );    // Uniform betwen 0-2Pi

  GPUVector dirVector;
  dirVector.x = r*cos(theta);   // Convert to cartesian coordinates
  dirVector.y = r*sin(theta);
  dirVector.z = plusMinus_one;

  return dirVector;
}


//______________________________________________________________________
//
__device__ GPUVector findRayDirectionHyperCubeDevice(hiprandState* randNumStates,
                                                     const int nDivQRays,
                                                     const int bin_i,
                                                     const int bin_j)
{
  // Random Points On Sphere
  double plusMinus_one = 2.0 *(randDblExcDevice( randNumStates ) + (double) bin_i)/nDivQRays - 1.0;

  // Radius of circle at z
  double r = sqrt(1.0 - plusMinus_one * plusMinus_one);

  // Uniform betwen 0-2Pi
  double phi = 2.0 * M_PI * (randDblExcDevice( randNumStates ) + (double) bin_j)/nDivQRays;

  GPUVector dirVector;
  dirVector[0] = r*cos(phi);                       // Convert to cartesian
  dirVector[1] = r*sin(phi);
  dirVector[2] = plusMinus_one;

  return dirVector;
}
//______________________________________________________________________
//  Populate vector with integers which have been randomly shuffled.
//  This is sampling without replacement and can be used to in a
//  Latin-Hyper-Cube sampling scheme.  The algorithm used is the
//  modern Fisher-Yates shuffle.
//______________________________________________________________________
__device__ void randVectorDevice( int int_array[],
                                  const int size,
                                  hiprandState* randNumStates ){

  for (int i=0; i<size; i++){   // populate sequential array from 0 to size-1
    int_array[i] = i;
  }

  for (int i=size-1; i>0; i--){  // fisher-yates shuffle starting with size-1
    int rand_int =  randIntDevice(randNumStates, i);    // Random number between 0 & i
    int swap = int_array[i];
    int_array[i] = int_array[rand_int];
    int_array[rand_int] = swap;
  }
}
//______________________________________________________________________
// Compute the Ray direction from a cell face
__device__ void rayDirection_cellFaceDevice( hiprandState* randNumStates,
                                             const GPUIntVector& origin,
                                             const GPUIntVector& indexOrder,
                                             const GPUIntVector& signOrder,
                                             const int iRay,
                                             GPUVector& directionVector,
                                             double& cosTheta )
{
  // Surface Way to generate a ray direction from the positive z face
  double phi = 2 * M_PI * randDblDevice(randNumStates);  // azimuthal angle.  Range of 0 to 2pi
  double theta = acos(randDblDevice(randNumStates));     // polar angle for the hemisphere
  cosTheta = cos(theta);
  double sinTheta = sin(theta);

  //Convert to Cartesian
  GPUVector tmp;
  tmp[0] = sinTheta * cos(phi);
  tmp[1] = sinTheta * sin(phi);
  tmp[2] = cosTheta;

  // Put direction vector as coming from correct face,
  directionVector[0] = tmp[indexOrder[0]] * signOrder[0];
  directionVector[1] = tmp[indexOrder[1]] * signOrder[1];
  directionVector[2] = tmp[indexOrder[2]] * signOrder[2];
}


//______________________________________________________________________
//

// Used in dataOnion.  This will be removed soon.
__device__ GPUVector rayOriginDevice( hiprandState* randNumStates,
                                      const GPUIntVector origin,
                                      const double DyDx,
                                      const double DzDx,
                                      const bool useCCRays )
{
  GPUVector rayOrigin;
  if (useCCRays == false) {
    rayOrigin.x = (double)origin.x + randDblDevice(randNumStates);
    rayOrigin.y = (double)origin.y + randDblDevice(randNumStates) * DyDx;
    rayOrigin.z = (double)origin.z + randDblDevice(randNumStates) * DzDx;
  }
  else {
    rayOrigin.x = origin.x + 0.5;
    rayOrigin.y = origin.y + 0.5 * DyDx;
    rayOrigin.z = origin.z + 0.5 * DzDx;
  }
  return rayOrigin;
}

__device__ 
GPUVector rayOriginDevice( hiprandState* randNumStates,
                           const GPUPoint  CC_pos,
                           const GPUVector dx,
                           const bool   useCCRays)
{
  GPUVector rayOrigin;
  if( useCCRays == false ){
    rayOrigin[0] =  CC_pos.x - 0.5*dx.x  + randDblDevice(randNumStates) * dx.x; 
    rayOrigin[1] =  CC_pos.y - 0.5*dx.y  + randDblDevice(randNumStates) * dx.y; 
    rayOrigin[2] =  CC_pos.z - 0.5*dx.z  + randDblDevice(randNumStates) * dx.z;
  }else{
    rayOrigin[0] = CC_pos.x;
    rayOrigin[1] = CC_pos.y;
    rayOrigin[2] = CC_pos.z;
  }
  return rayOrigin;
}

//______________________________________________________________________
//  Compute the Ray location from a cell face
__device__ void rayLocation_cellFaceDevice( hiprandState* randNumStates,
                                            const GPUIntVector& origin,
                                            const GPUIntVector &indexOrder,
                                            const GPUIntVector &shift,
                                            const double &DyDx,
                                            const double &DzDx,
                                            GPUVector& location )
{
  GPUVector tmp;
  tmp[0] = randDblDevice(randNumStates);
  tmp[1] = 0;
  tmp[2] = randDblDevice(randNumStates) * DzDx;

  // Put point on correct face
  location[0] = tmp[indexOrder[0]] + (double)shift[0];
  location[1] = tmp[indexOrder[1]] + (double)shift[1] * DyDx;
  location[2] = tmp[indexOrder[2]] + (double)shift[2] * DzDx;

  location[0] += (double)origin.x;
  location[1] += (double)origin.y;
  location[2] += (double)origin.z;
}
//______________________________________________________________________
//
//  Compute the Ray location on a cell face
__device__ void rayLocation_cellFaceDevice( hiprandState* randNumStates,
                                            const int face,
                                            const GPUVector Dx,
                                            const GPUPoint CC_pos,
                                            GPUVector& rayOrigin)
{
  double cellOrigin[3];
  // left, bottom, back corner of the cell
  cellOrigin[X] = CC_pos.x - 0.5 * Dx[X];
  cellOrigin[Y] = CC_pos.y - 0.5 * Dx[Y];
  cellOrigin[Z] = CC_pos.z - 0.5 * Dx[Z];

  switch(face)
  {
    case WEST:
      rayOrigin[X] = cellOrigin[X];
      rayOrigin[Y] = cellOrigin[Y] + randDblDevice(randNumStates) * Dx[Y];
      rayOrigin[Z] = cellOrigin[Z] + randDblDevice(randNumStates) * Dx[Z];
      break;
    case EAST:
      rayOrigin[X] = cellOrigin[X] +  Dx[X];
      rayOrigin[Y] = cellOrigin[Y] + randDblDevice(randNumStates) * Dx[Y];
      rayOrigin[Z] = cellOrigin[Z] + randDblDevice(randNumStates) * Dx[Z];
      break;
    case SOUTH:
      rayOrigin[X] = cellOrigin[X] + randDblDevice(randNumStates) * Dx[X];
      rayOrigin[Y] = cellOrigin[Y];
      rayOrigin[Z] = cellOrigin[Z] + randDblDevice(randNumStates) * Dx[Z];
      break;
    case NORTH:
      rayOrigin[X] = cellOrigin[X] + randDblDevice(randNumStates) * Dx[X];
      rayOrigin[Y] = cellOrigin[Y] + Dx[Y];
      rayOrigin[Z] = cellOrigin[Z] + randDblDevice(randNumStates) * Dx[Z];
      break;
    case BOT:
      rayOrigin[X] = cellOrigin[X] + randDblDevice(randNumStates) * Dx[X];
      rayOrigin[Y] = cellOrigin[Y] + randDblDevice(randNumStates) * Dx[Y];
      rayOrigin[Z] = cellOrigin[Z];
      break;
    case TOP:
      rayOrigin[X] = cellOrigin[X] + randDblDevice(randNumStates) * Dx[X];
      rayOrigin[Y] = cellOrigin[Y] + randDblDevice(randNumStates) * Dx[Y];
      rayOrigin[Z] = cellOrigin[Z] + Dx[Z];
      break;
    default:
//      throw InternalError("Ray::rayLocation_cellFace,  Invalid FaceType Specified", __FILE__, __LINE__);
      return;
  }
}
//______________________________________________________________________
//
__device__ bool has_a_boundaryDevice(const GPUIntVector &c,
                                     const GPUGridVariable<int>& celltype,
                                     BoundaryFaces &boundaryFaces){

  GPUIntVector adj = c;
  bool hasBoundary = false;

  adj[0] = c[0] - 1;     // west

  if ( celltype[adj]+1 ){              // cell type of flow is -1, so when cellType+1 isn't false, we
    boundaryFaces.addFace( WEST );     // know we're at a boundary
    hasBoundary = true;
  }

  adj[0] += 2;           // east

  if ( celltype[adj]+1 ){
    boundaryFaces.addFace( EAST );
    hasBoundary = true;
  }

  adj[0] -= 1;
  adj[1] = c[1] - 1;     // south

  if ( celltype[adj]+1 ){
    boundaryFaces.addFace( SOUTH );
    hasBoundary = true;
  }

  adj[1] += 2;           // north

  if ( celltype[adj]+1 ){
    boundaryFaces.addFace( NORTH );
    hasBoundary = true;
  }

  adj[1] -= 1;
  adj[2] = c[2] - 1;     // bottom

  if ( celltype[adj]+1 ){
    boundaryFaces.addFace( BOT );
    hasBoundary = true;
  }

  adj[2] += 2;           // top

  if ( celltype[adj]+1 ){
    boundaryFaces.addFace( TOP );
    hasBoundary = true;
  }

  return (hasBoundary);
}


//______________________________________________________________________
//
__device__ void findStepSizeDevice(int step[],
                                   bool sign[],
                                   const GPUVector& inv_direction_vector)
{
  // get new step and sign
  for ( int d= 0; d<3; d++ ){

    if (inv_direction_vector[d]>0){
      step[d] = 1;
      sign[d] = 1;
    }else{
      step[d] = -1;
      sign[d] = 0;
    }
  }
}

//______________________________________________________________________
//
//______________________________________________________________________
__device__ void raySignStepDevice(GPUVector& sign,
                                  int cellStep[],
                                  const GPUVector& inv_direction_vector)
{
  // get new step and sign
  for ( int d=0; d<3; d++){
    double me = copysign((double)1.0, inv_direction_vector[d]);  // +- 1
    
    sign[d] = max(0.0, me);    // 0, 1
    
    cellStep[d] = int(me);
  }
}


//______________________________________________________________________
//
__device__ bool containsCellDevice( GPUIntVector low,
                                    GPUIntVector high,
                                    GPUIntVector cell,
                                    const int dir)
{
  return  low[dir] <= cell[dir] &&
          high[dir] > cell[dir];
}

//______________________________________________________________________
//          // used by dataOnion it will be replaced
__device__ void reflect(double& fs,
                         GPUIntVector& cur,
                         GPUIntVector& prevCell,
                         const double abskg,
                         bool& in_domain,
                         int& step,
                         bool& sign,
                         double& ray_direction)
{
  fs = fs * (1 - abskg);

  //put cur back inside the domain
  cur = prevCell;
  in_domain = true;

  // apply reflection condition
  step *= -1;                // begin stepping in opposite direction
  sign = (sign==1) ? 0 : 1;  //  swap sign from 1 to 0 or vice versa
  ray_direction *= -1;
}


__device__ void reflect(double& fs,
                         GPUIntVector& cur,
                         GPUIntVector& prevCell,
                         const double abskg,
                         bool& in_domain,
                         int& step,
                         double& sign,
                         double& ray_direction)
{
  fs = fs * (1 - abskg);

  //put cur back inside the domain
  cur = prevCell;
  in_domain = true;

  // apply reflection condition
  step *= -1;                // begin stepping in opposite direction
  sign *= -1;
  ray_direction *= -1;
}


//______________________________________________________________________
template< class T >
__device__ void updateSumIDevice ( levelParams level,
                                   GPUVector& ray_direction,
                                   GPUVector& ray_origin,
                                   const GPUIntVector& origin,
                                   const GPUVector& Dx,
                                   const GPUGridVariable< T >& sigmaT4OverPi,
                                   const GPUGridVariable< T >& abskg,
                                   const GPUGridVariable<int>& celltype,
                                   double& sumI,
                                   hiprandState* randNumStates,
                                   RMCRT_flags RT_flags)

{


  GPUIntVector cur = origin;
  GPUIntVector prevCell = cur;
  // Step and sign for ray marching
  int step[3];                              // Gives +1 or -1 based on sign
  GPUVector sign;                           //   is 0 for negative ray direction

  GPUVector inv_ray_direction = 1.0/ray_direction;
/*`==========TESTING==========*/
#if DEBUG == 1
  if( isDbgCellDevice(origin) ) {
    printf("        updateSumI: [%d,%d,%d] ray_dir [%g,%g,%g] ray_loc [%g,%g,%g]\n", origin.x, origin.y, origin.z,ray_direction.x, ray_direction.y, ray_direction.z, ray_origin.x, ray_origin.y, ray_origin.z);
  }
#endif
/*===========TESTING==========`*/

  raySignStepDevice(sign, step, ray_direction);


  GPUPoint CC_pos = level.getCellPosition(origin);
  
  // rayDx is the distance from bottom, left, back, corner of cell to ray
  GPUVector rayDx;
  rayDx[0] = ray_origin.x - ( CC_pos.x - 0.5*Dx.x );         // this can be consolidated using GPUVector
  rayDx[1] = ray_origin.y - ( CC_pos.y - 0.5*Dx.y );
  rayDx[2] = ray_origin.z - ( CC_pos.z - 0.5*Dx.z );
  
  GPUVector tMax;
  tMax.x = (sign.x * Dx.x - rayDx.x) * inv_ray_direction.x;
  tMax.y = (sign.y * Dx.y - rayDx.y) * inv_ray_direction.y;
  tMax.z = (sign.z * Dx.z - rayDx.z) * inv_ray_direction.z;

  //Length of t to traverse one cell
  GPUVector tDelta;
  tDelta   = Abs(inv_ray_direction) * Dx;

  //Initializes the following values for each ray
  bool in_domain     = true;
  double tMax_prev   = 0;
  double intensity   = 1.0;
  double fs          = 1.0;
  int nReflect       = 0;                 // Number of reflections
  double optical_thickness      = 0;
  double expOpticalThick_prev   = 1.0;
  double rayLength              = 0.0;
  GPUVector ray_location        = ray_origin;


#ifdef RAY_SCATTER
  double scatCoeff = RT_flags.sigmaScat;          //[m^-1]  !! HACK !! This needs to come from data warehouse
  if (scatCoeff == 0) scatCoeff = 1e-99;  // avoid division by zero

  // Determine the length at which scattering will occur
  // See CCA/Components/Arches/RMCRT/PaulasAttic/MCRT/ArchesRMCRT/ray.cc
  double scatLength = -log( randDblExcDevice( randNumStates ) ) / scatCoeff;
#endif

  //+++++++Begin ray tracing+++++++++++++++++++
  //Threshold while loop
  while ( intensity > RT_flags.threshold ){

    DIR dir = NONE;

    while (in_domain){

      prevCell = cur;
      double disMin = -9;          // Represents ray segment length.

      //__________________________________
      //  Determine which cell the ray will enter next
      dir = NONE;
      if ( tMax.x < tMax.y ){        // X < Y
        if ( tMax.x < tMax.z ){      // X < Z
          dir = X;
        } else {
          dir = Z;
        }
      } else {
        if( tMax.y < tMax.z ){       // Y < Z
          dir = Y;
        } else {
          dir = Z;
        }
      }

      //__________________________________
      //  update marching variables
      cur[dir]  = cur[dir] + step[dir];
      disMin    = (tMax[dir] - tMax_prev);
      tMax_prev = tMax[dir];
      tMax[dir] = tMax[dir] + tDelta[dir];
      rayLength += disMin;

      ray_location.x = ray_location.x + (disMin  * ray_direction.x);
      ray_location.y = ray_location.y + (disMin  * ray_direction.y);
      ray_location.z = ray_location.z + (disMin  * ray_direction.z);

      in_domain = (celltype[cur]==-1);  //cellType of -1 is flow         HARDWIRED WARNING

      optical_thickness += abskg[prevCell]*disMin; 

      RT_flags.nRaySteps ++;
      
/*`==========TESTING==========*/
#if ( DEBUG >= 1 )
if( isDbgCellDevice(origin) ){
    printf( "            cur [%d,%d,%d] prev [%d,%d,%d] ", cur.x, cur.y, cur.z, prevCell.x, prevCell.y, prevCell.z);
    printf( " dir %d ", dir );
    printf( "tMax [%g,%g,%g] ",tMax.x,tMax.y, tMax.z);
    printf( "rayLoc [%g,%g,%g] ",ray_location.x,ray_location.y, ray_location.z);
    printf( "disMin %g tMax[dir]: %g tMax_prev: %g, Dx[dir]: %g\n",disMin, tMax[dir], tMax_prev, Dx[dir]);

    printf( "            abskg[prev] %g  \t sigmaT4OverPi[prev]: %g \n",abskg[prevCell],  sigmaT4OverPi[prevCell]);
    printf( "            abskg[cur]  %g  \t sigmaT4OverPi[cur]:  %g  \t  cellType: %i\n",abskg[cur], sigmaT4OverPi[cur], celltype[cur] );
    printf( "            optical_thickkness %g \t rayLength: %g\n", optical_thickness, rayLength);
}
#endif

/*===========TESTING==========`*/


      //Eqn 3-15(see below reference) while
      //Third term inside the parentheses is accounted for in Inet. Chi is accounted for in Inet calc.
      double expOpticalThick = exp(-optical_thickness);

      sumI += sigmaT4OverPi[prevCell] * ( expOpticalThick_prev - expOpticalThick ) * fs;

      expOpticalThick_prev = expOpticalThick;


#ifdef RAY_SCATTER
      if ( (rayLength > scatLength) && in_domain){

        // get new scatLength for each scattering event
        scatLength = -log( randDblExcDevice( randNumStates ) ) / scatCoeff;

        ray_direction     = findRayDirectionDevice( randNumStates );

        inv_ray_direction = 1.0/ray_direction;

        // get new step and sign
        int stepOld = step[dir];
        raySignStepDevice( sign, step, ray_direction);

        // if sign[dir] changes sign, put ray back into prevCell (back scattering)
        // a sign change only occurs when the product of old and new is negative
        if( step[dir] * stepOld < 0 ){
          cur = prevCell;
        }
        
        GPUPoint CC_pos = level.getCellPosition(cur);
        
         // rayDx is the distance from bottom, left, back, corner of cell to ray
        rayDx[0] = ray_origin.x - ( CC_pos.x - 0.5*Dx.x );         // this can be consolidated using GPUVector
        rayDx[1] = ray_origin.y - ( CC_pos.y - 0.5*Dx.y );
        rayDx[2] = ray_origin.z - ( CC_pos.z - 0.5*Dx.z );

        tMax.x = (sign.x * Dx.x - rayDx.x) * inv_ray_direction.x;
        tMax.y = (sign.y * Dx.y - rayDx.y) * inv_ray_direction.y;
        tMax.z = (sign.z * Dx.z - rayDx.z) * inv_ray_direction.z;

        // Length of t to traverse one cell
        tDelta    = Abs(inv_ray_direction) * Dx;
        tMax_prev = 0;
        rayLength = 0;  // allow for multiple scattering events per ray

/*`==========TESTING==========*/
#if (DEBUG == 3)
        if( isDbgCellDevice( origin)  ){
          printf( "            Scatter: [%i, %i, %i], rayLength: %g, tmax: %g, %g, %g  tDelta: %g, %g, %g  ray_dir: %g, %g, %g\n",cur.x, cur.y, cur.z,rayLength, tMax[0], tMax[1], tMax[2], tDelta.x, tDelta.y , tDelta.z, ray_direction.x, ray_direction.y , ray_direction.z);
          printf( "                    dir: %i sign: [%g, %g, %g], step [%i, %i, %i] cur: [%i, %i, %i], prevCell: [%i, %i, %i]\n", dir, sign[0], sign[1], sign[2], step[0], step[1], step[2], cur[0], cur[1], cur[2], prevCell[0], prevCell[1], prevCell[2] );
          printf( "                    ray_location: [%g, %g, %g]\n", rayLocation[0], rayLocation[1], rayLocation[2] );
//          printf("                     rayDx         [%g, %g, %g]  CC_pos[%g, %g, %g]\n", rayDx[0], rayDx[1], rayDx[2], CC_pos.x, CC_pos.y, CC_pos.z);
        }
#endif
/*===========TESTING==========`*/

      }
#endif

    } //end domain while loop.  ++++++++++++++

    //  wall emission 12/15/11
    double wallEmissivity = abskg[cur];

    if (wallEmissivity > 1.0){       // Ensure wall emissivity doesn't exceed one.
      wallEmissivity = 1.0;
    }

    intensity = exp(-optical_thickness);

    sumI += wallEmissivity * sigmaT4OverPi[cur] * intensity;

    intensity = intensity * fs;


    // when a ray reaches the end of the domain, we force it to terminate.
    if( !RT_flags.allowReflect ) intensity = 0;


/*`==========TESTING==========*/
#if DEBUG >0
if( isDbgCellDevice(origin) ){
    printf( "            cur [%d,%d,%d] intensity: %g expOptThick: %g, fs: %g allowReflect: %i \n",
            cur.x, cur.y, cur.z, intensity,  exp(-optical_thickness), fs, RT_flags.allowReflect );

}
__syncthreads();
#endif
/*===========TESTING==========`*/
    //__________________________________
    //  Reflections
    if ( (intensity > RT_flags.threshold) && RT_flags.allowReflect){
      reflect( fs, cur, prevCell, abskg[cur], in_domain, step[dir], sign[dir], ray_direction[dir]);
      ++nReflect;
    }

  }  // threshold while loop.
} // end of updateSumI function

//______________________________________________________________________
//  Multi-level
 template< class T>
 __device__ void updateSumI_MLDevice (  GPUVector& ray_direction,
                                        GPUVector& ray_location,
                                        const GPUIntVector& origin,
                                        gridParams gridP,
                                        const GPUIntVector& fineLevel_ROI_Lo,
                                        const GPUIntVector& fineLevel_ROI_Hi,
                                        const int3* regionLo,
                                        const int3* regionHi,
                                        const GPUGridVariable< T >* sigmaT4OverPi,
                                        const GPUGridVariable< T >* abskg,
                                        const GPUGridVariable<int>* cellType,
                                        double& sumI,
                                        hiprandState* randNumStates,
                                        RMCRT_flags RT_flags )
{
  /*`==========TESTING==========*/
#if DEBUG == 1
  if( isDbgCellDevice(origin) ) {
    printf("        A) updateSumI_ML: [%d,%d,%d] ray_dir [%g,%g,%g] ray_loc [%g,%g,%g]\n", origin.x, origin.y, origin.z,ray_direction.x, ray_direction.y, ray_direction.z, ray_location.x, ray_location.y, ray_location.z);
  }
#endif
  /*===========TESTING==========`*/
  int maxLevels = gridP.maxLevels;   // for readability
  int L = maxLevels - 1;       // finest level
  int prevLev = L;

  GPUIntVector cur = origin;
  GPUIntVector prevCell = cur;
  // Step and sign for ray marching
  int step[3];                                          // Gives +1 or -1 based on sign
  bool sign[3];

  GPUVector inv_ray_direction = 1.0 / ray_direction;
  findStepSizeDevice(step, sign, inv_ray_direction);

  //__________________________________
  // define tMax & tDelta on all levels
  // go from finest to coarset level so you can compare
  // with 1L rayTrace results.
  GPUVector tMax;         // (mixing bools, ints and doubles)
  tMax.x = (origin.x + sign[0] - ray_location.x) * inv_ray_direction.x;
  tMax.y = (origin.y + sign[1] * d_levels[L].DyDx - ray_location.y) * inv_ray_direction.y;
  tMax.z = (origin.z + sign[2] * d_levels[L].DzDx - ray_location.z) * inv_ray_direction.z;

  GPUVector tDelta[d_MAXLEVELS];
  for (int Lev = maxLevels - 1; Lev > -1; Lev--) {
    //Length of t to traverse one cell
    tDelta[Lev].x = fabs(inv_ray_direction[0]);
    tDelta[Lev].y = fabs(inv_ray_direction[1]) * d_levels[Lev].DyDx;
    tDelta[Lev].z = fabs(inv_ray_direction[2]) * d_levels[Lev].DzDx;
  }

  //Initializes the following values for each ray
  bool in_domain = true;
  double tMax_prev = 0;
  double intensity = 1.0;
  double fs = 1.0;
  int nReflect = 0;                 // Number of reflections
  double optical_thickness = 0;
  double expOpticalThick_prev = 1.0;
  bool onFineLevel = true;

  //______________________________________________________________________
  //  Threshold  loop

  while (intensity > RT_flags.threshold) {

    DIR dir = NONE;

    while (in_domain) {

      prevCell = cur;
      prevLev = L;
      double disMin = -9;          // Represents ray segment length.

      //__________________________________
      //  Determine which cell the ray will enter next
      if (tMax.x < tMax.y) {        // X < Y
        if (tMax.x < tMax.z) {      // X < Z
          dir = X;
        }
        else {
          dir = Z;
        }
      }
      else {
        if (tMax.y < tMax.z) {       // Y < Z
          dir = Y;
        }
        else {
          dir = Z;
        }
      }

      // next cell index and position
      cur[dir] = cur[dir] + step[dir];
      GPUVector dx_prev = d_levels[L].Dx;           //  Used to compute coarsenRatio
      //__________________________________
      // Logic for moving between levels
      //  - Currently you can only move from fine to coarse level
      //  - Don't jump levels if ray is at edge of domain

      GPUPoint pos = d_levels[L].getCellPosition(cur);         // position could be outside of domain
      in_domain = gridP.domain_BB.inside(pos);

      //in_domain = (cellType[L][cur] == d_flowCell);    // use this when direct comparison with 1L resullts

      bool ray_outside_ROI    = ( containsCellDevice(fineLevel_ROI_Lo, fineLevel_ROI_Hi, cur, dir) == false );
      bool ray_outside_Region = ( containsCellDevice(regionLo[L], regionHi[L], cur, dir) == false );

      bool jumpFinetoCoarserLevel   = ( onFineLevel &&  ray_outside_ROI && in_domain );
      bool jumpCoarsetoCoarserLevel = ( (onFineLevel == false) && ray_outside_Region && (L > 0) && in_domain );

#if (DEBUG == 1 || DEBUG == 4)
      if( isDbgCellDevice(origin) ) {
        printf( "        Ray: [%i,%i,%i] **jumpFinetoCoarserLevel %i jumpCoarsetoCoarserLevel %i containsCell: %i ", cur.x, cur.y, cur.z, jumpFinetoCoarserLevel, jumpCoarsetoCoarserLevel,
            containsCellDevice(fineLevel_ROI_Lo, fineLevel_ROI_Hi, cur, dir));
        printf( " onFineLevel: %i ray_outside_ROI: %i ray_outside_Region: %i in_domain: %i\n", onFineLevel, ray_outside_ROI, ray_outside_Region,in_domain );
        printf( " L: %i regionLo: [%i,%i,%i], regionHi: [%i,%i,%i]\n",L,regionLo[L].x,regionLo[L].y,regionLo[L].z, regionHi[L].x,regionHi[L].y,regionHi[L].z);
      }
#endif

      if (jumpFinetoCoarserLevel) {
        cur = d_levels[L].mapCellToCoarser(cur);
        L = d_levels[L].getCoarserLevelIndex();      // move to a coarser level
        onFineLevel = false;

#if (DEBUG == 1 || DEBUG == 4)
        if( isDbgCellDevice(origin) ) {
          printf( "        ** Jumping off fine patch switching Levels:  prev L: %i, L: %i, cur: [%i,%i,%i] \n",prevLev, L, cur.x, cur.y, cur.z);
        }
#endif

      }
      else if (jumpCoarsetoCoarserLevel) {
        GPUIntVector c_old = cur;                     // needed for debugging
        cur = d_levels[L].mapCellToCoarser(cur);
        L = d_levels[L].getCoarserLevelIndex();      // move to a coarser level
#if (DEBUG == 1 || DEBUG == 4)
        if( isDbgCellDevice(origin) ) {
          printf( "        ** Switching Levels:  prev L: %i, L: %i, cur: [%i,%i,%i], c_old: [%i,%i,%i]\n",prevLev, L, cur.x, cur.y, cur.z, c_old.x, c_old.y, c_old.z);
        }
#endif
      }


      //__________________________________
      //  update marching variables
      disMin = (tMax[dir] - tMax_prev);
      tMax_prev = tMax[dir];
      tMax[dir] = tMax[dir] + tDelta[L][dir];

      ray_location.x = ray_location.x + (disMin * ray_direction.x);
      ray_location.y = ray_location.y + (disMin * ray_direction.y);
      ray_location.z = ray_location.z + (disMin * ray_direction.z);

      //__________________________________
      // Account for uniqueness of first step after reaching a new level
      GPUVector dx = d_levels[L].Dx;
      GPUIntVector coarsenRatio = GPUIntVector(make_int3(1, 1, 1));

      coarsenRatio[0] = dx[0] / dx_prev[0];
      coarsenRatio[1] = dx[1] / dx_prev[1];
      coarsenRatio[2] = dx[2] / dx_prev[2];

      GPUVector lineup;
      for (int ii = 0; ii < 3; ii++) {
        if (sign[ii]) {
          lineup[ii] = -(cur[ii] % coarsenRatio[ii] - (coarsenRatio[ii] - 1));
        }
        else {
          lineup[ii] = cur[ii] % coarsenRatio[ii];
        }
      }

      tMax += lineup * tDelta[prevLev];

      /*`==========TESTING==========*/
#if DEBUG == 1
      if( isDbgCellDevice(origin) ) {
        printf( "        B) cur [%i,%i,%i] prev [%i,%i,%i]"
            " dir %i "
            " stepSize [%i,%i,%i] "
            " tMax [%g,%g,%g] "
            "rayLoc [%g,%g,%g] "
            "inv_dir [%g,%g,%g] "
            "disMin %g "
            "inDomain %i\n"
            "            abskg[prev] %g  \t sigmaT4OverPi[prev]: %g \n"
            "            abskg[cur]  %g  \t sigmaT4OverPi[cur]:  %g  \t  cellType: %i \n"
            "            Dx[prevLev].x  %g \n",
            cur.x, cur.y, cur.z, prevCell.x, prevCell.y, prevCell.z,
            dir,
            step[0],step[1],step[2],
            tMax.x,tMax.y, tMax.z,
            ray_location.x,ray_location.y, ray_location.z,
            inv_ray_direction.x,inv_ray_direction.y, inv_ray_direction.z,
            disMin,
            in_domain,
            abskg[prevLev][prevCell], sigmaT4OverPi[prevLev][prevCell],
            abskg[L][cur], sigmaT4OverPi[L][cur], cellType[L][cur],
            d_levels[prevLev].Dx.x);
      }
#endif
      /*===========TESTING==========`*/
      optical_thickness += d_levels[prevLev].Dx.x * abskg[prevLev][prevCell] * disMin;

      double expOpticalThick = exp(-optical_thickness);

      sumI += sigmaT4OverPi[prevLev][prevCell] * (expOpticalThick_prev - expOpticalThick) * fs;

      expOpticalThick_prev = expOpticalThick;

    }  //end domain while loop.  ++++++++++++++
    //__________________________________
    //
    double wallEmissivity = abskg[L][cur];

    if (wallEmissivity > 1.0) {       // Ensure wall emissivity doesn't exceed one.
      wallEmissivity = 1.0;
    }

    intensity = exp(-optical_thickness);

    sumI += wallEmissivity * sigmaT4OverPi[L][cur] * intensity;

    intensity = intensity * fs;

    // when a ray reaches the end of the domain, we force it to terminate.
    if (!RT_flags.allowReflect)
      intensity = 0;

    /*`==========TESTING==========*/
#if DEBUG == 1
    if( isDbgCellDevice(origin) ) {
      printf( "        C) intensity: %g OptThick: %g, fs: %g allowReflect: %i\n", intensity, optical_thickness, fs, RT_flags.allowReflect );
    }
#endif
    /*===========TESTING==========`*/
    //__________________________________
    //  Reflections
    if ((intensity > RT_flags.threshold) && RT_flags.allowReflect) {
      reflect(fs, cur, prevCell, abskg[L][cur], in_domain, step[dir], sign[dir], ray_direction[dir]);
      ++nReflect;
    }
  }  // threshold while loop.
}  // end of updateSumI function

//______________________________________________________________________
// Returns random number between 0 & 1.0 including 0 & 1.0
// See src/Core/Math/MersenneTwister.h for equation
//______________________________________________________________________
__device__ double randDblDevice(hiprandState* globalState)
{
  int tid = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
  hiprandState localState = globalState[tid];
  double val = hiprand(&localState);
  globalState[tid] = localState;

#ifdef FIXED_RANDOM_NUM
  return 0.3;
#else
  return (double)val * (1.0/4294967295.0);
#endif

}

//______________________________________________________________________
// Returns random number between 0 & 1.0 excluding 0 & 1.0
// See src/Core/Math/MersenneTwister.h for equation
//______________________________________________________________________
__device__ double randDblExcDevice(hiprandState* globalState)
{
  int tid = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;

  hiprandState localState = globalState[tid];
  double val = hiprand(&localState);
  globalState[tid] = localState;

#ifdef FIXED_RANDOM_NUM
  return 0.3;
#else
  return ( (double)val + 0.5 ) * (1.0/4294967296.0);
#endif
}

//______________________________________________________________________
// Returns random integer in [0,n]
// rnd_integer_from_A_to_B = A + hiprand() * (B-A);
//  A = 0
//______________________________________________________________________
__device__ int randIntDevice(hiprandState* globalState,
                             const int B )
{
  double val = randDblDevice( globalState );
  return val * B;
}

//______________________________________________________________________
//  Each thread gets same seed, a different sequence number, no offset
//  This will create repeatable results.
__global__ void setupRandNumKernel(hiprandState* randNumStates)
{
  int tID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
  hiprand_init(1234, tID, 0, &randNumStates[tID]);
}

//______________________________________________________________________
//  is cell a debug cell
__device__ bool isDbgCellDevice( GPUIntVector me )
{
  int size = 2;
  GPUIntVector dbgCell[2];
  dbgCell[0] = make_int3(0,0,0);
  dbgCell[1] = make_int3(5,5,5);



  for (int i = 0; i < size; i++) {
    if( me == dbgCell[i]){
      return true;
    }
  }
  return false;
}
//______________________________________________________________________
//   Perform some sanity checks on the Variable.  This is for debugging
template< class T>
__device__ void GPUVariableSanityCK(const GPUGridVariable<T>& Q,
                                    const GPUIntVector Lo,
                                    const GPUIntVector Hi)
{
#if SCI_ASSERTION_LEVEL > 0
  if (isThread0()) {
    GPUIntVector varLo = Q.getLowIndex();
    GPUIntVector varHi = Q.getHighIndex();

    if( Lo < varLo || varHi < Hi){
      printf ( "ERROR: GPUVariableSanityCK \n");
      printf("  Variable:          varLo:[%i,%i,%i], varHi[%i,%i,%i]\n", varLo.x, varLo.y, varLo.z, varHi.x, varHi.y, varHi.z);
      printf("  Requested extents: varLo:[%i,%i,%i], varHi[%i,%i,%i]\n", Lo.x, Lo.y, Lo.z, Hi.x, Hi.y, Hi.z);
      printf(" Now existing...");
      __threadfence();
      asm("trap;");
    }

    for (int i = Lo.x; i < Hi.x; i++) {
      for (int j = Lo.y; j < Hi.y; j++) {
        for (int k = Lo.z; k < Hi.z; k++) {
          GPUIntVector idx = make_int3(i, j, k);
          T me = Q[idx];
          if ( isnan(me) || isinf(me)){
            printf ( "isNan or isInf was detected at [%i,%i,%i]\n", i,j,k);
            printf(" Now existing...");
            __threadfence();
            asm("trap;");
          }

        }  // k loop
      }  // j loop
    }  // i loop
  }  // thread0
#endif
}
template
__device__ void GPUVariableSanityCK(const GPUGridVariable<float>& Q,
                                    const GPUIntVector Lo,
                                    const GPUIntVector Hi);
template
__device__ void GPUVariableSanityCK(const GPUGridVariable<double>& Q,
                                    const GPUIntVector Lo,
                                    const GPUIntVector Hi);
//______________________________________________________________________
//
template< class T>
__host__ void launchRayTraceKernel(dim3 dimGrid,
                                   dim3 dimBlock,
                                   const int matlIndx,
                                   levelParams level,
                                   patchParams patch,
                                   hipStream_t* stream,
                                   RMCRT_flags RT_flags,
                                   varLabelNames* labelNames,
                                   GPUDataWarehouse* abskg_gdw,
                                   GPUDataWarehouse* sigmaT4_gdw,
                                   GPUDataWarehouse* cellType_gdw,
                                   GPUDataWarehouse* old_gdw,
                                   GPUDataWarehouse* new_gdw)
{
  // setup random number generator states on the device, 1 for each thread
  hiprandState* randNumStates;
  int numStates = dimGrid.x * dimGrid.y * dimBlock.x * dimBlock.y * dimBlock.z;
  randNumStates = (hiprandState*)GPUMemoryPool::allocateCudaSpaceFromPool(0, numStates * sizeof(hiprandState));
  //CUDA_RT_SAFE_CALL( hipMalloc((void**)&randNumStates, numStates * sizeof(hiprandState)) );


  setupRandNumKernel<<< dimGrid, dimBlock, 0, *stream>>>( randNumStates );

  rayTraceKernel< T ><<< dimGrid, dimBlock, 0, *stream >>>( dimGrid,
                                                            dimBlock,
                                                            matlIndx,
                                                            level,
                                                            patch,
                                                            randNumStates,
                                                            RT_flags,
                                                            labelNames,
                                                            abskg_gdw,
                                                            sigmaT4_gdw,
                                                            cellType_gdw,
                                                            old_gdw,
                                                            new_gdw);
    // free device-side RNG states
    GPUMemoryPool::freeCudaSpaceFromPool(0,numStates * sizeof(hiprandState), (void*)randNumStates);
    //CUDA_RT_SAFE_CALL( hipFree(randNumStates) );
}

//______________________________________________________________________
//
template< class T>
__host__ void launchRayTraceDataOnionKernel( dim3 dimGrid,
                                             dim3 dimBlock,
                                             int matlIndex,
                                             patchParams patch,
                                             gridParams gridP,
                                             levelParams* levelP,
                                             GPUIntVector fineLevel_ROI_Lo,
                                             GPUIntVector fineLevel_ROI_Hi,
                                             hipStream_t* stream,
                                             RMCRT_flags RT_flags,
                                             GPUDataWarehouse* abskg_gdw,
                                             GPUDataWarehouse* sigmaT4_gdw,
                                             GPUDataWarehouse* cellType_gdw,
                                             GPUDataWarehouse* old_gdw,
                                             GPUDataWarehouse* new_gdw )
{
  // copy regionLo & regionHi to device memory
  int maxLevels = gridP.maxLevels;

  int3* dev_regionLo;
  int3* dev_regionHi;
  size_t size = d_MAXLEVELS *  sizeof(int3);
  dev_regionLo = (int3*)GPUMemoryPool::allocateCudaSpaceFromPool(0, size);
  dev_regionHi = (int3*)GPUMemoryPool::allocateCudaSpaceFromPool(0, size);

  //CUDA_RT_SAFE_CALL( hipMalloc( (void**)& dev_regionLo, size) );
  //CUDA_RT_SAFE_CALL( hipMalloc( (void**)& dev_regionHi, size) );

  int3 myLo[d_MAXLEVELS];
  int3 myHi[d_MAXLEVELS];
  for (int l = 0; l < maxLevels; ++l) {
    myLo[l] = levelP[l].regionLo;        // never use levelP regionLo or hi in the kernel.
    myHi[l] = levelP[l].regionHi;        // They are different on each patch
  }

  CUDA_RT_SAFE_CALL( hipMemcpyAsync( dev_regionLo, myLo, size, hipMemcpyHostToDevice, *stream) );
  CUDA_RT_SAFE_CALL( hipMemcpyAsync( dev_regionHi, myHi, size, hipMemcpyHostToDevice, *stream) );


  //__________________________________
  // copy levelParams array to constant memory on device
  CUDA_RT_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_levels), levelP, (maxLevels * sizeof(levelParams)),0, hipMemcpyHostToDevice,*stream));
  //CUDA_RT_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_levels), levelP, (maxLevels * sizeof(levelParams))));

  //__________________________________
  // setup random number generator states on the device, 1 for each thread
  hiprandState* randNumStates;
  int numStates = dimGrid.x * dimGrid.y * dimBlock.x * dimBlock.y * dimBlock.z;
  randNumStates = (hiprandState*)GPUMemoryPool::allocateCudaSpaceFromPool(0, numStates * sizeof(hiprandState));
  //CUDA_RT_SAFE_CALL( hipMalloc((void**)&randNumStates, (numStates * sizeof(hiprandState))) );

  setupRandNumKernel<<< dimGrid, dimBlock, 0, *stream>>>( randNumStates );

  rayTraceDataOnionKernel< T ><<< dimGrid, dimBlock, 0, *stream >>>( dimGrid,
                                                                     dimBlock,
                                                                     matlIndex,
                                                                     patch,
                                                                     gridP,
                                                                     fineLevel_ROI_Lo,
                                                                     fineLevel_ROI_Hi,
                                                                     dev_regionLo,
                                                                     dev_regionHi,
                                                                     randNumStates,
                                                                     RT_flags,
                                                                     abskg_gdw,
                                                                     sigmaT4_gdw,
                                                                     cellType_gdw,
                                                                     old_gdw,
                                                                     new_gdw);

  // free device-side RNG states
  //CUDA_RT_SAFE_CALL( hipFree(randNumStates) );
  //CUDA_RT_SAFE_CALL( hipFree(dev_regionLo) );
  //CUDA_RT_SAFE_CALL( hipFree(dev_regionHi) );
  GPUMemoryPool::freeCudaSpaceFromPool(0,numStates * sizeof(hiprandState), (void*)randNumStates);
  GPUMemoryPool::freeCudaSpaceFromPool(0, size, (int3*)dev_regionLo);
  GPUMemoryPool::freeCudaSpaceFromPool(0, size, (int3*)dev_regionHi);


}

//______________________________________________________________________
//  Explicit template instantiations

template
__host__ void launchRayTraceKernel<double>( dim3 dimGrid,
                                            dim3 dimBlock,
                                            const int matlIndx,
                                            levelParams level,
                                            patchParams patch,
                                            hipStream_t* stream,
                                            RMCRT_flags RT_flags,
                                            varLabelNames* labelNames,
                                            GPUDataWarehouse* abskg_gdw,
                                            GPUDataWarehouse* sigmaT4_gdw,
                                            GPUDataWarehouse* cellType_gdw,
                                            GPUDataWarehouse* old_gdw,
                                            GPUDataWarehouse* new_gdw );

//______________________________________________________________________
//
template
__host__ void launchRayTraceKernel<float>( dim3 dimGrid,
                                           dim3 dimBlock,
                                           const int matlIndx,
                                           levelParams level,
                                           patchParams patch,
                                           hipStream_t* stream,
                                           RMCRT_flags RT_flags,
                                           varLabelNames* labelNames,
                                           GPUDataWarehouse* abskg_gdw,
                                           GPUDataWarehouse* sigmaT4_gdw,
                                           GPUDataWarehouse* celltype_gdw,
                                           GPUDataWarehouse* old_gdw,
                                           GPUDataWarehouse* new_gdw );

//______________________________________________________________________
//
template
__host__ void launchRayTraceDataOnionKernel<double>( dim3 dimGrid,
                                                     dim3 dimBlock,
                                                     int matlIndex,
                                                     patchParams patch,
                                                     gridParams gridP,
                                                     levelParams*  levelP,
                                                     GPUIntVector fineLevel_ROI_Lo,
                                                     GPUIntVector fineLevel_ROI_Hi,
                                                     hipStream_t* stream,
                                                     RMCRT_flags RT_flags,
                                                     GPUDataWarehouse* abskg_gdw,
                                                     GPUDataWarehouse* sigmaT4_gdw,
                                                     GPUDataWarehouse* cellType_gdw,
                                                     GPUDataWarehouse* old_gdw,
                                                     GPUDataWarehouse* new_gdw );

//______________________________________________________________________
//
template
__host__ void launchRayTraceDataOnionKernel<float>( dim3 dimGrid,
                                                    dim3 dimBlock,
                                                    int matlIndex,
                                                    patchParams patch,
                                                    gridParams gridP,
                                                    levelParams* levelP,
                                                    GPUIntVector fineLevel_ROI_Lo,
                                                    GPUIntVector fineLevel_ROI_Hi,
                                                    hipStream_t* stream,
                                                    RMCRT_flags RT_flags,
                                                    GPUDataWarehouse* abskg_gdw,
                                                    GPUDataWarehouse* sigmaT4_gdw,
                                                    GPUDataWarehouse* cellType_gdw,
                                                    GPUDataWarehouse* old_gdw,
                                                    GPUDataWarehouse* new_gdw );

} //end namespace Uintah
