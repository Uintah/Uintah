#include "hip/hip_runtime.h"
/*
 * The MIT License
 *
 * Copyright (c) 1997-2017 The University of Utah
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to
 * deal in the Software without restriction, including without limitation the
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 */


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <random>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <MersenneTwister.h>

#define BLKWIDTH 32


//______________________________________________________________________
//
//
//  The following compares the random number generation on the CPU vs GPU
//
//
//______________________________________________________________________

//______________________________________________________________________
//
inline int RoundUp(double d)
{
  if(d>=0){
    if((d-(int)d) == 0){
      return (int)d;
    } else{
      return (int)(d+1);
    }
  } else {
    return (int)d;
  }
}
//______________________________________________________________________
//
void stopwatch( std::string message, time_t start)
{    
  double secs;
  time_t stop;                 /* timing variables             */

  stop = time(nullptr);
  secs = difftime(stop, start);               
  fprintf(stdout,"    %.f [s] %s  \n",secs, message.c_str());       
}
//______________________________________________________________________
//  CPU based random number generations
void randCPU( double *M, int nRandNums)
{
  unsigned int size = nRandNums;
  unsigned int Imem_size = sizeof(unsigned int) * size;
  unsigned int Dmem_size = sizeof(double) * size;
  
  int* org_randInt = (int*)malloc(Imem_size);
  int* new_randInt = (int*)malloc(Imem_size);
 
  double* org_randDbl = (double*)malloc(Dmem_size);
  double* new_randDbl = (double*)malloc(Dmem_size); 
  
  //__________________________________
  //  Orginal implementation
  MTRand mTwister;
  for (int i = 0; i< nRandNums; i++){
    mTwister.seed(i);
    org_randDbl[i] = mTwister.rand();
    org_randInt[i] = mTwister.randInt();
  }

  //__________________________________
  //  C++11 
  std::mt19937 mTwist;
  std::uniform_real_distribution<double> D_dist(0.0,1.0);
  std::uniform_int_distribution<int>     I_dist;  // 
  mTwist.seed(1234ULL);
  
  
  for (int i = 0; i< nRandNums; i++){  
    new_randDbl[i] = D_dist( mTwist );
    new_randInt[i] = I_dist( mTwist );
  }


  for (int i = 0; i< nRandNums; i++){
    M[i] = new_randDbl[i];
  }


  for (int i = 0; i< nRandNums; i++){
    printf( "%i org_randDbl: %g  new_randDbl: %g org_randInt: %i, new_randInt: %i\n",i, org_randDbl[i],  new_randDbl[i], org_randInt[i], new_randInt[i]);
  }
  
  free( org_randInt );
  free( new_randInt );
  free( org_randDbl );
  free( new_randDbl );
}


//______________________________________________________________________
//  Determine device properties
void deviceProperties( int &maxThreadsPerBlock )
{  
  // Number of CUDA devices
  int devCount;
  hipGetDeviceCount(&devCount);

  // Iterate through devices
  for (int deviceNum = 0; deviceNum < devCount; ++deviceNum){
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceNum);
   // printDevProp(deviceProp);

    maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
  }
}

//______________________________________________________________________
//  This is the host side random number generation using cuda
void randGPU_V1( double *M, int nRandNums)
{
  int size = nRandNums* sizeof(double);
  double* Md;

  //__________________________________
  //  allocate device memory and copy memory to the device
  hipMalloc( (void**)&Md, size);  
  
  hipMemcpy( Md, M, size, hipMemcpyHostToDevice );
  
  //__________________________________
  // Create pseudo-random number generator
  // set the seed 
  // generate the numbers
  hiprandGenerator_t randGen;
  
//  hiprandCreateGenerator(&randGen, HIPRAND_RNG_PSEUDO_DEFAULT);

  hiprandCreateGenerator(&randGen, HIPRAND_RNG_PSEUDO_MT19937);

  hiprandSetPseudoRandomGeneratorSeed(randGen, 1234ULL);

  hiprandGenerateUniformDouble(randGen, Md, nRandNums);
 
  //__________________________________
  //   copy from device memory and free device matrices
  hipMemcpy( M, Md, size, hipMemcpyDeviceToHost );
  hipFree( Md );
  hiprandDestroyGenerator(randGen);
}

//______________________________________________________________________
//    Returns an random number
__device__ double randDevice(hiprandState* globalState, const int tid)
{
  hiprandState localState = globalState[tid];
  double val = hiprand(&localState);
  globalState[tid] = localState;
  return (double)val * (1.0/4294967295.0);
}


//______________________________________________________________________
//    Returns an random number  excluding 0 & 1.0.  See MersenneTwister.h
//
__device__ double randDblExcDevice(hiprandState* globalState, const int tid)
{
  hiprandState localState = globalState[tid];
  double val = hiprand(&localState);
  globalState[tid] = localState;
  return ( double(val) + 0.5 ) * (1.0/4294967296.0);
}

//______________________________________________________________________
//
__global__ void setup_kernel(hiprandState* randNumStates)
{
   int tID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
   /* Each thread gets same seed, a different sequence number, no offset */
   hiprand_init(1234, tID, 0, &randNumStates[tID]);
}
//______________________________________________________________________
//    Kernel:  
__global__ void randNumKernel( hiprandState* randNumStates, double* M, double* N, int nRandNums )
{
  int tID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
//  for (int k = 0; k < nRandNums; ++k){
    M[tID] = randDblExcDevice( randNumStates, tID);
    N[tID] = randDevice( randNumStates, tID );
//  }
}

//______________________________________________________________________
//  Device side random number generator
void randGPU_V2( double *M, double *N,int nRandNums)
{
  int size = nRandNums* sizeof(double);
  double* Md;
  double* Nd;
  //__________________________________
  //  allocate device memory and copy memory to the device
  hipMalloc( (void**)&Md, size);  
  hipMalloc( (void**)&Nd, size);
  //__________________________________
  //  copy host memory -> device
  hipMemcpy( Md, M, size, hipMemcpyHostToDevice );
  hipMemcpy( Nd, N, size, hipMemcpyHostToDevice );  
  //__________________________________
  //
  int maxThreadsPerBlock = 0;
  deviceProperties( maxThreadsPerBlock );
  
  int xMaxThreadsPerBlock = BLKWIDTH;
  int yMaxThreadsPerBlock = BLKWIDTH;
  maxThreadsPerBlock = xMaxThreadsPerBlock * yMaxThreadsPerBlock;       // hardwired for now
  
  
  int threadsPerBlock = min(maxThreadsPerBlock, nRandNums);
  
  int xBlocks = 0;
  int yBlocks = 0;
  
  if( nRandNums > maxThreadsPerBlock){
    int nBlocks = RoundUp(  nRandNums/sqrt(maxThreadsPerBlock) );
    xBlocks = RoundUp(  nRandNums/xMaxThreadsPerBlock );
    yBlocks = RoundUp(  nRandNums/yMaxThreadsPerBlock );
  }else{
    xBlocks = 1;   // if matrix is smaller than 1 block
    yBlocks = 1;
  }
  
  int nBlocks = xBlocks = yBlocks;           // Assumption that
  int me = xBlocks * yBlocks * threadsPerBlock;
  
  fprintf(stdout, "    xBlocks: %d, yBlocks: %d, nRandNums: %d BLKWIDTH: %d, threadsPerBlock %d ",xBlocks, yBlocks, nRandNums, BLKWIDTH, threadsPerBlock);
  fprintf(stdout, "    number of threads: %d\n",me);
  
  //__________________________________
  //  Kernel invocation
  dim3 dimBlock(BLKWIDTH, BLKWIDTH, 1);
  dim3 dimGrid( xBlocks,  yBlocks,  1);
  
  // setup random number generator states on the device, 1 for each thread
  hiprandState* randNumStates;
  int numStates = dimGrid.x * dimGrid.y * dimBlock.x * dimBlock.y * dimBlock.z;
  hipMalloc((void**)&randNumStates, numStates * sizeof(hiprandState));

  //__________________________________
  //  Global Memory Kernel
  time_t start = time(nullptr);
  setup_kernel<<<dimGrid, dimBlock>>>( randNumStates );
  stopwatch("  randDeviceGPU setup_kernel: ", start);
  
  start = time(nullptr);
  randNumKernel<<<dimGrid, dimBlock>>>( randNumStates, Md, Nd, nRandNums );
  stopwatch("  randDeviceGPU randNumKernel: ", start);
  
  //__________________________________
  //   copy from device memory and free memory
  start = time(nullptr);
  hipMemcpy( M, Md, size, hipMemcpyDeviceToHost );
  hipMemcpy( N, Nd, size, hipMemcpyDeviceToHost );
  stopwatch(" randDeviceGPU memcopy: ", start);
  
  start = time(nullptr);
  hipFree( Md );
  hipFree( Nd );
  hipFree(randNumStates) ;
  stopwatch("  randDeviceGPU free memory: ", start);
}


//______________________________________________________________________
int main( int argc, char** argv)
{  

  FILE *fp;
  fp = fopen("randomNumbers.dat", "w");
  for(int power = 0; power<2; ++power) { 
    //int nRandNums = pow(10,power);
    int nRandNums = 8;   
    fprintf(stdout,"__________________________________\n");
    fprintf(stdout," nRand %d  \n", nRandNums);
    
    //__________________________________
    //  allocate memory
    unsigned int size = nRandNums;
    unsigned int mem_size = sizeof(double) * size;
    double* rand_CPU       = (double*)malloc(mem_size); 
    double* rand_GPU_L   = (double*)malloc(mem_size);
    double* rand_GPU_M  = (double*)malloc(mem_size);
    double* rand_GPU_N  = (double*)malloc(mem_size); 
       
    time_t start;
    start = time(nullptr);
    //__________________________________
    //  Compute the random numbers
    randCPU( rand_CPU, nRandNums );
    stopwatch(" randCPU: ", start);
    
    start = time(nullptr);
    randGPU_V1( rand_GPU_L, nRandNums);
    stopwatch(" randGPU_V1: ", start);
     
    start = time(nullptr);    
    randGPU_V2( rand_GPU_M, rand_GPU_N, nRandNums);
    stopwatch(" randGPU_V2: ", start);
    
    //__________________________________
    //  Output data

    fprintf( fp, "           #CPU,                 GPU_V1,               GPU_dblExc,            GPU_dblInc\n");
    for (int i = 0; i< nRandNums; i++){
      fprintf( fp, "%i:%i, %16.15E, %16.15E, %16.15E,  %16.15E\n",power,i, rand_CPU[i], rand_GPU_L[i], rand_GPU_M[i], rand_GPU_N[i] );
      //printf(      "%i, %16.15E, %16.15E, %16.15E,  %16.15E\n",i, rand_CPU[i], rand_GPU_L[i], rand_GPU_M[i], rand_GPU_N[i] );
    }

    
    //__________________________________
    //Free memory
    free( rand_CPU );
    free( rand_GPU_L );
    free( rand_GPU_M );
    free( rand_GPU_N );
  }   // loop 
  fclose(fp);
}




