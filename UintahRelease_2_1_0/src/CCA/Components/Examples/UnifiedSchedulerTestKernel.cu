#include "hip/hip_runtime.h"
/*
 * The MIT License
 *
 * Copyright (c) 1997-2017 The University of Utah
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to
 * deal in the Software without restriction, including without limitation the
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 */

#include <sci_defs/cuda_defs.h>

#include <Core/Grid/Variables/GPUGridVariable.h>
#include <Core/Parallel/Parallel.h>
#include <Core/Util/GPU.h>

#include <CCA/Components/Schedulers/GPUDataWarehouse.h>

#include <string>
using namespace std;

namespace Uintah {

//______________________________________________________________________
//
// @brief A GPU kernel for the Jacobi iterations in the Poisson 1-material solver
// @param patchID the patch this kernel will operate over
// @param matlIndex the material associated with the specified patchID
// @param domainLow a three component vector that gives the lower corner of the work area as (x,y,z)
// @param domainHigh a three component vector that gives the highest corner of the work area as (x,y,z)
// @param old_gpudw the old GPU DataWarehouse
// @param new_gpudw the new GPU DataWarehouse

__global__
void
unifiedSchedulerTestKernel( int                patchID,
                            uint3              patchNodeLowIndex,
                            uint3              patchNodeHighIndex,
                            uint3              domainLow,
                            uint3              domainHigh,
                            GPUDataWarehouse * old_gpudw,
                            GPUDataWarehouse * new_gpudw,
                            hipStream_t     * stream)
{

  const GPUGridVariable<double> phi;
  GPUGridVariable<double> newphi;
  old_gpudw->get(phi, "phi", patchID, 0, 0);

  new_gpudw->getModifiable(newphi, "phi", patchID, 0);
  //if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
  //  printf("*****For patch %d old phi %p and new phi %p*****\n", patchID, phi.getVoidPointer(), newphi.getVoidPointer());
  //}
  // calculate the thread indices
  int i = blockDim.x * blockIdx.x + threadIdx.x + patchNodeLowIndex.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y + patchNodeLowIndex.y;


  // If the threads are within the bounds of the patch
  // the algorithm is allowed to stream along the z direction
  // applying the stencil to a line of cells.  The z direction
  // is streamed because it allows access of x and y elements
  // that are close to one another which should allow coalesced
  // memory accesses.


  //Copy all face cells (any on an exposed face.)

  // These outer cells don't get computed, just preserved across iterations
  // newphi(i,j,k) = phi(i,j,k)
  if(i >= patchNodeLowIndex.x && j >= patchNodeLowIndex.y && i < patchNodeHighIndex.x && j < patchNodeHighIndex.y ) {
    if ((domainLow.x - patchNodeLowIndex.x == 1 && i == patchNodeLowIndex.x) ||  /*left face*/
        (domainLow.y - patchNodeLowIndex.y == 1 && j == patchNodeLowIndex.y) ||  /*bottom face*/
        (patchNodeHighIndex.x - domainHigh.x == 1 && i == patchNodeHighIndex.x - 1) ||  /*right face*/
        (patchNodeHighIndex.y - domainHigh.y == 1 && j == patchNodeHighIndex.y - 1)) {  /*top face*/

      for (int k = domainLow.z; k < domainHigh.z; k++) {
        newphi(i,j,k) = phi(i,j,k);
        //if (i == 0 && j == 1 && k == 9) {
        //  printf("gpu - border1 - newphi(%d, %d, %d) is %1.6lf at ptr %p\n", i,j,k,newphi(i,j,k), &newphi(i,j,k));
        //}
      }
    }

    if (domainLow.z - patchNodeLowIndex.z == 1){
      newphi(i,j,patchNodeLowIndex.z) = phi(i,j,patchNodeLowIndex.z);
      //printf("gpu - border2 - newphi(%d, %d, %d) is %1.6lf at ptr %p\n", i,j,patchNodeLowIndex.z,newphi(i,j,patchNodeLowIndex.z),&newphi(i,j,patchNodeLowIndex.z));
    }
    if (patchNodeHighIndex.z - domainHigh.z == 1) {
      newphi(i,j,patchNodeHighIndex.z-1) = phi(i,j,patchNodeHighIndex.z-1);
      //printf("gpu - border3 - newphi(%d, %d, %d) is %1.6lf at ptr %p\n", i,j,patchNodeHighIndex.z-1,newphi(i,j,patchNodeHighIndex.z-1),&newphi(i,j,patchNodeHighIndex.z-1));
    }
  }
  __syncthreads();

  if(i >= domainLow.x && j >= domainLow.y && i < domainHigh.x && j < domainHigh.y ) {

    for (int k = domainLow.z; k < domainHigh.z; k++) {

      newphi(i,j,k) = (1. / 6)
                  * (phi(i-1, j, k)
                   + phi(i+1, j, k)
                   + phi(i, j-1, k)
                   + phi(i, j+1, k)
                   + phi(i, j, k-1)
                   + phi(i, j, k+1));
      //if (i == 1 && j == 1 && k == 1) {
      //        printf("gpu - newphi(%d, %d, %d) is %1.6lf ptr %p from %1.6lf %1.6lf %1.6lf %1.6lf %1.6lf %1.6lf addresses %p %p %p %p %p %p\n",
      //            i, j, k, newphi(i, j, k), &newphi(i,j,k),
      //            phi(i-1, j, k), phi(i+1, j, k), phi(i, j-1, k), phi(i, j+1, k), phi(i, j, k-1), phi(i, j, k+1),
      //            &phi(i-1, j, k), &phi(i+1, j, k), &phi(i, j-1, k), &phi(i, j+1, k), &phi(i, j, k-1), &phi(i, j, k+1));
      //}
      //if (i == 1 && j == 1 && k == domainLow.z) {
      //        printf("gpu - newphi(%d, %d, %d) is %1.6lf from %1.6lf %1.6lf %1.6lf %1.6lf %1.6lf %1.6lf\n", i, j, k, newphi(i, j, k), phi(i-1, j, k), phi(i+1, j, k), phi(i, j-1, k), phi(i, j+1, k), phi(i, j, k-1), phi(i, j, k+1));
      //}

    }
  }

  //}
}

void
launchUnifiedSchedulerTestKernel( dim3               dimGrid,
                                  dim3               dimBlock,
                                  hipStream_t     * stream,
                                  int                patchID,
                                  uint3              patchNodeLowIndex,
                                  uint3              patchNodeHighIndex,
                                  uint3              domainLow,
                                  uint3              domainHigh,
                                  GPUDataWarehouse * old_gpudw,
                                  GPUDataWarehouse * new_gpudw)
{
  unifiedSchedulerTestKernel<<< dimGrid, dimBlock, 0, *stream>>>( patchID,
                                                                  patchNodeLowIndex,
                                                                  patchNodeHighIndex,
                                                                  domainLow,
                                                                  domainHigh,
                                                                  old_gpudw,
                                                                  new_gpudw,
                                                                  stream );
  //hipDeviceSynchronize();
}

} //end namespace Uintah
