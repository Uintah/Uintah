#include "hip/hip_runtime.h"
/*
 * The MIT License
 *
 * Copyright (c) 1997-2015 The University of Utah
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to
 * deal in the Software without restriction, including without limitation the
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
 * IN THE SOFTWARE.
 */

#include <CCA/Components/Models/Radiation/RMCRT/RayGPU.cuh>
#include <CCA/Components/Schedulers/GPUDataWarehouse.h>

#include <Core/Grid/Variables/GPUGridVariable.h>
#include <Core/Grid/Variables/GPUStencil7.h>
#include <Core/Grid/Variables/Stencil7.h>
#include <Core/Util/GPU.h>

#include <sci_defs/cuda_defs.h>
#include <sci_defs/uintah_defs.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DEBUG -9                 // 1: divQ, 2: boundFlux, 3: scattering
#define FIXED_RANDOM_NUM         // also edit in src/Core/Math/MersenneTwister.h to compare with Ray:CPU

//__________________________________
//  To Do
//  - Need to implement transferFrom so use can use calc_frequency > 1
//  - Temporal scheduling
//  - restarts are not working.
//  - Investigate using multiple GPUs per node.
//  - Implement fixed and dynamic ROI.
//  - dynamic block size?
//  - Implement labelNames in unified memory.
//  - investigate the performance with different patch configurations
//  - deterministic random numbers
//  - Ray steps


//__________________________________
//
//  To use cuda-gdb on a single GPU you must set the environmental variable
//  CUDA_DEBUGGER_SOFTWARE_PREEMPTION=1
//
// mpirun -np 1 xterm -e cuda-gdb sus -gpu -nthreads 2 <args>
//__________________________________

namespace Uintah {

//---------------------------------------------------------------------------
// Kernel: The GPU ray tracer kernel
//---------------------------------------------------------------------------
template< class T>
__global__ void rayTraceKernel( dim3 dimGrid,
                                dim3 dimBlock,
                                const int matl,
                                const int levelIndx,
                                patchParams patch,
                                hiprandState* randNumStates,
                                RMCRT_flags RT_flags,
                                varLabelNames* labelNames,
                                GPUDataWarehouse* abskg_gdw,
                                GPUDataWarehouse* sigmaT4_gdw,
                                GPUDataWarehouse* cellType_gdw,
                                GPUDataWarehouse* old_gdw,
                                GPUDataWarehouse* new_gdw )
{
    // Not used right now
//  int blockID  = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
//  int threadID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;

  // calculate the thread indices
  int tidX = threadIdx.x + blockIdx.x * blockDim.x + patch.loEC.x;
  int tidY = threadIdx.y + blockIdx.y * blockDim.y + patch.loEC.y;

  const GPUGridVariable< T > sigmaT4OverPi;
  const GPUGridVariable< T > abskg;              // Need to use getRegion() to get the data
  const GPUGridVariable<int> cellType;

  GPUGridVariable<double> divQ;
  GPUGridVariable<GPUStencil7> boundFlux;
  GPUGridVariable<double> radiationVolQ;

//  sigmaT4_gdw->print();
  
  sigmaT4_gdw->getLevel( sigmaT4OverPi, "sigmaT4",  matl, levelIndx);
  cellType_gdw->getLevel( cellType,     "cellType", matl, levelIndx);

  if(RT_flags.usingFloats){
    abskg_gdw->getLevel( abskg, "abskgRMCRT",  matl, levelIndx);
  }else{
    abskg_gdw->getLevel( abskg, "abskg",       matl, levelIndx);
  }

  if( RT_flags.modifies_divQ ){
    new_gdw->getModifiable( divQ,         "divQ",          patch.ID, matl );
    new_gdw->getModifiable( boundFlux,    "boundFlux",     patch.ID, matl );
    new_gdw->getModifiable( radiationVolQ,"radiationVolq", patch.ID, matl );
  }else{
    new_gdw->get( divQ,         "divQ",          patch.ID, matl );         // these should be allocateAntPut() calls
    new_gdw->get( boundFlux,    "boundFlux",     patch.ID, matl );
    new_gdw->get( radiationVolQ,"radiationVolq", patch.ID, matl );


    // Extra Cell Loop
    if ( (tidX >= patch.loEC.x) && (tidY >= patch.loEC.y) && (tidX < patch.hiEC.x) && (tidY < patch.hiEC.y) ) { // patch boundary check
      #pragma unroll
      for (int z = patch.loEC.z; z < patch.hiEC.z; z++) { // loop through z slices
        GPUIntVector c = make_int3(tidX, tidY, z);
        divQ[c]          = 0.0;
        radiationVolQ[c] = 0.0;
      }
    }
  }
  
  //__________________________________  
  //  Sanity checks                     
#if 0
  if (isThread0()) {
   printf("  GPUVariable Sanity check level: %i, patch: %i \n",levelIndx, patch.ID); 
  }
#endif
  GPUVariableSanityCK(abskg,         patch.loEC, patch.hiEC);
  GPUVariableSanityCK(sigmaT4OverPi, patch.loEC, patch.hiEC);


  double DyDx = patch.dx.y/patch.dx.x;
  double DzDx = patch.dx.z/patch.dx.x;

  //______________________________________________________________________
  //           R A D I O M E T E R
  //______________________________________________________________________
  // TO BE FILLED IN

  //______________________________________________________________________
  //          B O U N D A R Y F L U X
  //______________________________________________________________________
  if( RT_flags.solveBoundaryFlux ){

    __shared__ int3 dirIndexOrder[6];
    __shared__ int3 dirSignSwap[6];
    __shared__ int3 locationIndexOrder[6];
    __shared__ int3 locationShift[6];

    //_____________________________________________
    //   Ordering for Surface Method
    // This block of code is used to properly place ray origins, and orient ray directions
    // onto the correct face.  This is necessary, because by default, the rays are placed
    // and oriented onto a default face, then require adjustment onto the proper face.
    dirIndexOrder[EAST]   = make_int3(2, 1, 0);
    dirIndexOrder[WEST]   = make_int3(2, 1, 0);
    dirIndexOrder[NORTH]  = make_int3(0, 2, 1);
    dirIndexOrder[SOUTH]  = make_int3(0, 2, 1);
    dirIndexOrder[TOP]    = make_int3(0, 1, 2);
    dirIndexOrder[BOT]    = make_int3(0, 1, 2);

    // Ordering is slightly different from 6Flux since here, rays pass through origin cell from the inside faces.
    dirSignSwap[EAST]     = make_int3(-1, 1,  1);
    dirSignSwap[WEST]     = make_int3( 1, 1,  1);
    dirSignSwap[NORTH]    = make_int3( 1, -1, 1);
    dirSignSwap[SOUTH]    = make_int3( 1, 1,  1);
    dirSignSwap[TOP]      = make_int3( 1, 1, -1);
    dirSignSwap[BOT]      = make_int3( 1, 1,  1);

    locationIndexOrder[EAST]  = make_int3(1,0,2);
    locationIndexOrder[WEST]  = make_int3(1,0,2);
    locationIndexOrder[NORTH] = make_int3(0,1,2);
    locationIndexOrder[SOUTH] = make_int3(0,1,2);
    locationIndexOrder[TOP]   = make_int3(0,2,1);
    locationIndexOrder[BOT]   = make_int3(0,2,1);

    locationShift[EAST]   = make_int3(1, 0, 0);
    locationShift[WEST]   = make_int3(0, 0, 0);
    locationShift[NORTH]  = make_int3(0, 1, 0);
    locationShift[SOUTH]  = make_int3(0, 0, 0);
    locationShift[TOP]    = make_int3(0, 0, 1);
    locationShift[BOT]    = make_int3(0, 0, 0);

    //__________________________________
    // GPU equivalent of GridIterator loop - calculate sets of rays per thread
    if ( (tidX >= patch.lo.x) && (tidY >= patch.lo.y) && (tidX < patch.hi.x) && (tidY < patch.hi.y) ) { // patch boundary check
      #pragma unroll
      for (int z = patch.lo.z; z < patch.hi.z; z++) { // loop through z slices

        GPUIntVector origin = make_int3(tidX, tidY, z);  // for each thread

        boundFlux[origin].initialize(0.0);

        BoundaryFaces boundaryFaces;

         // which surrounding cells are boundaries
        boundFlux[origin].p = has_a_boundaryDevice(origin, cellType, boundaryFaces);

        //__________________________________
        // Loop over boundary faces of the cell and compute incident radiative flux
        #pragma unroll
        for( int i = 0; i<boundaryFaces.size(); i++) {

          int RayFace = boundaryFaces.faceArray[i];
          int UintahFace[6] = {WEST,EAST,SOUTH,NORTH,BOT,TOP};

          double sumI     = 0;
          double sumProjI = 0;
          double sumI_prev= 0;

          //__________________________________
          // Flux ray loop
          #pragma unroll
          for (int iRay=0; iRay < RT_flags.nFluxRays; iRay++){

            GPUVector direction_vector, ray_location;
            double cosTheta;

            rayDirection_cellFaceDevice( randNumStates, origin, dirIndexOrder[RayFace], dirSignSwap[RayFace], iRay,
                                   direction_vector, cosTheta );

            rayLocation_cellFaceDevice( randNumStates, origin, locationIndexOrder[RayFace], locationShift[RayFace],
                                  DyDx, DzDx, ray_location);

            updateSumIDevice< T >( direction_vector, ray_location, origin, patch.dx, sigmaT4OverPi, abskg, cellType, sumI, randNumStates, RT_flags);

            sumProjI += cosTheta * (sumI - sumI_prev);   // must subtract sumI_prev, since sumI accumulates intensity

            sumI_prev = sumI;

          } // end of flux ray loop

          //__________________________________
          //  Compute Net Flux to the boundary
          int face = UintahFace[RayFace];
          boundFlux[origin][ face ] = sumProjI * 2 *M_PI/RT_flags.nFluxRays;

/*`==========TESTING==========*/
#if DEBUG == 2
          printf( "\n      [%d, %d, %d]  face: %d sumProjI:  %g BF: %g\n",
                    origin.x, origin.y, origin.z, face, sumProjI, boundFlux[origin][ face ]);
#endif
/*===========TESTING==========`*/

        } // boundary faces loop
      }  // z slices loop
    }  // X-Y Thread loop
  }


  //______________________________________________________________________
  //         S O L V E   D I V Q
  //______________________________________________________________________
  if( RT_flags.solveDivQ ){
    // GPU equivalent of GridIterator loop - calculate sets of rays per thread
    if ( (tidX >= patch.lo.x) && (tidY >= patch.lo.y) && (tidX < patch.hi.x) && (tidY < patch.hi.y) ) { // patch boundary check
      #pragma unroll
      for (int z = patch.lo.z; z < patch.hi.z; z++) { // loop through z slices

        GPUIntVector origin = make_int3(tidX, tidY, z);  // for each thread
        double sumI = 0;

        //__________________________________
        // ray loop
        #pragma unroll
        for (int iRay = 0; iRay < RT_flags.nDivQRays; iRay++) {

          GPUVector direction_vector = findRayDirectionDevice( randNumStates );

          GPUVector ray_location = rayLocationDevice( randNumStates, origin, DyDx,  DzDx, RT_flags.CCRays );

          updateSumIDevice< T >( direction_vector, ray_location, origin, patch.dx,  sigmaT4OverPi, abskg, cellType, sumI, randNumStates, RT_flags);
        } //Ray loop

        //__________________________________
        //  Compute divQ
        divQ[origin] = 4.0 * M_PI * abskg[origin] * ( sigmaT4OverPi[origin] - (sumI/RT_flags.nDivQRays) );

        // radiationVolq is the incident energy per cell (W/m^3) and is necessary when particle heat transfer models (i.e. Shaddix) are used
        radiationVolQ[origin] = 4.0 * M_PI * abskg[origin] *  (sumI/RT_flags.nDivQRays) ;

/*`==========TESTING==========*/
#if DEBUG == 1
        if( isDbgCellDevice( origin ) ){
          printf( "\n      [%d, %d, %d]  sumI: %g  divQ: %g radiationVolq: %g  abskg: %g,    sigmaT4: %g \n",
                    origin.x, origin.y, origin.z, sumI,divQ[origin], radiationVolQ[origin],abskg[origin], sigmaT4OverPi[origin]);
        }
#endif
/*===========TESTING==========`*/
      }  // end z-slice loop
    }  // end domain boundary check
  }  // solve divQ
}  // end ray trace kernel

//---------------------------------------------------------------------------
// Kernel: The GPU ray tracer data onion kernel
//---------------------------------------------------------------------------
// hard-wired for 2-levels now, but this should be fast and fixes
__constant__ levelParams d_levels[d_MAXLEVELS];

template< class T>
__global__ void rayTraceDataOnionKernel( dim3 dimGrid,
                                         dim3 dimBlock,
                                         int matl,
                                         patchParams finePatch,
                                         gridParams gridP,
                                         GPUIntVector fineLevel_ROI_Lo,
                                         GPUIntVector fineLevel_ROI_Hi,
                                         int3* regionLo,
                                         int3* regionHi,
                                         hiprandState* randNumStates,
                                         RMCRT_flags RT_flags,
                                         GPUDataWarehouse* abskg_gdw,
                                         GPUDataWarehouse* sigmaT4_gdw,
                                         GPUDataWarehouse* cellType_gdw,
                                         GPUDataWarehouse* old_gdw,
                                         GPUDataWarehouse* new_gdw )
{
    // Not used right now
//  int blockID  = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
//  int threadID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;


  // calculate the thread indices
  int tidX = threadIdx.x + blockIdx.x * blockDim.x + finePatch.loEC.x;
  int tidY = threadIdx.y + blockIdx.y * blockDim.y + finePatch.loEC.y;

#if 0
  if (tidX == 1 && tidY == 1) {
    printf("\nGPU levelParams\n");

    printf("Level-0 ");
    d_levels[0].print();

    printf("Level-1 ");
    d_levels[1].print();
  }
#endif


  int maxLevels = gridP.maxLevels;
  int fineL = maxLevels - 1;

  //__________________________________
  //
  const GPUGridVariable<T>    abskg[d_MAXLEVELS];
  const GPUGridVariable<T>    sigmaT4OverPi[d_MAXLEVELS];
  const GPUGridVariable<int>  cellType[d_MAXLEVELS];

//  new_gdw->print();

  //__________________________________
  // coarse level data for the entire level
  for (int l = 0; l < maxLevels; ++l) {
    if (d_levels[l].hasFinerLevel) {
      abskg_gdw->getLevel( abskg[l],           "abskg",    matl, l);
      sigmaT4_gdw->getLevel( sigmaT4OverPi[l], "sigmaT4",  matl, l);
      cellType_gdw->getLevel( cellType[l],     "cellType", matl, l);

      GPUVariableSanityCK(abskg[l],        d_levels[l].regionLo,d_levels[l].regionHi);
      GPUVariableSanityCK(sigmaT4OverPi[l],d_levels[l].regionLo,d_levels[l].regionHi);
    }
  }

  //__________________________________
  //  fine level data for the region of interest.
  //  ToDo:  replace get with getRegion() calls so 
  //  so the halo can be > 0
  if ( RT_flags.whichROI_algo == patch_based ) {

    abskg_gdw->get(abskg[fineL],           "abskg",    finePatch.ID, matl, fineL);
    sigmaT4_gdw->get(sigmaT4OverPi[fineL], "sigmaT4",  finePatch.ID, matl, fineL);
    cellType_gdw->get(cellType[fineL],     "cellType", finePatch.ID, matl, fineL);

    GPUVariableSanityCK(abskg[fineL],        fineLevel_ROI_Lo,fineLevel_ROI_Hi);
    GPUVariableSanityCK(sigmaT4OverPi[fineL],fineLevel_ROI_Lo,fineLevel_ROI_Hi);
  }

  GPUGridVariable<double> divQ;
  GPUGridVariable<GPUStencil7> boundFlux;
  GPUGridVariable<double> radiationVolQ;

  //__________________________________
  //  fine level data for this patch
  if( RT_flags.modifies_divQ ){
    new_gdw->getModifiable( divQ,         "divQ",          finePatch.ID, matl, fineL );
    new_gdw->getModifiable( boundFlux,    "boundFlux",     finePatch.ID, matl, fineL );
    new_gdw->getModifiable( radiationVolQ,"radiationVolq", finePatch.ID, matl, fineL );
  }else{
    new_gdw->get( divQ,         "divQ",          finePatch.ID, matl, fineL );         // these should be allocateAntPut() calls
    new_gdw->get( boundFlux,    "boundFlux",     finePatch.ID, matl, fineL );
    new_gdw->get( radiationVolQ,"radiationVolq", finePatch.ID, matl, fineL );


    //__________________________________
    // initialize Extra Cell Loop
    if ( (tidX >= finePatch.loEC.x) && (tidY >= finePatch.loEC.y) && (tidX < finePatch.hiEC.x) && (tidY < finePatch.hiEC.y) ) { // finePatch boundary check
      #pragma unroll
      for (int z = finePatch.loEC.z; z < finePatch.hiEC.z; z++) { // loop through z slices
        GPUIntVector c = make_int3(tidX, tidY, z);
        divQ[c]          = 0.0;
        radiationVolQ[c] = 0.0;
      }
    }
  }

  //______________________________________________________________________
  //           R A D I O M E T E R
  //______________________________________________________________________
  // TO BE FILLED IN



  //______________________________________________________________________
  //          B O U N D A R Y F L U X
  //______________________________________________________________________
  if( RT_flags.solveBoundaryFlux ){
    // TO BE FILLED IN
  }


#if 1
  //______________________________________________________________________
  //         S O L V E   D I V Q
  //______________________________________________________________________
  if( RT_flags.solveDivQ ) {

    // GPU equivalent of GridIterator loop - calculate sets of rays per thread
    if ( (tidX >= finePatch.lo.x) && (tidY >= finePatch.lo.y) && (tidX < finePatch.hi.x) && (tidY < finePatch.hi.y) ) { // finePatch boundary check
      #pragma unroll
      for (int z = finePatch.lo.z; z < finePatch.hi.z; z++) { // loop through z slices

        GPUIntVector origin = make_int3(tidX, tidY, z);  // for each thread

/*`==========TESTING==========*/
#if 0
        if( !isDbgCellDevice( origin ) ){
          return;
        }
     printf(" origin[%i,%i,%i] finePatchID: %i \n", origin.x, origin.y, origin.z, finePatch.ID);
#endif
/*===========TESTING==========`*/

        double sumI = 0;

        //__________________________________
        // ray loop
        #pragma unroll
        for (int iRay = 0; iRay < RT_flags.nDivQRays; iRay++) {

          GPUVector ray_direction = findRayDirectionDevice( randNumStates );

          GPUVector ray_location = rayLocationDevice( randNumStates, origin, d_levels[fineL].DyDx, d_levels[fineL].DzDx , RT_flags.CCRays );

          updateSumI_MLDevice<T>(ray_direction, ray_location, origin, gridP, 
                                 fineLevel_ROI_Lo, fineLevel_ROI_Hi,
                                 regionLo, regionHi,
                                 sigmaT4OverPi, abskg, cellType, sumI, randNumStates, RT_flags);
        } //Ray loop

        //__________________________________
        //  Compute divQ
        divQ[origin] = 4.0 * M_PI * abskg[fineL][origin] * ( sigmaT4OverPi[fineL][origin] - (sumI/RT_flags.nDivQRays) );

        // radiationVolq is the incident energy per cell (W/m^3) and is necessary when particle heat transfer models (i.e. Shaddix) are used
        radiationVolQ[origin] = 4.0 * M_PI * abskg[fineL][origin] *  (sumI/RT_flags.nDivQRays) ;


/*`==========TESTING==========*/
#if DEBUG == 1
       if( isDbgCellDevice(origin) ){
          printf( "\n      [%d, %d, %d]  sumI: %g  divQ: %g radiationVolq: %g  abskg: %g,    sigmaT4: %g \n",
                    origin.x, origin.y, origin.z, sumI,divQ[origin], radiationVolQ[origin],abskg[fineL][origin], sigmaT4OverPi[fineL][origin]);
       }
#endif
/*===========TESTING==========`*/

      }  // end z-slice loop
    }  // end ROI loop
  }  // solve divQ
#endif

}

//______________________________________________________________________
//
//______________________________________________________________________
__device__ GPUVector findRayDirectionDevice( hiprandState* randNumStates )
{
  // Random Points On Sphere
  // add fuzz to prevent infs in 1/dirVector calculation
  double plusMinus_one = 2.0 * randDblExcDevice( randNumStates ) - 1.0 + DBL_EPSILON;
  double r = sqrt(1.0 - plusMinus_one * plusMinus_one);             // Radius of circle at z
  double theta = 2.0 * M_PI * randDblExcDevice( randNumStates );    // Uniform betwen 0-2Pi

  GPUVector dirVector;
  dirVector.x = r*cos(theta);   // Convert to cartesian coordinates
  dirVector.y = r*sin(theta);
  dirVector.z = plusMinus_one;

  return dirVector;
}

//______________________________________________________________________
// Compute the Ray direction from a cell face
__device__ void rayDirection_cellFaceDevice( hiprandState* randNumStates,
                                             const GPUIntVector& origin,
                                             const GPUIntVector& indexOrder,
                                             const GPUIntVector& signOrder,
                                             const int iRay,
                                             GPUVector& directionVector,
                                             double& cosTheta )
{
  // Surface Way to generate a ray direction from the positive z face
  double phi = 2 * M_PI * randDevice(randNumStates);  // azimuthal angle.  Range of 0 to 2pi
  double theta = acos(randDevice(randNumStates));     // polar angle for the hemisphere
  cosTheta = cos(theta);
  double sinTheta = sin(theta);

  //Convert to Cartesian
  GPUVector tmp;
  tmp[0] = sinTheta * cos(phi);
  tmp[1] = sinTheta * sin(phi);
  tmp[2] = cosTheta;

  // Put direction vector as coming from correct face,
  directionVector[0] = tmp[indexOrder[0]] * signOrder[0];
  directionVector[1] = tmp[indexOrder[1]] * signOrder[1];
  directionVector[2] = tmp[indexOrder[2]] * signOrder[2];
}


//______________________________________________________________________
//
__device__ GPUVector rayLocationDevice( hiprandState* randNumStates,
                                        const GPUIntVector origin,
                                        const double DyDx,
                                        const double DzDx,
                                        const bool useCCRays )
{
  GPUVector location;
  if (useCCRays == false) {
    location.x = (double)origin.x + randDevice(randNumStates);
    location.y = (double)origin.y + randDevice(randNumStates) * DyDx;
    location.z = (double)origin.z + randDevice(randNumStates) * DzDx;
  }
  else {
    location.x = origin.x + 0.5;
    location.y = origin.y + 0.5 * DyDx;
    location.z = origin.z + 0.5 * DzDx;
  }
  return location;
}

//______________________________________________________________________
//  Compute the Ray location from a cell face
__device__ void rayLocation_cellFaceDevice( hiprandState* randNumStates,
                                            const GPUIntVector& origin,
                                            const GPUIntVector &indexOrder,
                                            const GPUIntVector &shift,
                                            const double &DyDx,
                                            const double &DzDx,
                                            GPUVector& location )
{
  GPUVector tmp;
  tmp[0] = randDevice(randNumStates);
  tmp[1] = 0;
  tmp[2] = randDevice(randNumStates) * DzDx;

  // Put point on correct face
  location[0] = tmp[indexOrder[0]] + (double)shift[0];
  location[1] = tmp[indexOrder[1]] + (double)shift[1] * DyDx;
  location[2] = tmp[indexOrder[2]] + (double)shift[2] * DzDx;

  location[0] += (double)origin.x;
  location[1] += (double)origin.y;
  location[2] += (double)origin.z;
}

//______________________________________________________________________
//
__device__ bool has_a_boundaryDevice(const GPUIntVector &c,
                                     const GPUGridVariable<int>& celltype,
                                     BoundaryFaces &boundaryFaces){

  GPUIntVector adj = c;
  bool hasBoundary = false;

  adj[0] = c[0] - 1;     // west

  if ( celltype[adj]+1 ){              // cell type of flow is -1, so when cellType+1 isn't false, we
    boundaryFaces.addFace( WEST );     // know we're at a boundary
    hasBoundary = true;
  }

  adj[0] += 2;           // east

  if ( celltype[adj]+1 ){
    boundaryFaces.addFace( EAST );
    hasBoundary = true;
  }

  adj[0] -= 1;
  adj[1] = c[1] - 1;     // south

  if ( celltype[adj]+1 ){
    boundaryFaces.addFace( SOUTH );
    hasBoundary = true;
  }

  adj[1] += 2;           // north

  if ( celltype[adj]+1 ){
    boundaryFaces.addFace( NORTH );
    hasBoundary = true;
  }

  adj[1] -= 1;
  adj[2] = c[2] - 1;     // bottom

  if ( celltype[adj]+1 ){
    boundaryFaces.addFace( BOT );
    hasBoundary = true;
  }

  adj[2] += 2;           // top

  if ( celltype[adj]+1 ){
    boundaryFaces.addFace( TOP );
    hasBoundary = true;
  }

  return (hasBoundary);
}


//______________________________________________________________________
//
__device__ void findStepSizeDevice(int step[],
                                   bool sign[],
                                   const GPUVector&
                                   inv_direction_vector)
{
  // get new step and sign
  for ( int d= 0; d<3; d++ ){

    if (inv_direction_vector[d]>0){
      step[d] = 1;
      sign[d] = 1;
    }else{
      step[d] = -1;
      sign[d] = 0;
    }
  }
}

//______________________________________________________________________
//
__device__ bool containsCellDevice( GPUIntVector low,
                                    GPUIntVector high,
                                    GPUIntVector cell,
                                    const int dir)
{
  return  low[dir] <= cell[dir] &&
          high[dir] > cell[dir];
}

//______________________________________________________________________
//
__device__ void reflect(double& fs,
                         GPUIntVector& cur,
                         GPUIntVector& prevCell,
                         const double abskg,
                         bool& in_domain,
                         int& step,
                         bool& sign,
                         double& ray_direction)
{
  fs = fs * (1 - abskg);

  //put cur back inside the domain
  cur = prevCell;
  in_domain = true;

  // apply reflection condition
  step *= -1;                // begin stepping in opposite direction
  sign = (sign==1) ? 0 : 1;  //  swap sign from 1 to 0 or vice versa
  ray_direction *= -1;
}

//______________________________________________________________________
template< class T >
__device__ void updateSumIDevice ( GPUVector& ray_direction,
                                   GPUVector& ray_location,
                                   const GPUIntVector& origin,
                                   const GPUVector& Dx,
                                   const GPUGridVariable< T >& sigmaT4OverPi,
                                   const GPUGridVariable< T >& abskg,
                                   const GPUGridVariable<int>& celltype,
                                   double& sumI,
                                   hiprandState* randNumStates,
                                   RMCRT_flags RT_flags)

{


  GPUIntVector cur = origin;
  GPUIntVector prevCell = cur;
  // Step and sign for ray marching
  int step[3];                                          // Gives +1 or -1 based on sign
  bool sign[3];

  GPUVector inv_ray_direction = 1.0/ray_direction;
/*`==========TESTING==========*/
#if DEBUG == 1
  if( isDbgCellDevice(origin) ) {
    printf("        updateSumI: [%d,%d,%d] ray_dir [%g,%g,%g] ray_loc [%g,%g,%g]\n", origin.x, origin.y, origin.z,ray_direction.x, ray_direction.y, ray_direction.z, ray_location.x, ray_location.y, ray_location.z);
    printf("        inv_ray_dir [%g,%g,%g]\n", inv_ray_direction.x,inv_ray_direction.y,inv_ray_direction.z);
  }
#endif
/*===========TESTING==========`*/

  findStepSizeDevice(step, sign, inv_ray_direction);
  GPUVector D_DxRatio = make_double3(1, Dx.y/Dx.x, Dx.z/Dx.x );

  GPUVector tMax;         // (mixing bools, ints and doubles)
  tMax.x = (origin.x + sign[0]               - ray_location.x) * inv_ray_direction.x ;
  tMax.y = (origin.y + sign[1] * D_DxRatio.y - ray_location.y) * inv_ray_direction.y ;
  tMax.z = (origin.z + sign[2] * D_DxRatio.z - ray_location.z) * inv_ray_direction.z ;

  //Length of t to traverse one cell
  GPUVector tDelta;
  tDelta   = Abs(inv_ray_direction) * D_DxRatio;

  //Initializes the following values for each ray
  bool in_domain     = true;
  double tMax_prev   = 0;
  double intensity   = 1.0;
  double fs          = 1.0;
  int nReflect       = 0;                 // Number of reflections
  double optical_thickness      = 0;
  double expOpticalThick_prev   = 1.0;


#ifdef RAY_SCATTER
  double scatCoeff = RT_flags.sigmaScat;          //[m^-1]  !! HACK !! This needs to come from data warehouse
  if (scatCoeff == 0) scatCoeff = 1e-99;  // avoid division by zero

  // Determine the length at which scattering will occur
  // See CCA/Components/Arches/RMCRT/PaulasAttic/MCRT/ArchesRMCRT/ray.cc
  double scatLength = -log( randDblExcDevice( randNumStates ) ) / scatCoeff;
  double curLength = 0;
#endif

  //+++++++Begin ray tracing+++++++++++++++++++
  //Threshold while loop
  while ( intensity > RT_flags.threshold ){

    DIR dir = NONE;

    while (in_domain){

      prevCell = cur;
      double disMin = -9;          // Represents ray segment length.

      //__________________________________
      //  Determine which cell the ray will enter next
      if ( tMax.x < tMax.y ){        // X < Y
        if ( tMax.x < tMax.z ){      // X < Z
          dir = X;
        } else {
          dir = Z;
        }
      } else {
        if( tMax.y < tMax.z ){       // Y < Z
          dir = Y;
        } else {
          dir = Z;
        }
      }

      //__________________________________
      //  update marching variables
      cur[dir]  = cur[dir] + step[dir];
      disMin    = (tMax[dir] - tMax_prev);
      tMax_prev = tMax[dir];
      tMax[dir] = tMax[dir] + tDelta[dir];

      ray_location.x = ray_location.x + (disMin  * ray_direction.x);
      ray_location.y = ray_location.y + (disMin  * ray_direction.y);
      ray_location.z = ray_location.z + (disMin  * ray_direction.z);

/*`==========TESTING==========*/
#if DEBUG == 1
if( isDbgCellDevice(origin) ){
    printf( "            cur [%d,%d,%d] prev [%d,%d,%d] ", cur.x, cur.y, cur.z, prevCell.x, prevCell.y, prevCell.z);
    printf( " dir %d ", dir );
    printf( "tMax [%g,%g,%g] ",tMax.x,tMax.y, tMax.z);
    printf( "rayLoc [%g,%g,%g] ",ray_location.x,ray_location.y, ray_location.z);
    printf( "inv_dir [%g,%g,%g] ",inv_ray_direction.x,inv_ray_direction.y, inv_ray_direction.z);
    printf( "disMin %g \n",disMin );

    printf( "            abskg[prev] %g  \t sigmaT4OverPi[prev]: %g \n",abskg[prevCell],  sigmaT4OverPi[prevCell]);
    printf( "            abskg[cur]  %g  \t sigmaT4OverPi[cur]:  %g  \t  cellType: %i\n",abskg[cur], sigmaT4OverPi[cur], celltype[cur] );
}
#endif

/*===========TESTING==========`*/
      in_domain = (celltype[cur]==-1);  //cellType of -1 is flow

      optical_thickness += Dx.x * abskg[prevCell]*disMin; // as long as tDeltaY,Z tMax.y(),Z and ray_location[1],[2]..
      // were adjusted by DyDx  or DzDx, this line is now correct for noncubic domains.

      RT_flags.nRaySteps ++;

      //Eqn 3-15(see below reference) while
      //Third term inside the parentheses is accounted for in Inet. Chi is accounted for in Inet calc.
      double expOpticalThick = exp(-optical_thickness);

      sumI += sigmaT4OverPi[prevCell] * ( expOpticalThick_prev - expOpticalThick ) * fs;

      expOpticalThick_prev = expOpticalThick;


#ifdef RAY_SCATTER
      curLength += disMin * Dx.x;
      if ( (curLength > scatLength) && in_domain){

        // get new scatLength for each scattering event
        scatLength = -log( randDblExcDevice( randNumStates ) ) / scatCoeff;

        ray_direction     = findRayDirectionDevice( randNumStates );

        inv_ray_direction = 1.0/ray_direction;

        // get new step and sign
        int stepOld = step[dir];
        findStepSizeDevice( step, sign, inv_ray_direction);

        // if sign[face] changes sign, put ray back into prevCell (back scattering)
        // a sign change only occurs when the product of old and new is negative
        if( step[dir] * stepOld < 0 ){
          cur = prevCell;
        }

        // get new tMax (mixing bools, ints and doubles)
        tMax.x = ( ( cur.x + sign[0]               - ray_location.x) * inv_ray_direction.x );
        tMax.y = ( ( cur.y + sign[1] * D_DxRatio.y - ray_location.y) * inv_ray_direction.y );
        tMax.z = ( ( cur.z + sign[2] * D_DxRatio.z - ray_location.z) * inv_ray_direction.z );

        // Length of t to traverse one cell
        tDelta    = Abs(inv_ray_direction) * D_DxRatio;
        tMax_prev = 0;
        curLength = 0;  // allow for multiple scattering events per ray

/*`==========TESTING==========*/
#if DEBUG == 3
        printf( "%i, %i, %i, tmax: %g, %g, %g  tDelta: %g, %g, %g \n", cur.x, cur.y, cur.z, tMax.x, tMax.y, tMax.z, tDelta.x, tDelta.y , tDelta.z );
#endif
/*===========TESTING==========`*/

      }
#endif

    } //end domain while loop.  ++++++++++++++

    //  wall emission 12/15/11
    double wallEmissivity = abskg[cur];

    if (wallEmissivity > 1.0){       // Ensure wall emissivity doesn't exceed one.
      wallEmissivity = 1.0;
    }

    intensity = exp(-optical_thickness);

    sumI += wallEmissivity * sigmaT4OverPi[cur] * intensity;

    intensity = intensity * fs;


    // when a ray reaches the end of the domain, we force it to terminate.
    if( !RT_flags.allowReflect ) intensity = 0;


/*`==========TESTING==========*/
#if DEBUG == 1
if( isDbgCellDevice(origin) ){
    printf( "            cur [%d,%d,%d] intensity: %g expOptThick: %g, fs: %g allowReflect: %i \n",
            cur.x, cur.y, cur.z, intensity,  exp(-optical_thickness), fs,RT_flags.allowReflect );

}
__syncthreads();
#endif
/*===========TESTING==========`*/
    //__________________________________
    //  Reflections
    if ( (intensity > RT_flags.threshold) && RT_flags.allowReflect){
      reflect( fs, cur, prevCell, abskg[cur], in_domain, step[dir], sign[dir], ray_direction[dir]);
      ++nReflect;
    }

  }  // threshold while loop.
} // end of updateSumI function

//______________________________________________________________________
//  Multi-level
 template< class T>
 __device__ void updateSumI_MLDevice (  GPUVector& ray_direction,
                                        GPUVector& ray_location,
                                        const GPUIntVector& origin,
                                        gridParams gridP,
                                        const GPUIntVector& fineLevel_ROI_Lo,
                                        const GPUIntVector& fineLevel_ROI_Hi,
                                        const int3* regionLo,
                                        const int3* regionHi,
                                        const GPUGridVariable< T >* sigmaT4OverPi,
                                        const GPUGridVariable< T >* abskg,
                                        const GPUGridVariable<int>* cellType,
                                        double& sumI,
                                        hiprandState* randNumStates,
                                        RMCRT_flags RT_flags )
{
  /*`==========TESTING==========*/
#if DEBUG == 1
  if( isDbgCellDevice(origin) ) {
    printf("        A) updateSumI_ML: [%d,%d,%d] ray_dir [%g,%g,%g] ray_loc [%g,%g,%g]\n", origin.x, origin.y, origin.z,ray_direction.x, ray_direction.y, ray_direction.z, ray_location.x, ray_location.y, ray_location.z);
  }
#endif
  /*===========TESTING==========`*/
  int maxLevels = gridP.maxLevels;   // for readability
  int L = maxLevels - 1;       // finest level
  int prevLev = L;

  GPUIntVector cur = origin;
  GPUIntVector prevCell = cur;
  // Step and sign for ray marching
  int step[3];                                          // Gives +1 or -1 based on sign
  bool sign[3];

  GPUVector inv_ray_direction = 1.0 / ray_direction;
  findStepSizeDevice(step, sign, inv_ray_direction);

  //__________________________________
  // define tMax & tDelta on all levels
  // go from finest to coarset level so you can compare
  // with 1L rayTrace results.
  GPUVector tMax;         // (mixing bools, ints and doubles)
  tMax.x = (origin.x + sign[0] - ray_location.x) * inv_ray_direction.x;
  tMax.y = (origin.y + sign[1] * d_levels[L].DyDx - ray_location.y) * inv_ray_direction.y;
  tMax.z = (origin.z + sign[2] * d_levels[L].DzDx - ray_location.z) * inv_ray_direction.z;

  GPUVector tDelta[d_MAXLEVELS];
  for (int Lev = maxLevels - 1; Lev > -1; Lev--) {
    //Length of t to traverse one cell
    tDelta[Lev].x = fabs(inv_ray_direction[0]);
    tDelta[Lev].y = fabs(inv_ray_direction[1]) * d_levels[Lev].DyDx;
    tDelta[Lev].z = fabs(inv_ray_direction[2]) * d_levels[Lev].DzDx;
  }

  //Initializes the following values for each ray
  bool in_domain = true;
  double tMax_prev = 0;
  double intensity = 1.0;
  double fs = 1.0;
  int nReflect = 0;                 // Number of reflections
  double optical_thickness = 0;
  double expOpticalThick_prev = 1.0;
  bool onFineLevel = true;

  //______________________________________________________________________
  //  Threshold  loop

  while (intensity > RT_flags.threshold) {

    DIR dir = NONE;

    while (in_domain) {

      prevCell = cur;
      prevLev = L;
      double disMin = -9;          // Represents ray segment length.

      //__________________________________
      //  Determine which cell the ray will enter next
      if (tMax.x < tMax.y) {        // X < Y
        if (tMax.x < tMax.z) {      // X < Z
          dir = X;
        }
        else {
          dir = Z;
        }
      }
      else {
        if (tMax.y < tMax.z) {       // Y < Z
          dir = Y;
        }
        else {
          dir = Z;
        }
      }

      // next cell index and position
      cur[dir] = cur[dir] + step[dir];
      GPUVector dx_prev = d_levels[L].Dx;           //  Used to compute coarsenRatio
      //__________________________________
      // Logic for moving between levels
      //  - Currently you can only move from fine to coarse level
      //  - Don't jump levels if ray is at edge of domain
      
      GPUPoint pos = d_levels[L].getCellPosition(cur);         // position could be outside of domain
      in_domain = gridP.domain_BB.inside(pos);

      //in_domain = (cellType[L][cur] == d_flowCell);    // use this when direct comparison with 1L resullts      
      
      bool ray_outside_ROI    = ( containsCellDevice(fineLevel_ROI_Lo, fineLevel_ROI_Hi, cur, dir) == false );
      bool ray_outside_Region = ( containsCellDevice(regionLo[L], regionHi[L], cur, dir) == false );
      
      bool jumpFinetoCoarserLevel   = ( onFineLevel &&  ray_outside_ROI && in_domain );
      bool jumpCoarsetoCoarserLevel = ( (onFineLevel == false) && ray_outside_Region && (L > 0) && in_domain );

#if (DEBUG == 1 || DEBUG == 4)
      if( isDbgCellDevice(origin) ) {
        printf( "        Ray: [%i,%i,%i] **jumpFinetoCoarserLevel %i jumpCoarsetoCoarserLevel %i containsCell: %i ", cur.x, cur.y, cur.z, jumpFinetoCoarserLevel, jumpCoarsetoCoarserLevel,
            containsCellDevice(fineLevel_ROI_Lo, fineLevel_ROI_Hi, cur, dir));
        printf( " onFineLevel: %i ray_outside_ROI: %i ray_outside_Region: %i in_domain: %i\n", onFineLevel, ray_outside_ROI, ray_outside_Region,in_domain );
        printf( " L: %i regionLo: [%i,%i,%i], regionHi: [%i,%i,%i]\n",L,regionLo[L].x,regionLo[L].y,regionLo[L].z, regionHi[L].x,regionHi[L].y,regionHi[L].z); 
      }
#endif

      if (jumpFinetoCoarserLevel) {
        cur = d_levels[L].mapCellToCoarser(cur);
        L = d_levels[L].getCoarserLevelIndex();      // move to a coarser level
        onFineLevel = false;

#if (DEBUG == 1 || DEBUG == 4)
        if( isDbgCellDevice(origin) ) {
          printf( "        ** Jumping off fine patch switching Levels:  prev L: %i, L: %i, cur: [%i,%i,%i] \n",prevLev, L, cur.x, cur.y, cur.z);
        }
#endif

      }
      else if (jumpCoarsetoCoarserLevel) {
        GPUIntVector c_old = cur;                     // needed for debugging
        cur = d_levels[L].mapCellToCoarser(cur);
        L = d_levels[L].getCoarserLevelIndex();      // move to a coarser level
#if (DEBUG == 1 || DEBUG == 4)
        if( isDbgCellDevice(origin) ) {
          printf( "        ** Switching Levels:  prev L: %i, L: %i, cur: [%i,%i,%i], c_old: [%i,%i,%i]\n",prevLev, L, cur.x, cur.y, cur.z, c_old.x, c_old.y, c_old.z);
        }
#endif
      }


      //__________________________________
      //  update marching variables
      disMin = (tMax[dir] - tMax_prev);
      tMax_prev = tMax[dir];
      tMax[dir] = tMax[dir] + tDelta[L][dir];

      ray_location.x = ray_location.x + (disMin * ray_direction.x);
      ray_location.y = ray_location.y + (disMin * ray_direction.y);
      ray_location.z = ray_location.z + (disMin * ray_direction.z);

      //__________________________________
      // Account for uniqueness of first step after reaching a new level
      GPUVector dx = d_levels[L].Dx;
      GPUIntVector coarsenRatio = GPUIntVector(make_int3(1, 1, 1));

      coarsenRatio[0] = dx[0] / dx_prev[0];
      coarsenRatio[1] = dx[1] / dx_prev[1];
      coarsenRatio[2] = dx[2] / dx_prev[2];

      GPUVector lineup;
      for (int ii = 0; ii < 3; ii++) {
        if (sign[ii]) {
          lineup[ii] = -(cur[ii] % coarsenRatio[ii] - (coarsenRatio[ii] - 1));
        }
        else {
          lineup[ii] = cur[ii] % coarsenRatio[ii];
        }
      }

      tMax += lineup * tDelta[prevLev];

      /*`==========TESTING==========*/
#if DEBUG == 1
      if( isDbgCellDevice(origin) ) {
        printf( "        B) cur [%i,%i,%i] prev [%i,%i,%i]", cur.x, cur.y, cur.z, prevCell.x, prevCell.y, prevCell.z);
        printf( " dir %i ", dir );
        printf( " stepSize [%i,%i,%i] ",step[0],step[1],step[2]);
        printf( " tMax [%g,%g,%g] ",tMax.x,tMax.y, tMax.z);
        printf( "rayLoc [%g,%g,%g] ", ray_location.x,ray_location.y, ray_location.z);
        printf( "inv_dir [%g,%g,%g] ",inv_ray_direction.x,inv_ray_direction.y, inv_ray_direction.z);
        printf( "disMin %g inDomain %i\n",disMin, in_domain );

        printf( "            abskg[prev] %g  \t sigmaT4OverPi[prev]: %g \n",abskg[prevLev][prevCell], sigmaT4OverPi[prevLev][prevCell]);
        printf( "            abskg[cur]  %g  \t sigmaT4OverPi[cur]:  %g  \t  cellType: %i \n",abskg[L][cur], sigmaT4OverPi[L][cur], cellType[L][cur]);
        printf( "            Dx[prevLev].x  %g \n", d_levels[prevLev].Dx.x);
      }
#endif
      /*===========TESTING==========`*/
      optical_thickness += d_levels[prevLev].Dx.x * abskg[prevLev][prevCell] * disMin;

      double expOpticalThick = exp(-optical_thickness);

      sumI += sigmaT4OverPi[prevLev][prevCell] * (expOpticalThick_prev - expOpticalThick) * fs;

      expOpticalThick_prev = expOpticalThick;

    }  //end domain while loop.  ++++++++++++++
    //__________________________________
    //
    double wallEmissivity = abskg[L][cur];

    if (wallEmissivity > 1.0) {       // Ensure wall emissivity doesn't exceed one.
      wallEmissivity = 1.0;
    }

    intensity = exp(-optical_thickness);

    sumI += wallEmissivity * sigmaT4OverPi[L][cur] * intensity;

    intensity = intensity * fs;

    // when a ray reaches the end of the domain, we force it to terminate.
    if (!RT_flags.allowReflect)
      intensity = 0;

    /*`==========TESTING==========*/
#if DEBUG == 1
    if( isDbgCellDevice(origin) ) {
      printf( "        C) intensity: %g OptThick: %g, fs: %g allowReflect: %i\n", intensity, optical_thickness, fs, RT_flags.allowReflect );
    }
#endif
    /*===========TESTING==========`*/
    //__________________________________
    //  Reflections
    if ((intensity > RT_flags.threshold) && RT_flags.allowReflect) {
      reflect(fs, cur, prevCell, abskg[L][cur], in_domain, step[dir], sign[dir], ray_direction[dir]);
      ++nReflect;
    }
  }  // threshold while loop.
}  // end of updateSumI function

//---------------------------------------------------------------------------
// Returns random number between 0 & 1.0 including 0 & 1.0
// See src/Core/Math/MersenneTwister.h for equation
//---------------------------------------------------------------------------
__device__ double randDevice(hiprandState* globalState)
{
  int tid = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
  hiprandState localState = globalState[tid];
  double val = hiprand(&localState);
  globalState[tid] = localState;

#ifdef FIXED_RANDOM_NUM
  return 0.3;
#else
  return (double)val * (1.0/4294967295.0);
#endif

}

//---------------------------------------------------------------------------
// Returns random number between 0 & 1.0 excluding 0 & 1.0
// See src/Core/Math/MersenneTwister.h for equation
//---------------------------------------------------------------------------
__device__ double randDblExcDevice(hiprandState* globalState)
{
  int tid = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;

  hiprandState localState = globalState[tid];
  double val = hiprand(&localState);
  globalState[tid] = localState;

#ifdef FIXED_RANDOM_NUM
  return 0.3;
#else
  return ( (double)val + 0.5 ) * (1.0/4294967296.0);
#endif
}

//______________________________________________________________________
//  Each thread gets same seed, a different sequence number, no offset
//  This will create repeatable results.
__global__ void setupRandNumKernel(hiprandState* randNumStates)
{
  int tID = threadIdx.x +  blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
  hiprand_init(1234, tID, 0, &randNumStates[tID]);
}

//______________________________________________________________________
//  is cell a debug cell
__device__ bool isDbgCellDevice( GPUIntVector me )
{
  int size = 1;  
  GPUIntVector dbgCell[1];
  dbgCell[0] = make_int3(10,10,10);
  
 
  
  for (int i = 0; i < size; i++) {
    if( me == dbgCell[i]){
      return true;
    }
  }
  return false;
}
//______________________________________________________________________
//   Perform some sanity checks on the Variable.  This is for debugging
template< class T>
__device__ void GPUVariableSanityCK(const GPUGridVariable<T>& Q,
                                    const GPUIntVector Lo,
                                    const GPUIntVector Hi)
{
#if SCI_ASSERTION_LEVEL > 0
  if (isThread0()) {
    GPUIntVector varLo = Q.getLowIndex();
    GPUIntVector varHi = Q.getHighIndex();
    
    if( Lo < varLo || varHi < Hi){
      printf ( "ERROR: GPUVariableSanityCK \n");
      printf("  Variable:          varLo:[%i,%i,%i], varHi[%i,%i,%i]\n", varLo.x, varLo.y, varLo.z, varHi.x, varHi.y, varHi.z);
      printf("  Requested extents: varLo:[%i,%i,%i], varHi[%i,%i,%i]\n", Lo.x, Lo.y, Lo.z, Hi.x, Hi.y, Hi.z);
      printf(" Now existing...");
      __threadfence();
      asm("trap;");
    }

    for (int i = Lo.x; i < Hi.x; i++) {
      for (int j = Lo.y; j < Hi.y; j++) {
        for (int k = Lo.z; k < Hi.z; k++) {
          GPUIntVector idx = make_int3(i, j, k);
          T me = Q[idx];
          if ( isnan(me) || isinf(me)){
            printf ( "isNan or isInf was detected at [%i,%i,%i]\n", i,j,k);
            printf(" Now existing...");
            __threadfence();
            asm("trap;");
          }
          
        }  // k loop
      }  // j loop
    }  // i loop
  }  // thread0
#endif
}
template
__device__ void GPUVariableSanityCK(const GPUGridVariable<float>& Q,
                                    const GPUIntVector Lo,
                                    const GPUIntVector Hi);
template
__device__ void GPUVariableSanityCK(const GPUGridVariable<double>& Q,
                                    const GPUIntVector Lo,
                                    const GPUIntVector Hi);
//______________________________________________________________________
//
template< class T>
__host__ void launchRayTraceKernel(dim3 dimGrid,
                                   dim3 dimBlock,
                                   const int matlIndx,
                                   const int levelIndx,
                                   patchParams patch,
                                   hipStream_t* stream,
                                   RMCRT_flags RT_flags,
                                   varLabelNames* labelNames,
                                   GPUDataWarehouse* abskg_gdw,
                                   GPUDataWarehouse* sigmaT4_gdw,
                                   GPUDataWarehouse* cellType_gdw,
                                   GPUDataWarehouse* old_gdw,
                                   GPUDataWarehouse* new_gdw)
{
  // setup random number generator states on the device, 1 for each thread
  hiprandState* randNumStates;
  int numStates = dimGrid.x * dimGrid.y * dimBlock.x * dimBlock.y * dimBlock.z;
  CUDA_RT_SAFE_CALL( hipMalloc((void**)&randNumStates, numStates * sizeof(hiprandState)) );


  setupRandNumKernel<<< dimGrid, dimBlock>>>( randNumStates );

  rayTraceKernel< T ><<< dimGrid, dimBlock, 0, *stream >>>( dimGrid,
                                                            dimBlock,
                                                            matlIndx,
                                                            levelIndx,
                                                            patch,
                                                            randNumStates,
                                                            RT_flags,
                                                            labelNames,
                                                            abskg_gdw,
                                                            sigmaT4_gdw,
                                                            cellType_gdw,
                                                            old_gdw,
                                                            new_gdw);
    // free device-side RNG states
    CUDA_RT_SAFE_CALL( hipFree(randNumStates) );
}

//______________________________________________________________________
//
template< class T>
__host__ void launchRayTraceDataOnionKernel( dim3 dimGrid,
                                             dim3 dimBlock,
                                             int matlIndex,
                                             patchParams patch,
                                             gridParams gridP,
                                             levelParams* levelP,
                                             GPUIntVector fineLevel_ROI_Lo,
                                             GPUIntVector fineLevel_ROI_Hi,
                                             hipStream_t* stream,
                                             RMCRT_flags RT_flags,
                                             GPUDataWarehouse* abskg_gdw,
                                             GPUDataWarehouse* sigmaT4_gdw,
                                             GPUDataWarehouse* cellType_gdw,
                                             GPUDataWarehouse* old_gdw,
                                             GPUDataWarehouse* new_gdw )
{  
  // copy regionLo & regionHi to device memory
  int maxLevels = gridP.maxLevels;
  
  int3* dev_regionLo;
  int3* dev_regionHi;
  size_t size = d_MAXLEVELS *  sizeof(int3);
  CUDA_RT_SAFE_CALL( hipMalloc( (void**)& dev_regionLo, size) );
  CUDA_RT_SAFE_CALL( hipMalloc( (void**)& dev_regionHi, size) );
  
  int3 myLo[d_MAXLEVELS];
  int3 myHi[d_MAXLEVELS];
  for (int l = 0; l < maxLevels; ++l) {
    myLo[l] = levelP[l].regionLo;        // never use levelP regionLo or hi in the kernel.
    myHi[l] = levelP[l].regionHi;        // They are different on each patch
  }
  
  CUDA_RT_SAFE_CALL( hipMemcpy( dev_regionLo, myLo, size, hipMemcpyHostToDevice) );
  CUDA_RT_SAFE_CALL( hipMemcpy( dev_regionHi, myHi, size, hipMemcpyHostToDevice) );  
  

  //__________________________________
  // copy levelParams array to constant memory on device
  CUDA_RT_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_levels), levelP, (maxLevels * sizeof(levelParams))));

  //__________________________________
  // setup random number generator states on the device, 1 for each thread
  hiprandState* randNumStates;
  int numStates = dimGrid.x * dimGrid.y * dimBlock.x * dimBlock.y * dimBlock.z;
  CUDA_RT_SAFE_CALL( hipMalloc((void**)&randNumStates, (numStates * sizeof(hiprandState))) );

  setupRandNumKernel<<< dimGrid, dimBlock>>>( randNumStates );

  rayTraceDataOnionKernel< T ><<< dimGrid, dimBlock, 0, *stream >>>( dimGrid,
                                                                     dimBlock,
                                                                     matlIndex,
                                                                     patch,
                                                                     gridP,
                                                                     fineLevel_ROI_Lo,
                                                                     fineLevel_ROI_Hi,
                                                                     dev_regionLo,
                                                                     dev_regionHi,
                                                                     randNumStates,
                                                                     RT_flags,
                                                                     abskg_gdw,
                                                                     sigmaT4_gdw,
                                                                     cellType_gdw,
                                                                     old_gdw,
                                                                     new_gdw);
  // free device-side RNG states
  CUDA_RT_SAFE_CALL( hipFree(randNumStates) );
  CUDA_RT_SAFE_CALL( hipFree(dev_regionLo) );
  CUDA_RT_SAFE_CALL( hipFree(dev_regionHi) );

}

//______________________________________________________________________
//  Explicit template instantiations

template
__host__ void launchRayTraceKernel<double>( dim3 dimGrid,
                                            dim3 dimBlock,
                                            const int matlIndx,
                                            const int levelIndx,
                                            patchParams patch,
                                            hipStream_t* stream,
                                            RMCRT_flags RT_flags,
                                            varLabelNames* labelNames,
                                            GPUDataWarehouse* abskg_gdw,
                                            GPUDataWarehouse* sigmaT4_gdw,
                                            GPUDataWarehouse* cellType_gdw,
                                            GPUDataWarehouse* old_gdw,
                                            GPUDataWarehouse* new_gdw );

//______________________________________________________________________
//
template
__host__ void launchRayTraceKernel<float>( dim3 dimGrid,
                                           dim3 dimBlock,
                                           const int matlIndx,
                                           const int levelIndx,
                                           patchParams patch,
                                           hipStream_t* stream,
                                           RMCRT_flags RT_flags,
                                           varLabelNames* labelNames,
                                           GPUDataWarehouse* abskg_gdw,
                                           GPUDataWarehouse* sigmaT4_gdw,
                                           GPUDataWarehouse* celltype_gdw,
                                           GPUDataWarehouse* old_gdw,
                                           GPUDataWarehouse* new_gdw );

//______________________________________________________________________
//
template
__host__ void launchRayTraceDataOnionKernel<double>( dim3 dimGrid,
                                                     dim3 dimBlock,
                                                     int matlIndex,
                                                     patchParams patch,
                                                     gridParams gridP,
                                                     levelParams*  levelP,
                                                     GPUIntVector fineLevel_ROI_Lo,
                                                     GPUIntVector fineLevel_ROI_Hi,
                                                     hipStream_t* stream,
                                                     RMCRT_flags RT_flags,
                                                     GPUDataWarehouse* abskg_gdw,
                                                     GPUDataWarehouse* sigmaT4_gdw,
                                                     GPUDataWarehouse* cellType_gdw,
                                                     GPUDataWarehouse* old_gdw,
                                                     GPUDataWarehouse* new_gdw );

//______________________________________________________________________
//
template
__host__ void launchRayTraceDataOnionKernel<float>( dim3 dimGrid,
                                                    dim3 dimBlock,
                                                    int matlIndex,
                                                    patchParams patch,
                                                    gridParams gridP,
                                                    levelParams* levelP,
                                                    GPUIntVector fineLevel_ROI_Lo,
                                                    GPUIntVector fineLevel_ROI_Hi,
                                                    hipStream_t* stream,
                                                    RMCRT_flags RT_flags,
                                                    GPUDataWarehouse* abskg_gdw,
                                                    GPUDataWarehouse* sigmaT4_gdw,
                                                    GPUDataWarehouse* cellType_gdw,
                                                    GPUDataWarehouse* old_gdw,
                                                    GPUDataWarehouse* new_gdw );

} //end namespace Uintah
